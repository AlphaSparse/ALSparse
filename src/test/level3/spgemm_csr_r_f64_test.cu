
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
double* coo_values;

// parms for kernel
double *hmatB, *matC_ict, *matC_roc;
long long ldb, ldc;
long long B_size, nnz, nnz_c;
const double alpha = 2.f;
const double beta = 3.f;

int trials = 10, warm_up = 2;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  int nnz = rnnz;

  // Offload data to device
  int* dACsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  double* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dACsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dACsrRowPtr);

  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dACsrRowPtr,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_64F));

  int* dBCsrRowPtr = NULL;
  int* dBrow = NULL;
  int* dBcol = NULL;
  double* dBval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dBrow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dBcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dBval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dBCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dBrow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dBcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dBval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dBrow, nnz, A_rows, dBCsrRowPtr);

  hipsparseSpMatDescr_t matB;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matB,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dBCsrRowPtr,
                                   dBcol,
                                   dBval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_64F));

  int* dCCsrRowPtr = NULL;
  int* dCrow = NULL;
  int* dCcol = NULL;
  double* dCval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCrow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dCrow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dCcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dCval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dCrow, nnz, A_rows, dCCsrRowPtr);

  hipsparseSpMatDescr_t matC;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matC,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dCCsrRowPtr,
                                   dCcol,
                                   dCval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_64F));

  size_t bufferSize1 = 0;
  size_t bufferSize2 = 0;
  void* dBuffer1 = NULL;
  void* dBuffer2 = NULL;
  // SpGEMM Computation
  hipsparseSpGEMMDescr_t spgemmDesc;
  CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&spgemmDesc))
  hipEvent_t event_start, event_stop;
  float elapsed_time = 0.0;
  float elapsed_time_work = 0.0;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  // ask bufferSize1 bytes for external memory
  CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha,
                                               matA,
                                               matB,
                                               &beta,
                                               matC,
                                               HIP_R_64F,
                                               HIPSPARSE_SPGEMM_DEFAULT,
                                               spgemmDesc,
                                               &bufferSize1,
                                               NULL))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "cusparseSpGEMM_workEstimation1 time: " << elapsed_time << " microseconds" << std::endl;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  CHECK_CUDA(hipMalloc((void**)&dBuffer1, bufferSize1))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "cusparseSpGEMM_workEstimation1 buffer1 time: " << elapsed_time << " microseconds" << std::endl;
  elapsed_time_work += elapsed_time;
  // inspect the matrices A and B to understand the memory requirement for
  // the next step
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha,
                                               matA,
                                               matB,
                                               &beta,
                                               matC,
                                               HIP_R_64F,
                                               HIPSPARSE_SPGEMM_DEFAULT,
                                               spgemmDesc,
                                               &bufferSize1,
                                               dBuffer1))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  elapsed_time_work += elapsed_time;
  std::cout << "cusparseSpGEMM_workEstimation2 time: " << elapsed_time << " microseconds" << std::endl;
  
  // ask bufferSize2 bytes for external memory
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha,
                                        matA,
                                        matB,
                                        &beta,
                                        matC,
                                        HIP_R_64F,
                                        HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc,
                                        &bufferSize2,
                                        NULL))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "cusparseSpGEMM_compute1 time: " << elapsed_time << " microseconds, buffersize :" << bufferSize2 << std::endl;
  double time1 = get_time_us();
  CHECK_CUDA(hipMalloc((void**)&dBuffer2, bufferSize2))
  double time2 = get_time_us();
  std::cout << "cusparseSpGEMM_compute1 buffer time: " << (time2 - time1) / (1e3) << " microseconds" << std::endl;
  elapsed_time_work += (time2 - time1) / (1e3);
  // compute the intermediate product of A * B
  CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha,
                                        matA,
                                        matB,
                                        &beta,
                                        matC,
                                        HIP_R_64F,
                                        HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc,
                                        &bufferSize2,
                                        dBuffer2))
  // get matrix C non-zero entries C_nnz1
  int64_t C_num_rows1, C_num_cols1, C_nnz1;
  CHECK_CUSPARSE(
    hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1, &C_nnz1))
  nnz_c = C_nnz1;
  // allocate matrix C
  int *dC_csrOffsets, *dC_columns;
  double* dC_values;
  CHECK_CUDA(hipMalloc((void**)&dC_columns, C_nnz1 * sizeof(int)))
  CHECK_CUDA(hipMalloc((void**)&dC_values, C_nnz1 * sizeof(double)))
  CHECK_CUDA(hipMalloc((void**)&dC_csrOffsets, (A_rows + 1) * sizeof(int)))
  // NOTE: if 'beta' != 0, the values of C must be update after the allocation
  //       of dC_values, and before the call of hipsparseSpGEMM_copy
  
  // update matC with the new pointers
  CHECK_CUSPARSE(
    hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values))
  
  // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values

  // copy the final products to the matrix C
  CHECK_CUSPARSE(hipsparseSpGEMM_copy(handle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha,
                                     matA,
                                     matB,
                                     &beta,
                                     matC,
                                     HIP_R_64F,
                                     HIPSPARSE_SPGEMM_DEFAULT,
                                     spgemmDesc))
  // device result check
  matC_roc = (double*)alpha_malloc(C_nnz1 * sizeof(double));
  // std::cout << "C_nnz1: " << C_nnz1 << std::endl;
  CHECK_CUDA(hipMemcpy(
    matC_roc, dC_values, C_nnz1 * sizeof(double), hipMemcpyDeviceToHost))
  int* cst_ptr = (int*)alpha_malloc((A_rows + 1) * sizeof(int));
  int* col_ptr = (int*)alpha_malloc((C_nnz1 + 1) * sizeof(int));
  CHECK_CUDA(hipMemcpy(
    cst_ptr, dC_csrOffsets, (A_rows + 1) * sizeof(int), hipMemcpyDeviceToHost))
  CHECK_CUDA(hipMemcpy(
    col_ptr, dC_columns, (C_nnz1) * sizeof(int), hipMemcpyDeviceToHost))

  std::cout << "cusparse row ptr " <<std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << cst_ptr[i] << ", ";
  }
  std::cout << std::endl;
  std::cout << "cusparse col idx " <<std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << col_ptr[i] << ", ";
  }
  std::cout << std::endl;
  // destroy matrix/vector descriptors
  for (int i = 0; i < warm_up; i++)
  {
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha,
                                          matA,
                                          matB,
                                          &beta,
                                          matC,
                                          HIP_R_64F,
                                          HIPSPARSE_SPGEMM_DEFAULT,
                                          spgemmDesc,
                                          &bufferSize2,
                                          dBuffer2))
  }

  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  for (int i = 0; i < trials; i++)
  {
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha,
                                          matA,
                                          matB,
                                          &beta,
                                          matC,
                                          HIP_R_64F,
                                          HIPSPARSE_SPGEMM_DEFAULT,
                                          spgemmDesc,
                                          &bufferSize2,
                                          dBuffer2))
  }
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "cusparseSpGEMM_compute2 time: " << elapsed_time/trials<< " microseconds" << std::endl;
  std::cout << "cusparseSpGEMM_compute_total time: " << elapsed_time/trials + elapsed_time_work<< " microseconds" << std::endl;
  CHECK_CUSPARSE(hipsparseSpGEMM_destroyDescr(spgemmDesc))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matC))
  CHECK_CUSPARSE(hipsparseDestroy(handle))
  // Clear up on device
  hipFree(dBuffer2);
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dBrow);
  hipFree(dBcol);
  hipFree(dBval);
  hipFree(dCrow);
  hipFree(dCcol);
  hipFree(dCval);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  int nnz = rnnz;

  // Offload data to device
  int* dACsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  double* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dACsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dACsrRowPtr);

  alphasparseSpMatDescr_t matA;
  alphasparseCreateCsr(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dACsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_64F);

  int* dBCsrRowPtr = NULL;
  int* dBrow = NULL;
  int* dBcol = NULL;
  double* dBval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dBrow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dBcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dBval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dBCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dBrow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dBcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dBval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dBrow, nnz, A_rows, dBCsrRowPtr);

  alphasparseSpMatDescr_t matB;
  alphasparseCreateCsr(&matB,
                       A_rows,
                       A_cols,
                       nnz,
                       dBCsrRowPtr,
                       dBcol,
                       dBval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_64F);

  int* dCCsrRowPtr = NULL;
  int* dCrow = NULL;
  int* dCcol = NULL;
  double* dCval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCrow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCCsrRowPtr, sizeof(int) * (A_rows + 1)));
  // CHECK_CUDA(hipMemcpy(
  //   dCrow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  // CHECK_CUDA(hipMemcpy(
  //   dCcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  // CHECK_CUDA(
  //   hipMemcpy(dCval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  // alphasparseXcoo2csr(dCrow, nnz, A_rows, dCCsrRowPtr);

  alphasparseSpMatDescr_t matC;
  alphasparseCreateCsr(&matC,
                       A_rows,
                       A_cols,
                       nnz,
                       dCCsrRowPtr,
                       (int *)nullptr,
                       (double *)nullptr,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_64F);

  size_t bufferSize1 = 0;
  size_t bufferSize2 = 0;
  void* dBuffer1 = NULL;
  void* dBuffer2 = NULL;
  // SpGEMM Computation
  alphasparseSpGEMMDescr_t spgemmDesc;
  alphasparseSpGEMM_createDescr(&spgemmDesc);
  hipEvent_t event_start, event_stop;
  float elapsed_time = 0.0;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  alphasparseSpGEMM_compute(handle,
                            ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                            ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha,
                            matA,
                            matB,
                            &beta,
                            matC,
                            ALPHA_R_64F,
                            ALPHASPARSE_SPGEMM_DEFAULT,
                            spgemmDesc,
                            &bufferSize2,
                            NULL);
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "alphasparseSpGEMM_compute1 time: " << elapsed_time << " microseconds, buffersize :" << bufferSize2 << std::endl;
  
  double time1 = get_time_us();
  hipMalloc(&dBuffer2, bufferSize2);
  double time2 = get_time_us();
  std::cout << "alphasparseSpGEMM buffer malloc time: " << (time2 - time1) / (1e3) << " microseconds" << std::endl;
  double etime = elapsed_time + (time2 - time1) / (1e3);
  
  alphasparseSpGEMM_compute(handle,
                            ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                            ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha,
                            matA,
                            matB,
                            &beta,
                            matC,
                            ALPHA_R_64F,
                            ALPHASPARSE_SPGEMM_DEFAULT,
                            spgemmDesc,
                            &bufferSize2,
                            dBuffer2);
  // allocate matrix C
  nnz_c = matC->nnz;
  // allocate matrix C
  int *dC_csrOffsets = (int *)malloc((A_rows + 1) * sizeof(int));
  int *dC_columns = (int *)malloc(nnz_c * sizeof(int));
  matC_ict = (double*)alpha_malloc(matC->nnz * sizeof(double));
  CHECK_CUDA(hipMemcpy(dC_csrOffsets,
                        matC->row_data,
                       (A_rows + 1) * sizeof(int),
                        hipMemcpyDeviceToHost))
  CHECK_CUDA(hipMemcpy(dC_columns,
                        matC->col_data,
                        matC->nnz * sizeof(int),
                        hipMemcpyDeviceToHost))
  CHECK_CUDA(hipMemcpy(matC_ict,
                        matC->val_data,
                        matC->nnz * sizeof(double),
                        hipMemcpyDeviceToHost))
  std::cout << "alphasparse row ptr " <<std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << dC_csrOffsets[i] << ", ";
  }
  std::cout << std::endl;
  std::cout << "alphasparse col idx " <<std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << dC_columns[i] << ", ";
  }
  std::cout << std::endl;
  
  for (int i = 0; i < warm_up; i++)
  {
    alphasparseSpGEMM_compute(handle,
                              ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                              ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              matA,
                              matB,
                              &beta,
                              matC,
                              ALPHA_R_64F,
                              ALPHASPARSE_SPGEMM_DEFAULT,
                              spgemmDesc,
                              &bufferSize2,
                              dBuffer2);
  }

  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  for (int i = 0; i < trials; i++)
  {
    alphasparseSpGEMM_compute(handle,
                              ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                              ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              matA,
                              matB,
                              &beta,
                              matC,
                              ALPHA_R_64F,
                              ALPHASPARSE_SPGEMM_DEFAULT,
                              spgemmDesc,
                              &bufferSize2,
                              dBuffer2);
  }
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "alphasparseSpGEMM_compute2 time: " << elapsed_time/trials + etime<< " microseconds" << std::endl;
  
  // std::cout << "C_nnz2: " << matC->nnz << std::endl;

  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dBrow);
  hipFree(dBcol);
  hipFree(dBval);
  hipFree(dCrow);
  hipFree(dCcol);
  hipFree(dCval);
  hipFree(dBuffer2);
  alphasparse_destory_handle(handle);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  transB = alpha_args_get_transB(argc, argv);
  trials = args_get_iter(argc, argv);
  // hipSetDevice(1);
  // read coo
  alpha_read_coo<double>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, double>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, A_rows); // 默认C是方阵
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;

  cuda_mm();
  alpha_mm();
  
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  std::cout << std::endl;
  check((double*)matC_roc, nnz_c, (double*)matC_ict, nnz_c);
  return 0;
}
