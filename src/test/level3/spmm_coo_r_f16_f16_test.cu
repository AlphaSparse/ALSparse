
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_coo.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
half* coo_values;

// parms for kernel
half *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
long long B_rows, B_cols;
long long ldb, ldc;
long long B_size, C_size;
const float alpha = 2.f;
const float beta = 3.f;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  int* dArow = NULL;
  int* dAcol = NULL;
  half* dAval = NULL;

  int nnz = rnnz;

  half* dmatB = NULL;
  half* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(half) * B_size);
  hipMalloc((void**)&dmatC, sizeof(half) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half) * nnz));

  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half), hipMemcpyHostToDevice));

  hipMemcpy(dmatB, hmatB, sizeof(half) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(half) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, HIP_R_16F, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_R_16F, HIPSPARSE_ORDER_COL))
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCoo(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dArow,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_16F));
  size_t bufferSize = 0;
  hipsparseOperation_t cutransA, cutransB;
  if(transA == ALPHA_SPARSE_OPERATION_TRANSPOSE) cutransA = HIPSPARSE_OPERATION_TRANSPOSE;
  else if(transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) cutransA = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  else cutransA = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  if(transB == ALPHA_SPARSE_OPERATION_TRANSPOSE) cutransB = HIPSPARSE_OPERATION_TRANSPOSE;
  else if(transB == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) cutransB = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  else cutransB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

  CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle,
                                         cutransA,
                                         cutransB,
                                         &alpha,
                                         matA,
                                         matB,
                                         &beta,
                                         matC,
                                         HIP_R_32F,
                                         HIPSPARSE_SPMM_ALG_DEFAULT,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  printf(" CUDA TRANS %d\n",cutransA);
  CHECK_CUSPARSE(hipsparseSpMM(handle,
                              cutransA,
                              cutransB,
                              &alpha,
                              matA,
                              matB,
                              &beta,
                              matC,
                              HIP_R_32F,
                              HIPSPARSE_SPMM_ALG_DEFAULT,
                              dBuffer))
  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(half), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  int* dArow = NULL;
  int* dAcol = NULL;
  half* dAval = NULL;

  int nnz = rnnz;

  half* dmatB = NULL;
  half* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(half) * B_size);
  hipMalloc((void**)&dmatC, sizeof(half) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half) * nnz));

  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half), hipMemcpyHostToDevice));

  hipMemcpy(dmatB, hmatB, sizeof(half) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(half) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, ALPHA_R_16F, ALPHASPARSE_ORDER_COL);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_R_16F, ALPHASPARSE_ORDER_COL);
  alphasparseSpMatDescr_t matA;
  alphasparseCreateCoo(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dArow,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_16F);
  size_t bufferSize = 0;
  alphasparseSpMM_bufferSize(handle,
                             transA,
                             transB,
                             &alpha,
                             matA,
                             matB,
                             &beta,
                             matC,
                             ALPHA_R_32F,
                             ALPHASPARSE_SPMM_ALG_DEFAULT,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  printf(" ALPHA TRANS %d\n", transA);
  alphasparseSpMM(handle,
                  transA,
                  transB,
                  &alpha,
                  matA,
                  matB,
                  &beta,
                  matC,
                  ALPHA_R_32F,
                  ALPHASPARSE_SPMM_ALG_DEFAULT,
                  dBuffer);
  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(half), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  transB = alpha_args_get_transB(argc, argv);

  // read coo
  alpha_read_coo<half>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, half>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, A_cols); // 默认C是方阵
  
  C_cols = columns;
  B_cols = columns;
  C_rows = A_rows;
  B_rows = A_cols;
  
  ldb = columns;
  ldc = columns;
  B_size = ldb * B_rows;
  C_size = ldc * B_rows;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_row_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_col_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;
  // init x y
  // init B C
  hmatB = (half*)alpha_malloc(B_size * sizeof(half));
  matC_ict = (half*)alpha_malloc(C_size * sizeof(half));
  matC_roc = (half*)alpha_malloc(C_size * sizeof(half));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_fill_random(matC_roc, 1, C_size);

  cuda_mm();
  alpha_mm();

  for (int i = 0; i < 40; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 40; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  std::cout << std::endl;
  check((half*)matC_roc, C_size, (half*)matC_ict, C_size);
  return 0;
}
