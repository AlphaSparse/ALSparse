
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;
alpha_matrix_descr descrT;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
hipFloatComplex* coo_values;

// parms for kernel
hipFloatComplex *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
long long B_cols;
long long ldb, ldc;
long long B_size, C_size;
const hipFloatComplex alpha = { 2.f, 3.f };

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));
  hipsparseSpSMDescr_t spsmDescr;
  CHECK_CUSPARSE(hipsparseSpSM_createDescr(&spsmDescr))
  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  hipFloatComplex* dAval = NULL;

  int nnz = rnnz;

  hipFloatComplex* dmatB = NULL;
  hipFloatComplex* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(hipFloatComplex) * B_size);
  hipMalloc((void**)&dmatC, sizeof(hipFloatComplex) * C_size);
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(hipFloatComplex) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAval, coo_values, nnz * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);
  hipMemcpy(
    dmatB, hmatB, sizeof(hipFloatComplex) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(
    dmatC, matC_roc, sizeof(hipFloatComplex) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, HIP_C_32F, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_C_32F, HIPSPARSE_ORDER_COL))
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dCsrRowPtr,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_C_32F));
  hipsparseFillMode_t fillmode;
  hipsparseDiagType_t diagtype;
  if (descrT.mode == ALPHA_SPARSE_FILL_MODE_LOWER)
    fillmode = HIPSPARSE_FILL_MODE_LOWER;
  else
    fillmode = HIPSPARSE_FILL_MODE_UPPER;
  if (descrT.diag == ALPHA_SPARSE_DIAG_NON_UNIT)
    diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  else
    diagtype = HIPSPARSE_DIAG_TYPE_UNIT;
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(
    matA, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(fillmode)))
  // Specify Unit|Non-Unit diagonal type.
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(
    matA, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype)))
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpSM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         matB,
                                         matC,
                                         HIP_C_32F,
                                         HIPSPARSE_SPSM_ALG_DEFAULT,
                                         spsmDescr,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpSM_analysis(handle,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha,
                                       matA,
                                       matB,
                                       matC,
                                       HIP_C_32F,
                                       HIPSPARSE_SPSM_ALG_DEFAULT,
                                       spsmDescr,
                                       dBuffer))
  CHECK_CUSPARSE(hipsparseSpSM_solve(handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha,
                                    matA,
                                    matB,
                                    matC,
                                    HIP_C_32F,
                                    HIPSPARSE_SPSM_ALG_DEFAULT,
                                    spsmDescr))
  CHECK_CUDA(hipMemcpy(
    matC_roc, dmatC, C_size * sizeof(hipFloatComplex), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);
  alphasparseSpSMDescr_t spsmDescr;
  alphasparseSpSM_createDescr(&spsmDescr);
  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  hipFloatComplex* dAval = NULL;

  int nnz = rnnz;

  hipFloatComplex* dmatB = NULL;
  hipFloatComplex* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(hipFloatComplex) * B_size);
  hipMalloc((void**)&dmatC, sizeof(hipFloatComplex) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(hipFloatComplex) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAval, coo_values, nnz * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(
    dmatB, hmatB, sizeof(hipFloatComplex) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(
    dmatC, matC_ict, sizeof(hipFloatComplex) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, ALPHA_C_32F, ALPHASPARSE_ORDER_COL);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_C_32F, ALPHASPARSE_ORDER_COL);
  alphasparseSpMatDescr_t matA;
  alphasparseCreateCsr(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_C_32F);
  // Specify Lower|Upper fill mode.
  alphasparseSpMatSetAttribute(
    matA, ALPHASPARSE_SPMAT_FILL_MODE, &descrT.mode, sizeof(descrT.mode));
  // Specify Unit|Non-Unit diagonal type.
  alphasparseSpMatSetAttribute(
    matA, ALPHASPARSE_SPMAT_DIAG_TYPE, &descrT.diag, sizeof(descrT.diag));
  size_t bufferSize = 0;
  alphasparseSpSM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             matA,
                             matB,
                             matC,
                             ALPHA_C_32F,
                             ALPHASPARSE_SPSM_ALG_DEFAULT,
                             spsmDescr,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  alphasparseSpSM_solve(handle,
                        ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                        ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        matA,
                        matB,
                        matC,
                        ALPHA_C_32F,
                        ALPHASPARSE_SPSM_ALG_DEFAULT,
                        spsmDescr);
  CHECK_CUDA(hipMemcpy(
    matC_ict, dmatC, C_size * sizeof(hipFloatComplex), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  transB = alpha_args_get_transB(argc, argv);
  descrT = alpha_args_get_matrix_descrA(argc, argv);

  // read coo
  alpha_read_coo<hipFloatComplex>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, hipFloatComplex>(
    rnnz, coo_row_index, coo_col_index, coo_values);
  columns = 1024; // 默认C是方阵
  C_rows = A_rows;
  C_cols = columns;
  B_cols = columns;
  ldb = A_cols;
  ldc = C_rows;
  B_size = ldb * B_cols;
  C_size = ldc * B_cols;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_row_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_col_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;
  // init x y
  // init B C
  hmatB = (hipFloatComplex*)alpha_malloc(B_size * sizeof(hipFloatComplex));
  matC_ict = (hipFloatComplex*)alpha_malloc(C_size * sizeof(hipFloatComplex));
  matC_roc = (hipFloatComplex*)alpha_malloc(C_size * sizeof(hipFloatComplex));
  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_fill_random(matC_roc, 1, C_size);
  cuda_mm();
  alpha_mm();

  for (int i = 0; i < 20; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  check((hipFloatComplex*)matC_roc, C_size, (hipFloatComplex*)matC_ict, C_size);
  return 0;
}
