
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "../../format/csr2csc.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
half2* coo_values;

// parms for kernel
half2 *hmatB;
half2 *matC_ict, *matC_roc;
long long C_rows, C_cols;
long long B_cols;
long long ldb, ldc;
long long B_size, C_size;
const hipFloatComplex alpha = {2.f,3.f};
const hipFloatComplex beta = {3.f,2.f};

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  half2* dAval = NULL;

  int nnz = rnnz;

  half2* dmatB = NULL;
  half2* dmatC = NULL;

  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half2) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half2), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);
  int* dCscColPtr = NULL;
  int* dCscRowInd = NULL;
  half2* dCscVal = NULL;
  size_t csc_bufferSize = 0;
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscRowInd, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscVal, sizeof(half2) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCscColPtr, sizeof(int) * (A_cols + 1)));
  hipsparseCsr2cscEx2_bufferSize(handle,
                                A_rows,
                                A_cols,
                                nnz,
                                dAval,
                                dCsrRowPtr,
                                dAcol,
                                dCscVal,
                                dCscColPtr,
                                dCscRowInd,
                                HIP_C_16F,
                                HIPSPARSE_ACTION_NUMERIC,
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIPSPARSE_CSR2CSC_ALG1,
                                &csc_bufferSize);
  void* csc_dBuffer = NULL;
  CHECK_CUDA(hipMalloc((void**)&csc_dBuffer, csc_bufferSize * sizeof(half2)));
  hipsparseCsr2cscEx2(handle,
                     A_rows,
                     A_cols,
                     nnz,
                     dAval,
                     dCsrRowPtr,
                     dAcol,
                     dCscVal,
                     dCscColPtr,
                     dCscRowInd,
                     HIP_C_16F,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO,
                     HIPSPARSE_CSR2CSC_ALG1,
                     csc_dBuffer);

  hipsparseSpMatDescr_t csc;
  hipsparseCreateCsc(&csc,
                    A_rows,
                    A_cols,
                    nnz,
                    dCscColPtr,
                    dCscRowInd,
                    dCscVal,
                    HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO,
                    HIP_C_16F);

  hipMalloc((void**)&dmatB, sizeof(half2) * B_size);
  hipMalloc((void**)&dmatC, sizeof(half2) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half2) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half2), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(half2) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(half2) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, HIP_C_16F, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_C_16F, HIPSPARSE_ORDER_COL))
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         csc,
                                         matB,
                                         &beta,
                                         matC,
                                         HIP_C_32F,
                                         HIPSPARSE_SPMM_ALG_DEFAULT,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpMM(handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              csc,
                              matB,
                              &beta,
                              matC,
                              HIP_C_32F,
                              HIPSPARSE_SPMM_ALG_DEFAULT,
                              dBuffer))
  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(half2), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  half2* dAval = NULL;

  int nnz = rnnz;

  half2* dmatB = NULL;
  half2* dmatC = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half2) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));

  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half2), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  alphasparseSpMatDescr_t csr;
  alphasparseCreateCsr(&csr,
                       A_rows,
                       A_cols,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_C_16F);
  alphasparseSpMatDescr_t csc;
  alphasparseCsr2csc<int, half2>(csr, csc);

  hipMalloc((void**)&dmatB, sizeof(half2) * B_size);
  hipMalloc((void**)&dmatC, sizeof(half2) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half2) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half2), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(half2) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(half2) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, ALPHA_C_16F, ALPHASPARSE_ORDER_COL);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_C_16F, ALPHASPARSE_ORDER_COL);
  size_t bufferSize = 0;
  alphasparseSpMM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             csc,
                             matB,
                             &beta,
                             matC,
                             ALPHA_C_32F,
                             ALPHASPARSE_SPMM_ALG_DEFAULT,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  alphasparseSpMM(handle,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha,
                  csc,
                  matB,
                  &beta,
                  matC,
                  ALPHA_C_32F,
                  ALPHASPARSE_SPMM_ALG_DEFAULT,
                  dBuffer);
  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(half2), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  transB = alpha_args_get_transB(argc, argv);

  // read coo
  alpha_read_coo<half2>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, half2>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, A_rows); // 默认C是方阵
  C_rows = A_rows;
  C_cols = columns;
  B_cols = columns;
  ldb = A_cols;
  ldc = C_rows;
  B_size = ldb * B_cols;
  C_size = ldc * B_cols;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_row_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_col_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;
  // init x y
  // init B C
  hmatB = (half2*)alpha_malloc(B_size * sizeof(half2));
  matC_ict = (half2*)alpha_malloc(C_size * sizeof(half2));
  matC_roc = (half2*)alpha_malloc(C_size * sizeof(half2));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_fill_random(matC_roc, 1, C_size);

  cuda_mm();
  alpha_mm();

  for (int i = 0; i < 20; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  check((half2*)matC_roc, C_size, (half2*)matC_ict, C_size);
  return 0;
}
