#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../test_common.h"
#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transAT;
alphasparseOperation_t transBT;

long long columns;
int C_rows, C_cols, rnnz;
int *coo_row_index, *coo_col_index;
hipDoubleComplex* coo_values;

// parms for kernel
hipDoubleComplex *hmatA, *hmatB, *matC_ict, *matC_cuda, *matC_cpu;
long long A_rows, A_cols;
long long B_rows, B_cols;
long long lda, ldb;
long long A_size, B_size;
const hipDoubleComplex alpha = {2.f, 1.0f};
const hipDoubleComplex beta = {3.f, 1.0f};

#define CHECK_CUDA(func)                                                       \
    {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",               \
                __LINE__,                                                         \
                hipGetErrorString(status),                                       \
                status);                                                          \
        exit(-1);                                                                \
    }                                                                          \
    }

#define CHECK_CUSPARSE(func)                                                   \
    {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
                __LINE__,                                                         \
                hipsparseGetErrorString(status),                                   \
                status);                                                          \
        exit(-1);                                                                \
    }                                                                          \
    }

static void
cpu_sddmm()
{
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dC_offsets, *dC_columns, *dCCsrRowPtr, *HCCsrRowPtr;
    hipDoubleComplex *dC_values, *dB, *dA;
    HCCsrRowPtr = (int *)malloc(sizeof(int) * (C_rows + 1));

    hipMalloc((void**) &dC_offsets,
                            rnnz * sizeof(int));
    hipMalloc((void**) &dCCsrRowPtr,
                            (C_rows + 1) * sizeof(int));

    CHECK_CUDA( hipMemcpy(dC_offsets, coo_row_index,
                            rnnz * sizeof(int),
                            hipMemcpyHostToDevice) )

    alphasparseXcoo2csr(dC_offsets, rnnz, C_rows, dCCsrRowPtr);
    CHECK_CUDA( hipMemcpy(HCCsrRowPtr, dCCsrRowPtr,
                            (C_rows + 1) * sizeof(int),
                            hipMemcpyDeviceToHost) )
    
    int ldc = A_rows;
    hipDoubleComplex *hmatC = (hipDoubleComplex *)malloc(sizeof(hipDoubleComplex) * ldc * C_cols);
    memset(hmatC, '\0', sizeof(hipDoubleComplex) * ldc * C_cols);
    for (int i = 0; i < C_rows; i++) //M
    {        
        for (int j = 0; j < C_rows; j++)  //N
        {
            for(int p = 0; p < A_cols; p++)  //K
            {
                long long inda = i + p * lda;
                long long indb = p + j * ldb;
                long long indc = i + j * ldc;
                hmatC[indc] = hipCadd(hmatC[indc], hipCmul(hmatA[inda], hmatB[indb]));
            }
        }
    }

    for(int rows = 0; rows < C_rows; rows ++)
    {
        for(int r = HCCsrRowPtr[rows]; r < HCCsrRowPtr[rows + 1]; r ++)
        {
            int col = coo_col_index[r];
            matC_cpu[r] = hipCadd(hipCmul(alpha, hmatC[rows + col * ldc]), hipCmul(beta, coo_values[r]));
        }
    }

    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipFree(dCCsrRowPtr) )
    CHECK_CUDA( hipFree(dC_offsets) )
    free(hmatC);
}

static void
cuda_sddmm()
{
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dC_offsets, *dC_columns, *dCCsrRowPtr;
    hipDoubleComplex *dC_values, *dB, *dA;
    CHECK_CUDA( hipMalloc((void**) &dA, A_size * sizeof(hipDoubleComplex)) )
    CHECK_CUDA( hipMalloc((void**) &dB, B_size * sizeof(hipDoubleComplex)) )
    CHECK_CUDA( hipMalloc((void**) &dC_offsets,
                            rnnz * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dCCsrRowPtr,
                            (C_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_columns, rnnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  rnnz * sizeof(hipDoubleComplex)) )

    CHECK_CUDA( hipMemcpy(dA, hmatA, A_size * sizeof(hipDoubleComplex),
                            hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hmatB, B_size * sizeof(hipDoubleComplex),
                            hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_offsets, coo_row_index,
                            rnnz * sizeof(int),
                            hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_columns, coo_col_index, rnnz * sizeof(int),
                            hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_values, coo_values, rnnz * sizeof(hipDoubleComplex),
                            hipMemcpyHostToDevice) )
    alphasparseXcoo2csr(dC_offsets, rnnz, C_rows, dCCsrRowPtr);
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_rows, A_cols, lda, dA,
                                        HIP_C_64F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, B_rows, B_cols, ldb, dB,
                                        HIP_C_64F, HIPSPARSE_ORDER_COL) )
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, C_rows, C_cols, rnnz,
                                        dCCsrRowPtr, dC_columns, dC_values,
                                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                        HIPSPARSE_INDEX_BASE_ZERO, HIP_C_64F) )
    // allocate an external buffer if needed
    hipsparseOperation_t transA, transB;
    if(transAT == ALPHA_SPARSE_OPERATION_TRANSPOSE) transA = HIPSPARSE_OPERATION_TRANSPOSE;
    else if(transAT == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) transA = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
    else transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    if(transBT == ALPHA_SPARSE_OPERATION_TRANSPOSE) transB = HIPSPARSE_OPERATION_TRANSPOSE;
    else if(transBT == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) transB = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
    else transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    printf("CUDA transA %d transB %d\n", transA, transB);
    CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                                    handle,
                                    transA,
                                    transB,
                                    &alpha, matA, matB, &beta, matC, HIP_C_64F,
                                    HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSDDMM_preprocess(
                                    handle,
                                    transA,
                                    transB,
                                    &alpha, matA, matB, &beta, matC, HIP_C_64F,
                                    HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // execute SpMM
    CHECK_CUSPARSE( hipsparseSDDMM(handle,
                                    transA,
                                    transB,
                                    &alpha, matA, matB, &beta, matC, HIP_C_64F,
                                    HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(matC_cuda, dC_values, rnnz * sizeof(hipDoubleComplex),
                            hipMemcpyDeviceToHost) )

    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC_offsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
}

static void
alpha_sddmm()
{
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dC_offsets, *dC_columns, *dCCsrRowPtr;
    hipDoubleComplex *dC_values, *dB, *dA;
    hipMalloc((void**) &dA, A_size * sizeof(hipDoubleComplex));
    hipMalloc((void**) &dB, B_size * sizeof(hipDoubleComplex));
    hipMalloc((void**) &dC_offsets,
                            rnnz * sizeof(int));
    hipMalloc((void**) &dCCsrRowPtr,
                            (C_rows + 1) * sizeof(int));
    hipMalloc((void**) &dC_columns, rnnz * sizeof(int));
    hipMalloc((void**) &dC_values,  rnnz * sizeof(hipDoubleComplex));

    hipMemcpy(dA, hmatA, A_size * sizeof(hipDoubleComplex),
                            hipMemcpyHostToDevice);
    hipMemcpy(dB, hmatB, B_size * sizeof(hipDoubleComplex),
                            hipMemcpyHostToDevice);
    CHECK_CUDA( hipMemcpy(dC_offsets, coo_row_index,
                            rnnz * sizeof(int),
                            hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_columns, coo_col_index, rnnz * sizeof(int),
                            hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_values, coo_values, rnnz * sizeof(hipDoubleComplex),
                            hipMemcpyHostToDevice) )
    alphasparseXcoo2csr(dC_offsets, rnnz, C_rows, dCCsrRowPtr);
    //--------------------------------------------------------------------------

    alphasparseHandle_t     handle = NULL;
    alphasparseDnMatDescr_t matA, matB;
    alphasparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    initHandle(&handle);
    alphasparseGetHandle(&handle);
    
    // Create dense matrix A
    alphasparseCreateDnMat(&matA, A_rows, A_cols, lda, dA,
                                        ALPHA_C_64F, ALPHASPARSE_ORDER_COL);
    // Create dense matrix B
    alphasparseCreateDnMat(&matB, B_rows, B_cols, ldb, dB,
                                        ALPHA_C_64F, ALPHASPARSE_ORDER_COL);
    // Create sparse matrix C in CSR format
    alphasparseCreateCsr(&matC, C_rows, C_cols, rnnz,
                                        dCCsrRowPtr, dC_columns, dC_values,
                                        ALPHA_SPARSE_INDEXTYPE_I32, ALPHA_SPARSE_INDEXTYPE_I32,
                                        ALPHA_SPARSE_INDEX_BASE_ZERO, ALPHA_C_64F);
    // allocate an external buffer if needed
    printf("ALPHA transA %d transB %d\n", transAT, transBT);
    alphasparseSDDMM_bufferSize(handle,
                            transAT,
                            transBT,
                            &alpha, matA, matB, &beta, matC, ALPHA_C_64F,
                            ALPHASPARSE_SDDMM_ALG_DEFAULT, &bufferSize);
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    hipMemset(dBuffer, '\0', bufferSize);
    // execute preprocess (optional)
    alphasparseSDDMM_preprocess(handle,
                            transAT,
                            transBT,
                            &alpha, matA, matB, &beta, matC, ALPHA_C_64F,
                            ALPHASPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    CHECK_CUDA( hipDeviceSynchronize() )
    // execute SpMM
    alphasparseSDDMM(handle,
                transAT,
                transBT,
                &alpha, matA, matB, &beta, matC, ALPHA_C_64F,
                ALPHASPARSE_SDDMM_ALG_DEFAULT, dBuffer);
    CHECK_CUDA( hipDeviceSynchronize() )
    // destroy matrix/vector descriptors
    // hipsparseDestroyDnMat(matA);
    // hipsparseDestroyDnMat(matB);
    // hipsparseDestroySpMat(matC);
    // alphasparseDestroy(handle);
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA(hipMemcpy(matC_ict, dC_values, rnnz * sizeof(hipDoubleComplex),  hipMemcpyDeviceToHost) )

    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC_offsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
}

int main(int argc, const char *argv[]) {
    // Host problem definition
    args_help(argc, argv);
    file = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    transAT = alpha_args_get_transA(argc, argv);
    transBT = alpha_args_get_transB(argc, argv);

    // read coo
    alpha_read_coo<hipDoubleComplex>(
    file, &C_rows, &C_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
    coo_order<int32_t, hipDoubleComplex>(rnnz, coo_row_index, coo_col_index, coo_values);
    columns = args_get_cols(argc, argv, C_rows); // 默认C是方阵
    A_rows = C_rows;
    A_cols = columns;
    B_rows = A_cols;
    B_cols = C_cols;
    lda = A_rows;
    ldb = B_rows;

    A_size = lda * A_cols;
    B_size = ldb * B_cols;
    
    for (int i = 0; i < 20; i++) {
    std::cout << coo_row_index[i] << ", ";
    }
    std::cout << std::endl;
    for (int i = 0; i < 20; i++) {
    std::cout << coo_col_index[i] << ", ";
    }
    std::cout << std::endl;
    for (int i = 0; i < 20; i++) {
    std::cout << coo_values[i] << ", ";
    }
    std::cout << std::endl;
    // init x y
    // init B C
    hmatA = (hipDoubleComplex*)alpha_malloc(A_size * sizeof(hipDoubleComplex));
    hmatB = (hipDoubleComplex*)alpha_malloc(B_size * sizeof(hipDoubleComplex));
    matC_ict = (hipDoubleComplex*)alpha_malloc(rnnz * sizeof(hipDoubleComplex));
    matC_cuda = (hipDoubleComplex*)alpha_malloc(rnnz * sizeof(hipDoubleComplex));
    matC_cpu = (hipDoubleComplex*)alpha_malloc(rnnz * sizeof(hipDoubleComplex));

    alpha_fill_random(hmatA, 1, A_size);
    alpha_fill_random(hmatB, 1, B_size);
    memset(matC_ict, 0, rnnz * sizeof(hipDoubleComplex));
    memset(matC_cuda, 0, rnnz * sizeof(hipDoubleComplex));
    memset(matC_cpu, 0, rnnz * sizeof(hipDoubleComplex));

    cuda_sddmm();
    alpha_sddmm();
    cpu_sddmm();
    
    for (int i = 0; i < 20; i++) {
    std::cout << matC_cuda[i] << ", ";
    }
    std::cout << std::endl;
    for (int i = 0; i < 20; i++) {
        std::cout << matC_ict[i] << ", ";
    }
    std::cout << std::endl;
    for (int i = 0; i < 20; i++) {
        std::cout << matC_cpu[i] << ", ";
    }
    check((hipDoubleComplex*)matC_cuda, rnnz, (hipDoubleComplex*)matC_ict, rnnz);
    check((hipDoubleComplex*)matC_cuda, rnnz, (hipDoubleComplex*)matC_cpu, rnnz);
    check((hipDoubleComplex*)matC_ict, rnnz, (hipDoubleComplex*)matC_cpu, rnnz);
    
    return EXIT_SUCCESS;
}