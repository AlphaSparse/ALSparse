
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char *file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
float *coo_values;

// parms for kernel
float *hmatB, *matC_ict, *matC_cuda;
long long ldb, ldc;
long long B_size, nnz, nnz_c;
const float alpha = 1.f;
const float beta = 3.f;

int trials = 10, warm_up = 5;

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  int nnz = rnnz;

  // Offload data to device
  int *dACsrRowPtr = NULL;
  int *dArow = NULL;
  int *dAcol = NULL;
  float *dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
      hipMalloc((void **)&dACsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
      dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
      dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dACsrRowPtr);

  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dACsrRowPtr,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));

  int *dBCsrRowPtr = NULL;
  int *dBrow = NULL;
  int *dBcol = NULL;
  float *dBval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dBrow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dBcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dBval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
      hipMalloc((void **)&dBCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
      dBrow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
      dBcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(dBval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dBrow, nnz, A_rows, dBCsrRowPtr);

  hipsparseSpMatDescr_t matB;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matB,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dBCsrRowPtr,
                                   dBcol,
                                   dBval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));

  int *dCCsrRowPtr = NULL;
  int *dCrow = NULL;
  int *dCcol = NULL;
  float *dCval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dCrow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dCcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dCval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
      hipMalloc((void **)&dCCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
      dCrow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
      dCcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(dCval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dCrow, nnz, A_rows, dCCsrRowPtr);

  hipsparseSpMatDescr_t matC;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matC,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dCCsrRowPtr,
                                   dCcol,
                                   dCval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));

  size_t bufferSize1 = 0;
  size_t bufferSize2 = 0;
  void *dBuffer1 = NULL;
  void *dBuffer2 = NULL;
  // SpGEMM Computation
  hipsparseSpGEMMDescr_t spgemmDesc;
  CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&spgemmDesc))
  hipEvent_t event_start, event_stop;
  float elapsed_time = 0.0;
  float elapsed_time_work = 0.0;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  // ask bufferSize1 bytes for external memory
  CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha,
                                               matA,
                                               matB,
                                               &beta,
                                               matC,
                                               HIP_R_32F,
                                               HIPSPARSE_SPGEMM_DEFAULT,
                                               spgemmDesc,
                                               &bufferSize1,
                                               NULL))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "cusparseSpGEMM_workEstimation1 time: " << elapsed_time << " microseconds" << std::endl;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  CHECK_CUDA(hipMalloc((void**)&dBuffer1, bufferSize1))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "cusparseSpGEMM_workEstimation1 buffer1 time: " << elapsed_time << " microseconds" << std::endl;
  elapsed_time_work += elapsed_time;
  // inspect the matrices A and B to understand the memory requirement for
  // the next step
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(handle,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               &alpha,
                                               matA,
                                               matB,
                                               &beta,
                                               matC,
                                               HIP_R_32F,
                                               HIPSPARSE_SPGEMM_DEFAULT,
                                               spgemmDesc,
                                               &bufferSize1,
                                               dBuffer1))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  elapsed_time_work += elapsed_time;
  std::cout << "cusparseSpGEMM_workEstimation2 time: " << elapsed_time << " microseconds" << std::endl;
  // ask bufferSize2 bytes for external memory
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha,
                                        matA,
                                        matB,
                                        &beta,
                                        matC,
                                        HIP_R_32F,
                                        HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc,
                                        &bufferSize2,
                                        NULL))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "cusparseSpGEMM_compute1 time: " << elapsed_time << " microseconds, buffersize :" << bufferSize2 << std::endl;

  double time1 = get_time_us();
  CHECK_CUDA(hipMalloc((void**)&dBuffer2, bufferSize2))
  double time2 = get_time_us();
  std::cout << "cusparseSpGEMM buffer malloc time: " << (time2 - time1) / (1e3) << " microseconds" << std::endl;
  elapsed_time_work += (time2 - time1) / (1e3);

  // compute the intermediate product of A * B
  CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha,
                                        matA,
                                        matB,
                                        &beta,
                                        matC,
                                        HIP_R_32F,
                                        HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc,
                                        &bufferSize2,
                                        dBuffer2))
  // get matrix C non-zero entries C_nnz1
  int64_t C_num_rows1, C_num_cols1, C_nnz1;
  CHECK_CUSPARSE(
    hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1, &C_nnz1))
  nnz_c = C_nnz1;
  // allocate matrix C
  int *dC_csrOffsets, *dC_columns;
  double* dC_values;
  CHECK_CUDA(hipMalloc((void**)&dC_columns, C_nnz1 * sizeof(int)))
  CHECK_CUDA(hipMalloc((void**)&dC_values, C_nnz1 * sizeof(double)))
  CHECK_CUDA(hipMalloc((void**)&dC_csrOffsets, (A_rows + 1) * sizeof(int)))
  // NOTE: if 'beta' != 0, the values of C must be update after the allocation
  //       of dC_values, and before the call of hipsparseSpGEMM_copy

  // update matC with the new pointers
  CHECK_CUSPARSE(
    hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values))

  // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values

  // copy the final products to the matrix C
  CHECK_CUSPARSE(hipsparseSpGEMM_copy(handle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha,
                                     matA,
                                     matB,
                                     &beta,
                                     matC,
                                     HIP_R_32F,
                                     HIPSPARSE_SPGEMM_DEFAULT,
                                     spgemmDesc))
  // device result check
  matC_cuda = (float*)alpha_malloc(C_nnz1 * sizeof(float));
  // std::cout << "C_nnz1: " << C_nnz1 << std::endl;
  CHECK_CUDA(hipMemcpy(
    matC_cuda, dC_values, C_nnz1 * sizeof(float), hipMemcpyDeviceToHost))

  // compute the intermediate product of A * B
  for (int i = 0; i < warm_up; i++)
  {
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha,
                                          matA,
                                          matB,
                                          &beta,
                                          matC,
                                          HIP_R_32F,
                                          HIPSPARSE_SPGEMM_DEFAULT,
                                          spgemmDesc,
                                          &bufferSize2,
                                          dBuffer2))
  }

  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  for (int i = 0; i < trials; i++)
  {
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha,
                                          matA,
                                          matB,
                                          &beta,
                                          matC,
                                          HIP_R_32F,
                                          HIPSPARSE_SPGEMM_DEFAULT,
                                          spgemmDesc,
                                          &bufferSize2,
                                          dBuffer2))
  }
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "cusparseSpGEMM_compute2 time: " << elapsed_time/trials<< " microseconds" << std::endl;
  std::cout << "cusparseSpGEMM_compute_total time: " << elapsed_time/trials + elapsed_time_work<< " microseconds" << std::endl;

  CHECK_CUSPARSE(hipsparseSpGEMM_destroyDescr(spgemmDesc))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
  CHECK_CUSPARSE(hipsparseDestroySpMat(matC))
  CHECK_CUSPARSE(hipsparseDestroy(handle))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dBrow);
  hipFree(dBcol);
  hipFree(dBval);
  hipFree(dCrow);
  hipFree(dCcol);
  hipFree(dCval);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  int nnz = rnnz;

  // Offload data to device
  int *dACsrRowPtr = NULL;
  int *dArow = NULL;
  int *dAcol = NULL;
  float *dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
      hipMalloc((void **)&dACsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
      dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
      dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dACsrRowPtr);

  alphasparseSpMatDescr_t matA;
  alphasparseCreateCsr(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dACsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_32F);

  int *dBCsrRowPtr = NULL;
  int *dBrow = NULL;
  int *dBcol = NULL;
  float *dBval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dBrow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dBcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dBval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
      hipMalloc((void **)&dBCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
      dBrow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
      dBcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(dBval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dBrow, nnz, A_rows, dBCsrRowPtr);

  alphasparseSpMatDescr_t matB;
  alphasparseCreateCsr(&matB,
                       A_rows,
                       A_cols,
                       nnz,
                       dBCsrRowPtr,
                       dBcol,
                       dBval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_32F);

  int *dCCsrRowPtr = NULL;
  int *dCrow = NULL;
  int *dCcol = NULL;
  float *dCval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dCrow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dCcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dCval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
      hipMalloc((void **)&dCCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
      dCrow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
      dCcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(dCval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  // alphasparseXcoo2csr(dCrow, nnz, A_rows, dCCsrRowPtr);

  alphasparseSpMatDescr_t matC;
  alphasparseCreateCsr(&matC,
                       A_rows,
                       A_cols,
                       nnz,
                       dCCsrRowPtr,
                       dCcol,
                       dCval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_32F);

  size_t bufferSize1 = 0;
  size_t bufferSize2 = 0;
  void *dBuffer1 = NULL;
  void *dBuffer2 = NULL;
  // SpGEMM Computation
  alphasparseSpGEMMDescr_t spgemmDesc;
  alphasparseSpGEMM_createDescr(&spgemmDesc);
  hipEvent_t event_start, event_stop;
  float elapsed_time = 0.0;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  alphasparseSpGEMM_compute(handle,
                            ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                            ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha,
                            matA,
                            matB,
                            &beta,
                            matC,
                            ALPHA_R_32F,
                            ALPHASPARSE_SPGEMM_DEFAULT,
                            spgemmDesc,
                            &bufferSize2,
                            NULL);
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "alphasparseSpGEMM_compute1 time: " << elapsed_time << " microseconds, buffersize :" << bufferSize2 <<std::endl;
  double time1 = get_time_us();
  hipMalloc(&dBuffer2, bufferSize2);
  double time2 = get_time_us();
  std::cout << "alphasparseSpGEMM buffer malloc time: " << (time2 - time1) / (1e3) << " microseconds" << std::endl;
  double etime = elapsed_time + (time2 - time1) / (1e3);
  alphasparseSpGEMM_compute(handle,
                            ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                            ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha,
                            matA,
                            matB,
                            &beta,
                            matC,
                            ALPHA_R_32F,
                            ALPHASPARSE_SPGEMM_DEFAULT,
                            spgemmDesc,
                            &bufferSize2,
                            dBuffer2);
  // allocate matrix C
  nnz_c = matC->nnz;
  matC_ict = (float*)alpha_malloc(matC->nnz * sizeof(float));
  CHECK_CUDA(hipMemcpy(matC_ict,
                        matC->val_data,
                        matC->nnz * sizeof(float),
                        hipMemcpyDeviceToHost))

  for (int i = 0; i < warm_up; i++)
  {
    alphasparseSpGEMM_compute(handle,
                              ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                              ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              matA,
                              matB,
                              &beta,
                              matC,
                              ALPHA_R_32F,
                              ALPHASPARSE_SPGEMM_DEFAULT,
                              spgemmDesc,
                              &bufferSize2,
                              dBuffer2);
  }

  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  for (int i = 0; i < trials; i++)
  {
    alphasparseSpGEMM_compute(handle,
                              ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                              ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              matA,
                              matB,
                              &beta,
                              matC,
                              ALPHA_R_32F,
                              ALPHASPARSE_SPGEMM_DEFAULT,
                              spgemmDesc,
                              &bufferSize2,
                              dBuffer2);
  }
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  std::cout << "alphasparseSpGEMM_compute2 time: " << elapsed_time/trials<< " microseconds" << std::endl;
  
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dBrow);
  hipFree(dBcol);
  hipFree(dBval);
  hipFree(dCrow);
  hipFree(dCcol);
  hipFree(dCval);
  hipFree(dBuffer2);
  alphasparse_destory_handle(handle);
}

int main(int argc, const char *argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  transB = alpha_args_get_transB(argc, argv);
  trials = args_get_iter(argc, argv);
  // read coo
  alpha_read_coo<float>(
      file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, float>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, A_rows); // 默认C是方阵
  printf("rnnz %d\n", rnnz);
  for (int i = 0; i < 20; i++)
  {
    std::cout << coo_row_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++)
  {
    std::cout << coo_col_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++)
  {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;

  cuda_mm();
  alpha_mm();
  std::cout << std::endl;
  for (int i = 0; i < 20; i++)
  {
    std::cout << matC_cuda[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++)
  {
    std::cout << matC_ict[i] << ", ";
  }
  check((float *)matC_cuda, nnz_c, (float *)matC_ict, nnz_c);
  // std::cout << std::endl << "check" << std::endl;
  // for (int i = 0; i < nnz_c; i++)
  // {
  //   if((matC_ict[i] - matC_roc[i])/matC_roc[i] > 1e-5)
  //     std::cout << i << ":" << matC_ict[i] << ", " << matC_roc[i] << std::endl;
  // }
  return 0;
}
