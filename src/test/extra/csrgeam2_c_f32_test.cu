#include "../test_common.h"

/**
 * @brief ict csr geam2 test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* fileA; 
const char* fileB;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transAT;
alphasparseOperation_t transBT;

long long Cnnz_ict, Cnnz_cuda;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
hipFloatComplex* coo_values;

int B_rows, B_cols, Bnnz;
int *B_coo_row_index, *B_coo_col_index;
hipFloatComplex* B_coo_values;

// parms for kernel
hipFloatComplex *csrValC_ict, *csrValC_cuda;
const hipFloatComplex alpha = {1.1f,2.4f};
const hipFloatComplex beta = {3.2f,4.3f};

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void cuda_csrgeam()
{
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    int baseC, nnzC;
    /* alpha, nnzTotalDevHostPtr points to host memory */
    size_t BufferSizeInBytes;
    char *buffer = NULL;
    int *nnzTotalDevHostPtr = &nnzC;
    int m = A_rows, n = A_cols;

    int* csrRowPtrA = NULL;
    int* dArow = NULL;
    int* csrColIndA = NULL;
    hipFloatComplex* csrValA = NULL;

    int* csrRowPtrB = NULL;
    int* dBrow = NULL;
    int* csrColIndB = NULL;
    hipFloatComplex* csrValB = NULL;

    int* csrRowPtrC = NULL;
    int* csrColIndC = NULL;
    hipFloatComplex* csrValC = NULL;

    int nnzA = rnnz;
    int nnzB = Bnnz;

    hipsparseMatDescr_t descrA;
    hipsparseMatDescr_t descrB;
    hipsparseMatDescr_t descrC;

    hipsparseOperation_t transA, transB;
    if(transAT == ALPHA_SPARSE_OPERATION_TRANSPOSE) transA = HIPSPARSE_OPERATION_TRANSPOSE;
    else if(transAT == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) transA = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
    else transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    if(transBT == ALPHA_SPARSE_OPERATION_TRANSPOSE) transB = HIPSPARSE_OPERATION_TRANSPOSE;
    else if(transBT == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) transB = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
    else transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrB));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));

    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnzA));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&csrColIndA, sizeof(int) * nnzA));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&csrValA, sizeof(hipFloatComplex) * nnzA));

    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dBrow, sizeof(int) * nnzB));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&csrColIndB, sizeof(int) * nnzB));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&csrValB, sizeof(hipFloatComplex) * nnzB));

    PRINT_IF_CUDA_ERROR(
        hipMalloc((void**)&csrRowPtrA, sizeof(int) * (m + 1)));
    CHECK_CUDA(hipMemcpy(
        dArow, coo_row_index, nnzA * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(
        csrColIndA, coo_col_index, nnzA * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(
        csrValA, coo_values, nnzA * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    hipsparseXcoo2csr(handle, dArow, nnzA, m, csrRowPtrA, HIPSPARSE_INDEX_BASE_ZERO);

    PRINT_IF_CUDA_ERROR(
        hipMalloc((void**)&csrRowPtrB, sizeof(int) * (m + 1)));
    CHECK_CUDA(hipMemcpy(
        dBrow, B_coo_row_index, nnzB * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(
        csrColIndB, B_coo_col_index, nnzB * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(
        hipMemcpy(csrValB, B_coo_values, nnzB * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    hipsparseXcoo2csr(handle, dBrow, nnzB, m, csrRowPtrB, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

    hipMalloc((void**)&csrRowPtrC, sizeof(int)*(m+1));
    /* prepare buffer */
    CHECK_CUSPARSE(hipsparseCcsrgeam2_bufferSizeExt(handle, m, n,
                                    &alpha,
                                    descrA, nnzA,
                                    csrValA, csrRowPtrA, csrColIndA,
                                    &beta,
                                    descrB, nnzB,
                                    csrValB, csrRowPtrB, csrColIndB,
                                    descrC,
                                    csrValC, csrRowPtrC, csrColIndC,
                                    &BufferSizeInBytes));

    CHECK_CUDA(hipMalloc((void**)&buffer, sizeof(char)*BufferSizeInBytes));
    CHECK_CUSPARSE(hipsparseXcsrgeam2Nnz(handle, m, n,
                        descrA, nnzA, csrRowPtrA, csrColIndA,
                        descrB, nnzB, csrRowPtrB, csrColIndB,
                        descrC, csrRowPtrC, nnzTotalDevHostPtr,
                        buffer));
    if (NULL != nnzTotalDevHostPtr){
        nnzC = *nnzTotalDevHostPtr;
    }else{
        hipMemcpy(&nnzC, csrRowPtrC+m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, csrRowPtrC, sizeof(int), hipMemcpyDeviceToHost);
        nnzC -= baseC;
    }
    Cnnz_cuda = nnzC;
    csrValC_cuda = (hipFloatComplex *)malloc(sizeof(hipFloatComplex)*nnzC);

    hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);
    hipMalloc((void**)&csrValC, sizeof(hipFloatComplex)*nnzC);

    CHECK_CUSPARSE(hipsparseCcsrgeam2(handle, m, n,
                    &alpha,
                    descrA, nnzA,
                    csrValA, csrRowPtrA, csrColIndA,
                    &beta,
                    descrB, nnzB,
                    csrValB, csrRowPtrB, csrColIndB,
                    descrC,
                    csrValC, csrRowPtrC, csrColIndC,
                    buffer));

    CHECK_CUDA(hipMemcpy(csrValC_cuda, csrValC, sizeof(hipFloatComplex)*nnzC, hipMemcpyDeviceToHost));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrB));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrC));
}

static void alpha_csrgeam()
{
    alphasparseHandle_t handle = NULL;
    initHandle(&handle);
    alphasparseGetHandle(&handle);
    int baseC, nnzC;
    /* alpha, nnzTotalDevHostPtr points to host memory */
    size_t BufferSizeInBytes;
    char *buffer = NULL;
    int *nnzTotalDevHostPtr = &nnzC;
    int m = A_rows, n = A_cols;

    int* csrRowPtrA = NULL;
    int* dArow = NULL;
    int* csrColIndA = NULL;
    hipFloatComplex* csrValA = NULL;

    int* csrRowPtrB = NULL;
    int* dBrow = NULL;
    int* csrColIndB = NULL;
    hipFloatComplex* csrValB = NULL;

    int* csrRowPtrC = NULL;
    int* csrColIndC = NULL;
    hipFloatComplex* csrValC = NULL;

    int nnzA = rnnz;
    int nnzB = Bnnz;

    alphasparseMatDescr_t descrA;
    alphasparseMatDescr_t descrB;
    alphasparseMatDescr_t descrC;

    alphasparseCreateMatDescr(&descrA);
    alphasparseCreateMatDescr(&descrB);
    alphasparseCreateMatDescr(&descrC);

    hipMalloc((void**)&dArow, sizeof(int) * nnzA);
    hipMalloc((void**)&csrColIndA, sizeof(int) * nnzA);
    hipMalloc((void**)&csrValA, sizeof(hipFloatComplex) * nnzA);

    hipMalloc((void**)&dBrow, sizeof(int) * nnzB);
    hipMalloc((void**)&csrColIndB, sizeof(int) * nnzB);
    hipMalloc((void**)&csrValB, sizeof(hipFloatComplex) * nnzB);
    
    hipMalloc((void**)&csrRowPtrA, sizeof(int) * (m + 1));
    hipMemcpy(
        dArow, coo_row_index, nnzA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(
        csrColIndA, coo_col_index, nnzA * sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(csrValA, coo_values, nnzA * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    alphasparseXcoo2csr(dArow, nnzA, m, csrRowPtrA);
    
    hipMalloc((void**)&csrRowPtrB, sizeof(int) * (m + 1));
    hipMemcpy(
        dBrow, B_coo_row_index, nnzB * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(
        csrColIndB, B_coo_col_index, nnzB * sizeof(int), hipMemcpyHostToDevice);
    
    hipMemcpy(csrValB, B_coo_values, nnzB * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    alphasparseXcoo2csr(dBrow, nnzB, m, csrRowPtrB);

    hipMalloc((void**)&csrRowPtrC, sizeof(int)*(m+1));
    /* prepare buffer */
    alphasparseCcsrgeam2_bufferSizeExt(handle, m, n,
                                    &alpha,
                                    descrA, nnzA,
                                    csrValA, csrRowPtrA, csrColIndA,
                                    &beta,
                                    descrB, nnzB,
                                    csrValB, csrRowPtrB, csrColIndB,
                                    descrC,
                                    csrValC, csrRowPtrC, csrColIndC,
                                    &BufferSizeInBytes);

    hipMalloc((void**)&buffer, sizeof(char)*BufferSizeInBytes);
    alphasparseXcsrgeam2Nnz(handle, m, n,
                        descrA, nnzA, csrRowPtrA, csrColIndA,
                        descrB, nnzB, csrRowPtrB, csrColIndB,
                        descrC, csrRowPtrC, nnzTotalDevHostPtr,
                        buffer);
                        
    if (NULL != nnzTotalDevHostPtr){
        nnzC = *nnzTotalDevHostPtr;
    }
    Cnnz_ict = nnzC;
    csrValC_ict = (hipFloatComplex *)malloc(sizeof(hipFloatComplex)*nnzC);

    hipMalloc((void**)&csrColIndC, sizeof(int)*nnzC);
    hipMalloc((void**)&csrValC, sizeof(hipFloatComplex)*nnzC);

    alphasparseCcsrgeam2(handle, m, n,
                    &alpha,
                    descrA, nnzA,
                    csrValA, csrRowPtrA, csrColIndA,
                    &beta,
                    descrB, nnzB,
                    csrValB, csrRowPtrB, csrColIndB,
                    descrC,
                    csrValC, csrRowPtrC, csrColIndC,
                    buffer);

    hipMemcpy(csrValC_ict, csrValC, sizeof(hipFloatComplex)*nnzC, hipMemcpyDeviceToHost);

    // alphasparseDestroyMatDescr(descrA);
    // alphasparseDestroyMatDescr(descrB);
    // alphasparseDestroyMatDescr(descrC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  fileA = args_get_data_fileA(argc, argv);
  fileB = args_get_data_fileB(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transAT = alpha_args_get_transA(argc, argv);
  transBT = alpha_args_get_transB(argc, argv);

  // read coo
  alpha_read_coo<hipFloatComplex>(
    fileA, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  alpha_read_coo<hipFloatComplex>(
    fileB, &B_rows, &B_cols, &Bnnz, &B_coo_row_index, &B_coo_col_index, &B_coo_values);
  
  if(A_rows != B_rows || A_cols != B_cols)
  {
    printf("INVALID SIZE OF MATRIX A AND B!\n");
    return 0;
  }
  coo_order<int32_t, hipFloatComplex>(rnnz, coo_row_index, coo_col_index, coo_values);
  coo_order<int32_t, hipFloatComplex>(Bnnz, B_coo_row_index, B_coo_col_index, B_coo_values);

  cuda_csrgeam();
  CHECK_CUDA(hipDeviceSynchronize());
  alpha_csrgeam();

  std::cout.precision( 10 );
  for (int i = 0; i < 20; i++) {
    std::cout << csrValC_cuda[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << csrValC_ict[i] << ", ";
  }
  check((hipFloatComplex*)csrValC_cuda, Cnnz_cuda, (hipFloatComplex*)csrValC_ict, Cnnz_ict);

  return 0;
}

