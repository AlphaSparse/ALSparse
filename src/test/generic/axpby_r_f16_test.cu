#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "alphasparse.h"

#include "../test_common.h"

const char *file;
bool check_flag;
int iter;

// sparse vector
int nnz;
int *alpha_x_idx;
int *cuda_x_idx;
half *x_val, *cuda_y, *alpha_y, *cpu_y;
half alpha = 2.;
half beta = 3.;

int idx_n = 1000;

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        exit(-1);                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(-1);                                                   \
    }                                                                          \
}

static void cuda_axpby() {
    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    half *dx_val = NULL;
    half *dy = NULL;

    hipMalloc((void **)&dx_idx, sizeof(int) * idx_n);
    hipMalloc((void **)&dx_val, sizeof(half) * idx_n);
    hipMalloc((void **)&dy, sizeof(half) * idx_n * 20);

    hipMemcpy(dx_idx, cuda_x_idx, sizeof(int) * idx_n,
            hipMemcpyHostToDevice);
    hipMemcpy(dx_val, x_val, sizeof(half) * idx_n, hipMemcpyHostToDevice);
    hipMemcpy(dy, cuda_y, sizeof(half) * idx_n * 20, hipMemcpyHostToDevice);

    // cudaSPARSE handle
    hipsparseHandle_t     handle = NULL;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )

    hipsparseSpVecDescr_t x;
    hipsparseCreateSpVec(&x, idx_n * 20, idx_n, (void *)dx_idx,
                                (void *)dx_val, HIPSPARSE_INDEX_32I,
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIP_R_16F);

    hipsparseDnVecDescr_t y;
    hipsparseCreateDnVec(&y, idx_n * 20, (void *)dy,
                                HIP_R_16F);

    // Call cudasparse csrmv
    roc_call_exit(hipsparseAxpby(handle, (void *)&alpha, x, (void *)&beta, y),
                "cudasparse_axpby");

    // Device synchronization
    hipDeviceSynchronize();

    hipMemcpy(cuda_y, dy, sizeof(half) * idx_n * 20, hipMemcpyDeviceToHost);

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    hipsparseDestroy(handle);
}

static void alpha_axpby()
{
    // cudaSPARSE handle
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    half *dx_val     = NULL;
    half *dy         = NULL;

    hipMalloc((void **)&dx_idx, sizeof(int) * idx_n);
    hipMalloc((void **)&dx_val, sizeof(half) * idx_n);
    hipMalloc((void **)&dy, sizeof(half) * idx_n * 20);

    hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * idx_n, hipMemcpyHostToDevice);
    hipMemcpy(dx_val, x_val, sizeof(half) * idx_n, hipMemcpyHostToDevice);
    hipMemcpy(dy, alpha_y, sizeof(half) * idx_n * 20, hipMemcpyHostToDevice);

    alphasparseSpVecDescr_t x{};
    alphasparseCreateSpVec(&x,idx_n * 20,idx_n,(void *)dx_idx,(void *)dx_val,ALPHA_SPARSE_INDEXTYPE_I32,ALPHA_SPARSE_INDEX_BASE_ZERO,ALPHA_R_16F);

    alphasparseDnVecDescr_t y{};
    alphasparseCreateDnVec(&y,idx_n * 20,(void *)dy,ALPHA_R_16F);

    // Call cudasparse csrmv
    alphasparseAxpby(handle, (void *)&alpha, x, (void *)&beta, y),

    // Device synchronization
    hipDeviceSynchronize();

    hipMemcpy(alpha_y, dy, sizeof(half) * idx_n * 20, hipMemcpyDeviceToHost);

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    alphasparse_destory_handle(handle);
}

__global__ static void cpu_axpby_kernel(int size, int nnz, half alpha, half beta, int *dx_idx, half *dx_val, half *dy)
{
    int tid    = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < size; i ++) {
        dy[i] = beta * dy[i];
    }
    for (int i = tid; i < nnz; i ++) {
        dy[dx_idx[i]] = alpha * dx_val[i] + dy[dx_idx[i]];
    }
}

static void cpu_axpby()
{
    int *dx_idx = NULL;
    half *dx_val     = NULL;
    half *dy         = NULL;

    hipMalloc((void **)&dx_idx, sizeof(int) * idx_n);
    hipMalloc((void **)&dx_val, sizeof(half) * idx_n);
    hipMalloc((void **)&dy, sizeof(half) * idx_n * 20);

    hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * idx_n, hipMemcpyHostToDevice);
    hipMemcpy(dx_val, x_val, sizeof(half) * idx_n, hipMemcpyHostToDevice);
    hipMemcpy(dy, cpu_y, sizeof(half) * idx_n * 20, hipMemcpyHostToDevice);
    
    cpu_axpby_kernel<<<1,1>>>(idx_n * 20, idx_n, alpha, beta, dx_idx, dx_val, dy);
    hipDeviceSynchronize();
    hipMemcpy(cpu_y, dy, sizeof(half) * idx_n * 20, hipMemcpyDeviceToHost);    

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
}

int main(int argc, const char *argv[])
{
    // args
    args_help(argc, argv);
    file  = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    iter  = args_get_iter(argc, argv);
    idx_n  = args_get_nnz(argc, argv);
    alpha_x_idx =
        (int *)alpha_memalign(sizeof(int) * idx_n, DEFAULT_ALIGNMENT);
    cuda_x_idx = (int *)alpha_memalign(sizeof(int) * idx_n,
                                                DEFAULT_ALIGNMENT);
    x_val     = (half *)alpha_memalign(sizeof(half) * idx_n, DEFAULT_ALIGNMENT);
    alpha_y   = (half *)alpha_memalign(sizeof(half) * idx_n * 20, DEFAULT_ALIGNMENT);
    cuda_y     = (half *)alpha_memalign(sizeof(half) * idx_n * 20, DEFAULT_ALIGNMENT);
    cpu_y     = (half *)alpha_memalign(sizeof(half) * idx_n * 20, DEFAULT_ALIGNMENT);

    alpha_fill_random(alpha_y, 1, idx_n * 20);
    alpha_fill_random(cuda_y, 1, idx_n * 20);
    alpha_fill_random(cpu_y, 1, idx_n * 20);
    alpha_fill_random(x_val, 0, idx_n);

    for (int i = 0; i < idx_n; i++) {
        alpha_x_idx[i] = i * 20;
        cuda_x_idx[i]   = i * 20;
    }

    alpha_axpby();

    if (check_flag) {
        cuda_axpby();
        cpu_axpby();
        check(alpha_y, idx_n * 20, cuda_y, idx_n * 20);
        check(cpu_y, idx_n * 20, cuda_y, idx_n * 20);
        check(cpu_y, idx_n * 20, alpha_y, idx_n * 20);
    }
    printf("\n");

    alpha_free(x_val);
    alpha_free(cuda_x_idx);
    alpha_free(alpha_x_idx);
    return 0;
}
