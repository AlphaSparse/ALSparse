
#include "../test_common.h"

/**
 * @brief ict mv csr test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "alphasparse.h"

#include "alphasparse/util/auxiliary.h"

const char *file;
bool check_flag;
int iter;

// sparse vector
int nnz;
int *alpha_x_idx;
int *roc_x_idx;
nv_bfloat162 *x_val, *cuda_y, *alpha_y;
hipFloatComplex alpha = {2., 3.}, beta = {3., 2.};
// hipFloatComplex alpha = make_hipFloatComplex(1., 1.);
// hipFloatComplex beta = make_hipFloatComplex(1., 1.);

int idx_n = 1000;

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            exit(-1);                                                  \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            exit(-1);                                                      \
        }                                                                  \
    }

static void roc_axpby()
{
    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    nv_bfloat162 *dx_val = NULL;
    nv_bfloat162 *dy = NULL;

    CHECK_CUDA(hipMalloc((void **)&dx_idx, sizeof(int) * idx_n))
    CHECK_CUDA(hipMalloc((void **)&dx_val, sizeof(nv_bfloat162) * idx_n))
    CHECK_CUDA(hipMalloc((void **)&dy, sizeof(nv_bfloat162) * idx_n * 20))

    CHECK_CUDA(hipMemcpy(dx_idx, roc_x_idx, sizeof(int) * idx_n,
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dx_val, x_val, sizeof(nv_bfloat162) * idx_n, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dy, cuda_y, sizeof(nv_bfloat162) * idx_n * 20, hipMemcpyHostToDevice))

    // rocSPARSE handle
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle))

    hipsparseSpVecDescr_t x;
    CHECK_CUSPARSE(hipsparseCreateSpVec(&x, idx_n * 20, idx_n, (void *)dx_idx,
                                       (void *)dx_val, HIPSPARSE_INDEX_32I,
                                       HIPSPARSE_INDEX_BASE_ZERO,
                                       HIP_C_16BF))

    hipsparseDnVecDescr_t y;
    CHECK_CUSPARSE(hipsparseCreateDnVec(&y, idx_n * 20, (void *)dy,
                                       HIP_C_16BF))

    // Call rocsparse csrmv
    CHECK_CUSPARSE(hipsparseAxpby(handle, (void *)&alpha, x, (void *)&beta, y))

    // Device synchronization
    CHECK_CUDA(hipDeviceSynchronize())

    CHECK_CUDA(hipMemcpy(cuda_y, dy, sizeof(nv_bfloat162) * idx_n * 20, hipMemcpyDeviceToHost))

    // Clear up on device
    CHECK_CUDA(hipFree(dx_val))
    CHECK_CUDA(hipFree(dx_idx))
    CHECK_CUDA(hipFree(dy))
    CHECK_CUSPARSE(hipsparseDestroy(handle))
}

static void alpha_axpyi()
{
    // rocSPARSE handle
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    nv_bfloat162 *dx_val = NULL;
    nv_bfloat162 *dy = NULL;

    CHECK_CUDA(hipMalloc((void **)&dx_idx, sizeof(int) * idx_n))
    CHECK_CUDA(hipMalloc((void **)&dx_val, sizeof(nv_bfloat162) * idx_n))
    CHECK_CUDA(hipMalloc((void **)&dy, sizeof(nv_bfloat162) * idx_n * 20))

    CHECK_CUDA(hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * idx_n, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dx_val, x_val, sizeof(nv_bfloat162) * idx_n, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dy, alpha_y, sizeof(nv_bfloat162) * idx_n * 20, hipMemcpyHostToDevice))

    alphasparseSpVecDescr_t x{};
    alphasparseCreateSpVec(&x, idx_n * 20, idx_n, (void *)dx_idx, (void *)dx_val,
                           ALPHA_SPARSE_INDEXTYPE_I32, ALPHA_SPARSE_INDEX_BASE_ZERO, ALPHA_C_16BF);

    alphasparseDnVecDescr_t y{};
    alphasparseCreateDnVec(&y, idx_n * 20, (void *)dy, ALPHA_C_16BF);

    // Call rocsparse csrmv
    alphasparseAxpby(handle, (void *)&alpha, x, (void *)&beta, y);

    // Device synchronization
    CHECK_CUDA(hipDeviceSynchronize())

    CHECK_CUDA(hipMemcpy(alpha_y, dy, sizeof(nv_bfloat162) * idx_n * 20, hipMemcpyDeviceToHost))

    // Clear up on device
    CHECK_CUDA(hipFree(dx_val))
    CHECK_CUDA(hipFree(dx_idx))
    CHECK_CUDA(hipFree(dy))
    alphasparse_destory_handle(handle);
}

int main(int argc, const char *argv[])
{
    // args
    args_help(argc, argv);
    file = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    iter = args_get_iter(argc, argv);
    idx_n  = args_get_nnz(argc, argv);

    alpha_x_idx =
        (int *)malloc(sizeof(int) * idx_n);
    roc_x_idx = (int *)malloc(sizeof(int) * idx_n);
    x_val = (nv_bfloat162 *)malloc(sizeof(nv_bfloat162) * idx_n);
    alpha_y = (nv_bfloat162 *)malloc(sizeof(nv_bfloat162) * idx_n * 20);
    cuda_y = (nv_bfloat162 *)malloc(sizeof(nv_bfloat162) * idx_n * 20);

    alpha_fill_random(alpha_y, 1, idx_n * 20);
    alpha_fill_random(cuda_y, 1, idx_n * 20);
    alpha_fill_random(x_val, 0, idx_n);

    for (int i = 0; i < idx_n; i++)
    {
        alpha_x_idx[i] = i * 20;
        roc_x_idx[i] = i * 20;
    }

    alpha_axpyi();

    if (check_flag)
    {
        roc_axpby();
        // printf("\n===================\n");
        // for(int i=0;i<20;i++)
        //     std::cout<<alpha_y[i]<<" ";
        // printf("\n====alpha_y end========\n");
        // for(int i=0;i<20;i++)
        //     std::cout<<cuda_y[i]<<" ";
        // printf("\n====cuda_y========\n");
        check(alpha_y, idx_n * 20, cuda_y, idx_n * 20);
    }
    printf("\n");

    alpha_free(x_val);
    alpha_free(roc_x_idx);
    alpha_free(alpha_x_idx);
    alpha_free(alpha_y);
    alpha_free(cuda_y);
    return 0;
}
