#include "../test_common.h"

/**
 * @brief ict dcu mv csr test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include <alphasparse.h>

const char *file;
bool check_flag;
int iter;

// sparse vector
int nnz = 10000;
int *alpha_x_idx;
int *roc_x_idx;
float *x_val;
float *cuda_y, *alpha_y;

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        exit(-1);                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(-1);                                                   \
    }                                                                          \
}

static void roc_sctr() {
  // cusparse handle
  hipsparseHandle_t handle;
  CHECK_CUSPARSE( hipsparseCreate(&handle) )

  hipDeviceProp_t devProp;
  int device_id = 0;

  hipGetDevice(&device_id);
  hipGetDeviceProperties(&devProp, device_id);
  std::cout << "Device: " << devProp.name << std::endl;

  // Offload data to device
  int *dx_idx = NULL;
  float *dx_val = NULL;
  float *dy = NULL;

  hipMalloc((void **)&dx_idx, sizeof(int) * nnz);
  hipMalloc((void **)&dx_val, sizeof(float) * nnz);
  hipMalloc((void **)&dy, sizeof(float) * nnz * 20);

  hipMemcpy(dx_idx, roc_x_idx, sizeof(int) * nnz,
            hipMemcpyHostToDevice);
  hipMemcpy(dx_val, x_val, sizeof(float) * nnz, hipMemcpyHostToDevice);
  hipMemcpy(dy, cuda_y, sizeof(float) * nnz * 20, hipMemcpyHostToDevice);

hipsparseSpVecDescr_t x;
    hipsparseCreateSpVec(&x, nnz * 20, nnz, (void *)dx_idx,
                                (void *)dx_val, HIPSPARSE_INDEX_32I,
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIP_R_32F);

hipsparseDnVecDescr_t y;
hipsparseCreateDnVec(&y, nnz * 20, (void *)dy,
                            HIP_R_32F);

  // Call cusparse csrmv
  CHECK_CUSPARSE( hipsparseScatter(handle, x, y) )

  // Device synchronization
  hipDeviceSynchronize();

  hipMemcpy(cuda_y, dy, sizeof(float) * nnz * 20, hipMemcpyDeviceToHost);

  // Clear up on device
  hipFree(dx_val);
  hipFree(dx_idx);
  hipFree(dy);
  hipsparseDestroy(handle);
}

static void alpha_sctr()
{
    // cusparse handle
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    float *dx_val     = NULL;
    float *dy         = NULL;

    hipMalloc((void **)&dx_idx, sizeof(int) * nnz);
    hipMalloc((void **)&dx_val, sizeof(float) * nnz);
    hipMalloc((void **)&dy, sizeof(float) * nnz * 20);

    hipMemcpy(dx_idx, roc_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dx_val, x_val, sizeof(float) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dy, alpha_y, sizeof(float) * nnz * 20, hipMemcpyHostToDevice);

    alphasparseSpVecDescr_t x{};
    alphasparseCreateSpVec(&x, nnz * 20,nnz,(void *)dx_idx,(void *)dx_val,ALPHA_SPARSE_INDEXTYPE_I32,ALPHA_SPARSE_INDEX_BASE_ZERO,ALPHA_R_32F);

    alphasparseDnVecDescr_t y{};
    alphasparseCreateDnVec(&y, nnz * 20,(void *)dy,ALPHA_R_32F);

    // Call cusparse csrmv
    alphasparseScatter(handle, x, y);

    // Device synchronization
    hipDeviceSynchronize();

    hipMemcpy(alpha_y, dy, sizeof(float) * nnz * 20, hipMemcpyDeviceToHost);

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    alphasparse_destory_handle(handle);
}

int main(int argc, const char *argv[])
{
    // args
    args_help(argc, argv);
    file  = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    iter  = args_get_iter(argc, argv);
    nnz  = args_get_nnz(argc, argv);

    alpha_x_idx = (int *)alpha_memalign(sizeof(int) * nnz, DEFAULT_ALIGNMENT);
    roc_x_idx   = (int *)alpha_memalign(sizeof(int) * nnz,
                                                DEFAULT_ALIGNMENT);
    x_val       = (float *)alpha_memalign(sizeof(float) * nnz, DEFAULT_ALIGNMENT);
    cuda_y       = (float *)alpha_memalign(sizeof(float) * nnz * 20, DEFAULT_ALIGNMENT);
    alpha_y     = (float *)alpha_memalign(sizeof(float) * nnz * 20, DEFAULT_ALIGNMENT);

    alpha_fill_random(cuda_y, 1, nnz * 20);
    alpha_fill_random(alpha_y, 1, nnz * 20);
    alpha_fill_random(x_val, 0, nnz);

    for (int i = 0; i < nnz; i++) {
        alpha_x_idx[i] = i * 20;
        roc_x_idx[i]   = i * 20;
    }

    alpha_sctr();

    if (check_flag) {
      roc_sctr();
      check(cuda_y, nnz * 20, alpha_y, nnz * 20);
    }
    printf("\n");

    alpha_free(x_val);
    alpha_free(roc_x_idx);
    alpha_free(alpha_x_idx);
    alpha_free(cuda_y);
    alpha_free(alpha_y);
    return 0;
}
