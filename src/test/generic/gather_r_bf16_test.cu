#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv csr test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>
#include <alphasparse.h>

const char *file;
bool check_flag;
int iter;

// sparse vector
int nnz = 1000;
int *alpha_x_idx;
int *roc_x_idx;
hip_bfloat16 *alpha_x_val, *roc_x_val;
hip_bfloat16 *y;

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            exit(-1);                                                  \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            exit(-1);                                                      \
        }                                                                  \
    }

static void roc_gthr()
{
    // cusparse handle
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle))

    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    hip_bfloat16 *dx_val = NULL;
    hip_bfloat16 *dy = NULL;

    hipMalloc((void **)&dx_idx, sizeof(int) * nnz);
    hipMalloc((void **)&dx_val, sizeof(hip_bfloat16) * nnz);
    hipMalloc((void **)&dy, sizeof(hip_bfloat16) * nnz * 20);

    hipMemcpy(dx_idx, roc_x_idx, sizeof(int) * nnz,
               hipMemcpyHostToDevice);
    hipMemcpy(dx_val, roc_x_val, sizeof(hip_bfloat16) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dy, y, sizeof(hip_bfloat16) * nnz * 20, hipMemcpyHostToDevice);

    hipsparseSpVecDescr_t x;
    hipsparseCreateSpVec(&x, nnz * 20, nnz, (void *)dx_idx,
                        (void *)dx_val, HIPSPARSE_INDEX_32I,
                        HIPSPARSE_INDEX_BASE_ZERO,
                        HIP_R_16BF);

    hipsparseDnVecDescr_t y;
    hipsparseCreateDnVec(&y, nnz * 20, (void *)dy,
                        HIP_R_16BF);

    // Call cusparse csrmv
    CHECK_CUSPARSE(hipsparseGather(handle, y, x))

    // Device synchronization
    hipDeviceSynchronize();

    hipMemcpy(roc_x_val, dx_val, sizeof(hip_bfloat16) * nnz, hipMemcpyDeviceToHost);

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    hipsparseDestroy(handle);
}

static void alpha_gthr()
{
    // cusparse handle
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    hip_bfloat16 *dx_val = NULL;
    hip_bfloat16 *dy = NULL;

    hipMalloc((void **)&dx_idx, sizeof(int) * nnz);
    hipMalloc((void **)&dx_val, sizeof(hip_bfloat16) * nnz);
    hipMalloc((void **)&dy, sizeof(hip_bfloat16) * nnz * 20);

    hipMemcpy(dx_idx, roc_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dx_val, alpha_x_val, sizeof(hip_bfloat16) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dy, y, sizeof(hip_bfloat16) * nnz * 20, hipMemcpyHostToDevice);

    alphasparseSpVecDescr_t x{};
    alphasparseCreateSpVec(&x, nnz * 20, nnz, (void *)dx_idx, (void *)dx_val,
                           ALPHA_SPARSE_INDEXTYPE_I32, ALPHA_SPARSE_INDEX_BASE_ZERO, ALPHA_R_16BF);

    alphasparseDnVecDescr_t y{};
    alphasparseCreateDnVec(&y, nnz * 20, (void *)dy, ALPHA_R_16BF);

    // Call cusparse csrmv
    alphasparseGather(handle, y, x);

    // Device synchronization
    hipDeviceSynchronize();

    hipMemcpy(alpha_x_val, dx_val, sizeof(hip_bfloat16) * nnz, hipMemcpyDeviceToHost);

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    alphasparse_destory_handle(handle);
}

int main(int argc, const char *argv[])
{
    // args
    args_help(argc, argv);
    file = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    iter = args_get_iter(argc, argv);
    nnz  = args_get_nnz(argc, argv);

    alpha_x_idx = (int *)alpha_memalign(sizeof(int) * nnz, DEFAULT_ALIGNMENT);
    roc_x_idx = (int *)alpha_memalign(sizeof(int) * nnz,
                                      DEFAULT_ALIGNMENT);
    alpha_x_val = (hip_bfloat16 *)alpha_memalign(sizeof(hip_bfloat16) * nnz, DEFAULT_ALIGNMENT);
    roc_x_val = (hip_bfloat16 *)alpha_memalign(sizeof(hip_bfloat16) * nnz, DEFAULT_ALIGNMENT);
    y = (hip_bfloat16 *)alpha_memalign(sizeof(hip_bfloat16) * nnz * 20, DEFAULT_ALIGNMENT);

    alpha_fill_random(y, 0, nnz * 20);
    alpha_fill_random(alpha_x_val, 1, nnz);
    alpha_fill_random(roc_x_val, 1, nnz);

    for (int i = 0; i < nnz; i++)
    {
        alpha_x_idx[i] = i * 20;
        roc_x_idx[i] = i * 20;
    }

    alpha_gthr();

    if (check_flag)
    {
        roc_gthr();
        check(alpha_x_val, nnz, roc_x_val, nnz);
    }
    printf("\n");

    alpha_free(roc_x_val);
    alpha_free(alpha_x_val);
    alpha_free(roc_x_idx);
    alpha_free(alpha_x_idx);
    alpha_free(y);
    return 0;
}
