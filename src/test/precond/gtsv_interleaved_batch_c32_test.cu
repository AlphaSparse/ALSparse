#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "alphasparse.h"

bool check_flag;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int algo = 0;
int m = 128, batchCount = 8, batchStride = 32, size;
hipFloatComplex *hdl, *hd, *hdu, *hict_x, *hcuda_x;
float error;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void
cuda_gtsv2()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  hipFloatComplex* ddl = NULL;
  hipFloatComplex* dd = NULL;
  hipFloatComplex* ddu = NULL;
  hipFloatComplex* dx = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dx, sizeof(hipFloatComplex) * size))

  // Copy data to device
  CHECK_CUDA(
    hipMemcpy(ddl, hdl, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dd, hd, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(ddu, hdu, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(
    dx, hcuda_x, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  CHECK_CUSPARSE(hipsparseCgtsvInterleavedBatch_bufferSizeExt(
    handle, algo, m, ddl, dd, ddu, dx, batchCount, &buffer_size))

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));
  CHECK_CUSPARSE(hipsparseCgtsvInterleavedBatch(
    handle, algo, m, ddl, dd, ddu, dx, batchCount, temp_buffer));

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(
    hcuda_x, dx, sizeof(hipFloatComplex) * size, hipMemcpyDeviceToHost));

  CHECK_CUSPARSE(hipsparseDestroy(handle))
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dx))
  CHECK_CUDA(hipFree(temp_buffer))
}

void
alpha_gtsv2()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  hipFloatComplex* ddl = NULL;
  hipFloatComplex* dd = NULL;
  hipFloatComplex* ddu = NULL;
  hipFloatComplex* dx = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dx, sizeof(hipFloatComplex) * size))

  // Copy data to device
  CHECK_CUDA(
    hipMemcpy(ddl, hdl, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dd, hd, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(ddu, hdu, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(
    dx, hict_x, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  alphasparseCgtsvInterleavedBatch_bufferSizeExt(
    handle, algo, m, ddl, dd, ddu, dx, batchCount, &buffer_size);

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));

  alphasparseCgtsvInterleavedBatch(
    handle, algo, m, ddl, dd, ddu, dx, batchCount, temp_buffer);

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(
    hict_x, dx, sizeof(hipFloatComplex) * size, hipMemcpyDeviceToHost));

  alphasparse_destory_handle(handle);
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dx))
  CHECK_CUDA(hipFree(temp_buffer))
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  m = args_get_rows(argc, argv, m);
  batchCount = args_get_batch_count(argc, argv, batchCount);
  batchStride = args_get_batch_stride(argc, argv, batchStride);
  size = m * batchCount;

  hdl = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hd = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hdu = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hcuda_x = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hict_x = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));

  alpha_fill_random(hdl, 899, size);
  alpha_fill_random(hd, 101, size);
  alpha_fill_random(hdu, 77, size);
  for (size_t i = 0; i < m; ++i)
  {
    if (hd[i].x > 0)
      hd[i].x += m/2;
    else
      hd[i].x -= m/2;

    if (hd[i].y > 0)
      hd[i].y += m/2;
    else
      hd[i].y -= m/2;
  }
  for(int i = 0; i < batchCount; i++)
    hdl[i * m] = {};
  for(int i = 0; i < batchCount; i++)
    hdu[i* m - 1] = {};
  alpha_fill_random(hcuda_x, 1, size);
  alpha_fill_random(hict_x, 1, size);
  std::cout << "===========hdl=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hdl[i] << ", ";
  }
  std::cout << std::endl << "===========hdl=============" << std::endl;
  std::cout << std::endl << "===========hd=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hd[i] << ", ";
  }
  std::cout << std::endl << "===========hd=============" << std::endl;
  std::cout << std::endl << "===========hdu=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hdu[i] << ", ";
  }
  std::cout << std::endl << "===========hdu=============" << std::endl;
  std::cout << std::endl << "===========hcuda_x=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcuda_x[i] << ", ";
  }
  std::cout << std::endl << "===========hcuda_x=============" << std::endl;
  std::cout << std::endl << "===========hict_x=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hict_x[i] << ", ";
  }
  std::cout << std::endl << "===========hict_x=============" << std::endl;

  cuda_gtsv2();
  alpha_gtsv2();

  std::cout << std::endl << "===========result=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcuda_x[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hict_x[i] << ", ";
  }
  std::cout << std::endl << "===========result=============" << std::endl;
  check(hict_x, size, hcuda_x, size, &error);

  return 0;
}
