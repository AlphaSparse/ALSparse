#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "alphasparse.h"

bool check_flag;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int algo = 0;
int m = 32, batchCount = 16, batchStride = 64, size;
hipDoubleComplex *hdl, *hd, *hdu, *hict_x, *hcuda_x;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void
cuda_gtsv2()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  hipDoubleComplex* ddl = NULL;
  hipDoubleComplex* dd = NULL;
  hipDoubleComplex* ddu = NULL;
  hipDoubleComplex* dx = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipDoubleComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipDoubleComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipDoubleComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dx, sizeof(hipDoubleComplex) * size))

  // Copy data to device
  CHECK_CUDA(hipMemcpy(ddl, hdl, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dd, hd, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(ddu, hdu, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dx, hcuda_x, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  CHECK_CUSPARSE(hipsparseZgtsvInterleavedBatch_bufferSizeExt(
    handle, algo, m, ddl, dd, ddu, dx, batchCount, &buffer_size))

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));
  CHECK_CUSPARSE(hipsparseZgtsvInterleavedBatch(
    handle, algo, m, ddl, dd, ddu, dx, batchCount, temp_buffer));

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(
    hipMemcpy(hcuda_x, dx, sizeof(hipDoubleComplex) * size, hipMemcpyDeviceToHost));

  CHECK_CUSPARSE(hipsparseDestroy(handle))
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dx))
  CHECK_CUDA(hipFree(temp_buffer))
}

void
alpha_gtsv2()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  hipDoubleComplex* ddl = NULL;
  hipDoubleComplex* dd = NULL;
  hipDoubleComplex* ddu = NULL;
  hipDoubleComplex* dx = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipDoubleComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipDoubleComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipDoubleComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dx, sizeof(hipDoubleComplex) * size))

  // Copy data to device
  CHECK_CUDA(hipMemcpy(ddl, hdl, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dd, hd, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(ddu, hdu, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dx, hict_x, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  alphasparseZgtsvInterleavedBatch_bufferSizeExt(
    handle, algo, m, ddl, dd, ddu, dx, batchCount, &buffer_size);

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));

  alphasparseZgtsvInterleavedBatch(
    handle, algo, m, ddl, dd, ddu, dx, batchCount, temp_buffer);

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(
    hipMemcpy(hict_x, dx, sizeof(hipDoubleComplex) * size, hipMemcpyDeviceToHost));

  alphasparse_destory_handle(handle);
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dx))
  CHECK_CUDA(hipFree(temp_buffer))
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  m = args_get_rows(argc, argv, m);
  batchCount = args_get_batch_count(argc, argv, batchCount);
  batchStride = args_get_batch_stride(argc, argv, batchStride);
  size = m * batchStride;
  
  hdl = (hipDoubleComplex*)alpha_malloc(size * sizeof(hipDoubleComplex));
  hd = (hipDoubleComplex*)alpha_malloc(size * sizeof(hipDoubleComplex));
  hdu = (hipDoubleComplex*)alpha_malloc(size * sizeof(hipDoubleComplex));
  hcuda_x = (hipDoubleComplex*)alpha_malloc(size * sizeof(hipDoubleComplex));
  hict_x = (hipDoubleComplex*)alpha_malloc(size * sizeof(hipDoubleComplex));

  alpha_fill_random(hdl, 899, size);
  hdl[0] = {};
  alpha_fill_random(hd, 101, size);
  alpha_fill_random(hdu, 77, size);
  hdu[m - 1] = {};
  alpha_fill_random(hcuda_x, 1, size);
  alpha_fill_random(hict_x, 1, size);
  std::cout << "===========hdl=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hdl[i] << ", ";
  }
  std::cout << std::endl << "===========hdl=============" << std::endl;
  std::cout << std::endl << "===========hd=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hd[i] << ", ";
  }
  std::cout << std::endl << "===========hd=============" << std::endl;
  std::cout << std::endl << "===========hdu=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hdu[i] << ", ";
  }
  std::cout << std::endl << "===========hdu=============" << std::endl;
  std::cout << std::endl << "===========hcuda_x=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcuda_x[i] << ", ";
  }
  std::cout << std::endl << "===========hcuda_x=============" << std::endl;
  std::cout << std::endl << "===========hict_x=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hict_x[i] << ", ";
  }
  std::cout << std::endl << "===========hict_x=============" << std::endl;

  cuda_gtsv2();
  alpha_gtsv2();

  std::cout << std::endl << "===========result=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcuda_x[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hict_x[i] << ", ";
  }
  std::cout << std::endl << "===========result=============" << std::endl;
  check(hict_x, size, hcuda_x, size);

  return 0;
}
