#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int m, n, nnz;
int* csrRowPtr = NULL;
int *coo_row_index, *coo_col_index;
hipFloatComplex* coo_values;

// coo format
hipFloatComplex* x_val;
hipFloatComplex* ict_y;
hipFloatComplex* cuda_y;

hipFloatComplex* ict_val;
hipFloatComplex* cuda_val;

// parms for kernel
const hipFloatComplex alpha = {2.3f, 3.4f};

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void cuda_ic02()
{
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Offload data to device
    int* d_csrRowPtr = NULL;
    int* dArow = NULL;
    int* d_csrColInd = NULL;
    hipFloatComplex* d_csrVal = NULL;
    hipFloatComplex* d_x = NULL;
    hipFloatComplex* d_y = NULL;
    hipFloatComplex* d_z = NULL;

    CHECK_CUDA(hipMalloc((void**)&d_x, n * sizeof(hipFloatComplex)));
    CHECK_CUDA(hipMalloc((void**)&d_y, m * sizeof(hipFloatComplex)));
    CHECK_CUDA(hipMalloc((void**)&d_z, m * sizeof(hipFloatComplex)));
    CHECK_CUDA(hipMemcpy(d_x, x_val, n * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, cuda_y, m * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrColInd, sizeof(int) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrVal, sizeof(hipFloatComplex) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrRowPtr, sizeof(int) * (m + 1)));
    
    CHECK_CUDA(hipMemcpy(
        dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(
        d_csrColInd, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(
        hipMemcpy(d_csrVal, coo_values, nnz * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    alphasparseXcoo2csr(dArow, nnz, m, d_csrRowPtr);
    hipsparseMatDescr_t descr_M = 0;
    hipsparseMatDescr_t descr_L = 0;
    csric02Info_t info_M = 0;

    int pBufferSize_M;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseOperation_t trans_L = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;
    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has non-unit diagonal
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr_M));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr_L));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER));
    CHECK_CUSPARSE(hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    // step 2: create a empty info structure
    // we need one info for csric02 and two info's for csrsv2
    CHECK_CUSPARSE(hipsparseCreateCsric02Info(&info_M));
    // step 3: query how much memory used in csric02 and csrsv2, and allocate the buffer
    CHECK_CUSPARSE(hipsparseCcsric02_bufferSize(handle, m, nnz,
    descr_M, d_csrVal, d_csrRowPtr, d_csrColInd, info_M, &pBufferSize_M));
    pBufferSize = pBufferSize_M;
    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize);
    // step 4: perform analysis of incomplete Cholesky on M
    // perform analysis of triangular solve on L
    // perform analysis of triangular solve on L'
    // The lower triangular part of M has the same sparsity pattern as L, so
    // we can do analysis of csric02 and csrsv2 simultaneously.
    CHECK_CUSPARSE(hipsparseCcsric02_analysis(handle, m, nnz, descr_M,
    d_csrVal, d_csrRowPtr, d_csrColInd, info_M,
    policy_M, pBuffer));
    hipsparseStatus_t status = hipsparseXcsric02_zeroPivot(handle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }
    // CHECK_CUSPARSE(hipsparseCcsrsv2_analysis(handle, trans_L, m, nnz, descr_L,
    // d_csrVal, d_csrRowPtr, d_csrColInd,
    // info_L, policy_L, pBuffer));
    // CHECK_CUSPARSE(hipsparseCcsrsv2_analysis(handle, trans_Lt, m, nnz, descr_L,
    // d_csrVal, d_csrRowPtr, d_csrColInd,
    // info_Lt, policy_Lt, pBuffer));
    // step 5: M = L * L'
    CHECK_CUSPARSE(hipsparseCcsric02(handle, m, nnz, descr_M,
    d_csrVal, d_csrRowPtr, d_csrColInd, info_M, policy_M, pBuffer));
    status = hipsparseXcsric02_zeroPivot(handle, info_M, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }
    // // step 6: solve L*z = x
    // CHECK_CUSPARSE(hipsparseCcsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L,
    // d_csrVal, d_csrRowPtr, d_csrColInd, info_L,
    // d_x, d_z, policy_L, pBuffer));
    // // step 7: solve L'*y = z
    // CHECK_CUSPARSE(hipsparseCcsrsv2_solve(handle, trans_Lt, m, nnz, &alpha, descr_L,
    // d_csrVal, d_csrRowPtr, d_csrColInd, info_Lt,
    // d_z, d_y, policy_Lt, pBuffer));
    hipMemcpy(cuda_val, d_csrVal, sizeof(hipFloatComplex)*nnz, hipMemcpyDeviceToHost);
    // step 6: free resources
    hipFree(pBuffer);
    hipsparseDestroyMatDescr(descr_M);
    hipsparseDestroyMatDescr(descr_L);
    hipsparseDestroyCsric02Info(info_M);
    hipsparseDestroy(handle);
}

void alpha_ic02()
{
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    // Offload data to device
    int* d_csrRowPtr = NULL;
    int* dArow = NULL;
    int* d_csrColInd = NULL;
    hipFloatComplex* d_csrVal = NULL;
    hipFloatComplex* d_x = NULL;
    hipFloatComplex* d_y = NULL;
    hipFloatComplex* d_z = NULL;

    CHECK_CUDA(hipMalloc((void**)&d_x, n * sizeof(hipFloatComplex)));
    CHECK_CUDA(hipMalloc((void**)&d_y, m * sizeof(hipFloatComplex)));
    CHECK_CUDA(hipMalloc((void**)&d_z, m * sizeof(hipFloatComplex)));
    CHECK_CUDA(hipMemcpy(d_x, x_val, n * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y, cuda_y, m * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrColInd, sizeof(int) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrVal, sizeof(hipFloatComplex) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrRowPtr, sizeof(int) * (m + 1)));
    
    CHECK_CUDA(hipMemcpy(
        dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(
        d_csrColInd, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(
        hipMemcpy(d_csrVal, coo_values, nnz * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    alphasparseXcoo2csr(dArow, nnz, m, d_csrRowPtr);
    alphasparseMatDescr_t descr_M = 0;
    alphasparseMatDescr_t descr_L = 0;
    alpha_csric02Info_t info_M = ALPHA_SPARSE_OPAQUE;

    int pBufferSize_M;
    size_t pBufferSize_L;
    size_t pBufferSize_Lt;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const alphasparseSolvePolicy_t policy_M = ALPHA_SPARSE_SOLVE_POLICY_NO_LEVEL;
    const alphasparseSolvePolicy_t policy_L = ALPHA_SPARSE_SOLVE_POLICY_NO_LEVEL;
    const alphasparseSolvePolicy_t policy_Lt = ALPHA_SPARSE_SOLVE_POLICY_NO_LEVEL;
    const alphasparseOperation_t trans_L = ALPHA_SPARSE_OPERATION_NON_TRANSPOSE;
    const alphasparseOperation_t trans_Lt = ALPHA_SPARSE_OPERATION_TRANSPOSE;
      
    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has non-unit diagonal
    alphasparseCreateMatDescr(&descr_M);

    // step 2: create a empty info structure
    // we need one info for csric02 and two info's for csrsv2
    // step 3: query how much memory used in csric02 and csrsv2, and allocate the buffer
    alphasparseCcsric02_bufferSize(handle, m, nnz,
    descr_M, d_csrVal, d_csrRowPtr, d_csrColInd, info_M, &pBufferSize_M);
    alphasparseDnVecDescr_t x{};
    alphasparseCreateDnVec(&x, n, (void*)d_x, ALPHA_R_32F);

    alphasparseDnVecDescr_t y_ict{};
    alphasparseCreateDnVec(&y_ict, m, (void*)d_y, ALPHA_R_32F);
    alphasparseDnVecDescr_t d_zz{};
    alphasparseCreateDnVec(&d_zz, m, (void*)d_z, ALPHA_R_32F);

    alphasparseSpSVDescr_t spsvDescr;
    alphasparseSpSV_createDescr(&spsvDescr);

    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize_M);
    // step 4: perform analysis of incomplete Cholesky on M
    // perform analysis of triangular solve on L
    // perform analysis of triangular solve on L'
    // The lower triangular part of M has the same sparsity pattern as L, so
    // we can do analysis of csric02 and csrsv2 simultaneously.
    alphasparseCcsric02_analysis(handle, m, nnz, descr_M,
    d_csrVal, d_csrRowPtr, d_csrColInd, info_M,
    policy_M, pBuffer);
    // step 5: M = L * L'
    alphasparseCcsric02(handle, m, nnz, descr_M,
    d_csrVal, d_csrRowPtr, d_csrColInd, info_M, policy_M, pBuffer);

    hipMemcpy(ict_val, d_csrVal, sizeof(hipFloatComplex)*nnz, hipMemcpyDeviceToHost);
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  dir_alpha = (alphasparseDirection_t)alpha_args_get_layout(argc, argv);

  // read coo
  alpha_read_coo<hipFloatComplex>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, hipFloatComplex>(nnz, coo_row_index, coo_col_index, coo_values);
  csrRowPtr = (int*)alpha_malloc(sizeof(int) * (m + 1));
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  for (int i = 0; i < 3; i++) {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;
  // init x y
  x_val = (hipFloatComplex*)alpha_malloc(n * sizeof(hipFloatComplex));
  ict_y = (hipFloatComplex*)alpha_malloc(m * sizeof(hipFloatComplex));
  cuda_y = (hipFloatComplex*)alpha_malloc(m * sizeof(hipFloatComplex));

  ict_val = (hipFloatComplex*)alpha_malloc(nnz * sizeof(hipFloatComplex));
  cuda_val = (hipFloatComplex*)alpha_malloc(nnz * sizeof(hipFloatComplex));

  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  cuda_ic02();
  alpha_ic02();
  check((hipFloatComplex*)cuda_val, nnz, (hipFloatComplex*)ict_val, nnz);
  for (int i = 0; i < min(50,nnz); i++) {
    std::cout << cuda_val[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(50,nnz); i++) {
    std::cout << ict_val[i] << ", ";
  }
  std::cout << std::endl;
  
  for (int i = 0; i < nnz; i++)
  {
    if((cuda_val[i].x - ict_val[i].x) / cuda_val[i].x > 1e-6 || (cuda_val[i].y - ict_val[i].y) / cuda_val[i].y > 1e-6) 
    {
      std::cout << "pos " << i << " col indx " << coo_col_index[i] <<" cuda val ("<< std::setprecision(10) << cuda_val[i].x << "," <<std::setprecision(10) << cuda_val[i].y << ") ict val (" << std::setprecision(10) << ict_val[i].x << "," <<std::setprecision(10) << ict_val[i].y << ")" << std::endl;
    }
  }
  return 0;
}