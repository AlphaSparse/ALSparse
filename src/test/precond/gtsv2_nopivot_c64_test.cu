#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "alphasparse.h"

bool check_flag;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int m = 1024, n = 1024, ldb, size;
hipDoubleComplex *hdl, *hd, *hdu, *hictB, *hcudaB;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void
cuda_gtsv2()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  hipDoubleComplex* ddl = NULL;
  hipDoubleComplex* dd = NULL;
  hipDoubleComplex* ddu = NULL;
  hipDoubleComplex* dB = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipDoubleComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipDoubleComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipDoubleComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&dB, sizeof(hipDoubleComplex) * size))

  // Copy data to device
  CHECK_CUDA(
    hipMemcpy(ddl, hdl, sizeof(hipDoubleComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dd, hd, sizeof(hipDoubleComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(ddu, hdu, sizeof(hipDoubleComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(
    dB, hcudaB, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  CHECK_CUSPARSE(hipsparseZgtsv2_nopivot_bufferSizeExt(
    handle, m, n, ddl, dd, ddu, dB, ldb, &buffer_size))

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));
  CHECK_CUSPARSE(
    hipsparseZgtsv2_nopivot(handle, m, n, ddl, dd, ddu, dB, ldb, temp_buffer));

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(
    hcudaB, dB, sizeof(hipDoubleComplex) * size, hipMemcpyDeviceToHost));

  CHECK_CUSPARSE(hipsparseDestroy(handle))
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(temp_buffer))
}

void
alpha_gtsv2()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  hipDoubleComplex* ddl = NULL;
  hipDoubleComplex* dd = NULL;
  hipDoubleComplex* ddu = NULL;
  hipDoubleComplex* dB = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipDoubleComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipDoubleComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipDoubleComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&dB, sizeof(hipDoubleComplex) * size))

  // Copy data to device
  CHECK_CUDA(
    hipMemcpy(ddl, hdl, sizeof(hipDoubleComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dd, hd, sizeof(hipDoubleComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(ddu, hdu, sizeof(hipDoubleComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(
    dB, hictB, sizeof(hipDoubleComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  alphasparseZgtsv2_nopivot_bufferSizeExt(
    handle, m, n, ddl, dd, ddu, dB, ldb, &buffer_size);

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));

  alphasparseZgtsv2_nopivot(handle, m, n, ddl, dd, ddu, dB, ldb, temp_buffer);

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(
    hictB, dB, sizeof(hipDoubleComplex) * size, hipMemcpyDeviceToHost));

  alphasparse_destory_handle(handle);
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(temp_buffer))
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  m = args_get_rows(argc, argv, m);
  n = args_get_cols(argc, argv, n);
  ldb = m;
  size = ldb * n;
  hdl = (hipDoubleComplex*)alpha_malloc(m * sizeof(hipDoubleComplex));
  hd = (hipDoubleComplex*)alpha_malloc(m * sizeof(hipDoubleComplex));
  hdu = (hipDoubleComplex*)alpha_malloc(m * sizeof(hipDoubleComplex));
  hcudaB = (hipDoubleComplex*)alpha_malloc(size * sizeof(hipDoubleComplex));
  hictB = (hipDoubleComplex*)alpha_malloc(size * sizeof(hipDoubleComplex));

  alpha_fill_random(hdl, 55, m);
  hdl[0] = hipDoubleComplex{};
  alpha_fill_random(hd, 99, m);
  alpha_fill_random(hdu, 1, m);
  hdu[m - 1] = hipDoubleComplex{};
  alpha_fill_random(hcudaB, 1, size);
  alpha_fill_random(hictB, 1, size);
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcudaB[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hictB[i] << ", ";
  }
  std::cout << std::endl;
  cuda_gtsv2();
  alpha_gtsv2();

  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcudaB[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hictB[i] << ", ";
  }
  std::cout << std::endl;

  check(hictB, size, hcudaB, size);

  return 0;
}
