#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "alphasparse.h"

bool check_flag;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int m = 8192, n = 1024, ldb, size;
hipFloatComplex *hdl, *hd, *hdu, *hictB, *hcudaB;
float error;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void
cuda_gtsv2()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  hipFloatComplex* ddl = NULL;
  hipFloatComplex* dd = NULL;
  hipFloatComplex* ddu = NULL;
  hipFloatComplex* dB = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipFloatComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipFloatComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipFloatComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&dB, sizeof(hipFloatComplex) * size))

  // Copy data to device
  CHECK_CUDA(
    hipMemcpy(ddl, hdl, sizeof(hipFloatComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dd, hd, sizeof(hipFloatComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(ddu, hdu, sizeof(hipFloatComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(
    dB, hcudaB, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  CHECK_CUSPARSE(hipsparseCgtsv2_bufferSizeExt(
    handle, m, n, ddl, dd, ddu, dB, ldb, &buffer_size))

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));
  CHECK_CUSPARSE(
    hipsparseCgtsv2(handle, m, n, ddl, dd, ddu, dB, ldb, temp_buffer));

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(
    hcudaB, dB, sizeof(hipFloatComplex) * size, hipMemcpyDeviceToHost));

  CHECK_CUSPARSE(hipsparseDestroy(handle))
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(temp_buffer))
}

void
alpha_gtsv2()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  hipFloatComplex* ddl = NULL;
  hipFloatComplex* dd = NULL;
  hipFloatComplex* ddu = NULL;
  hipFloatComplex* dB = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipFloatComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipFloatComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipFloatComplex) * m))
  CHECK_CUDA(hipMalloc((void**)&dB, sizeof(hipFloatComplex) * size))

  // Copy data to device
  CHECK_CUDA(
    hipMemcpy(ddl, hdl, sizeof(hipFloatComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dd, hd, sizeof(hipFloatComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(ddu, hdu, sizeof(hipFloatComplex) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(
    dB, hictB, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  alphasparseCgtsv2_bufferSizeExt(
    handle, m, n, ddl, dd, ddu, dB, ldb, &buffer_size);

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));

  alphasparseCgtsv2(handle, m, n, ddl, dd, ddu, dB, ldb, temp_buffer);

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipMemcpy(
    hictB, dB, sizeof(hipFloatComplex) * size, hipMemcpyDeviceToHost));

  alphasparse_destory_handle(handle);
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(temp_buffer))
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  m = args_get_rows(argc, argv, m);
  n = args_get_cols(argc, argv, n);
  ldb = m;
  size = ldb * n;
  hdl = (hipFloatComplex*)alpha_malloc(m * sizeof(hipFloatComplex));
  hd = (hipFloatComplex*)alpha_malloc(m * sizeof(hipFloatComplex));
  hdu = (hipFloatComplex*)alpha_malloc(m * sizeof(hipFloatComplex));
  hcudaB = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hictB = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));

  alpha_fill_random(hdl, 899, m);
  hdl[0] = {};
  alpha_fill_random(hd, 101, m);
  alpha_fill_random(hdu, 77, m);
  hdu[m - 1] = {};
  for (size_t i = 0; i < m; ++i)
  {
    if (hd[i].x > 0)
      hd[i].x += 1.0;
    else
      hd[i].x -= 1.0;

    if (hd[i].y > 0)
      hd[i].x += 1.0;
    else
      hd[i].x -= 1.0;
  }
  alpha_fill_random(hcudaB, 1, size);
  alpha_fill_random(hictB, 1, size);
  std::cout << "===========hdl=============" << std::endl;
  for (int i = 0; i < min(20, m); i++) {
    std::cout << hdl[i] << ", ";
  }
  std::cout << std::endl << "===========hdl=============" << std::endl;
  std::cout << std::endl << "===========hd=============" << std::endl;
  for (int i = 0; i < min(20, m); i++) {
    std::cout << hd[i] << ", ";
  }
  std::cout << std::endl << "===========hd=============" << std::endl;
  std::cout << std::endl << "===========hdu=============" << std::endl;
  for (int i = 0; i < min(20, m); i++) {
    std::cout << hdu[i] << ", ";
  }
  std::cout << std::endl << "===========hdu=============" << std::endl;
  std::cout << std::endl << "===========hcudaB=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcudaB[i] << ", ";
  }
  std::cout << std::endl << "===========hcudaB=============" << std::endl;
  std::cout << std::endl << "===========hictB=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hictB[i] << ", ";
  }
  std::cout << std::endl << "===========hictB=============" << std::endl;

  cuda_gtsv2();
  alpha_gtsv2();

  std::cout << std::endl << "===========result=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcudaB[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hictB[i] << ", ";
  }
  std::cout << std::endl << "===========result=============" << std::endl;
  check(hictB, size, hcudaB, size, &error);

  return 0;
}
