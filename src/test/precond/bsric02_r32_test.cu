#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;
struct alpha_matrix_descr descrT;

int m, n, nnz, blockdim = 2;
int cuda_nnzb, ict_nnzb;
int* csrRowPtr = NULL;
int *coo_row_index, *coo_col_index;
float* coo_values;

// coo format
float* x_val;
float* ict_y;
float* cuda_y;

float* ict_val;
float* cuda_val;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void cuda_ilu02()
{
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Offload data to device
    int* d_csrRowPtr = NULL;
    int* dArow = NULL;
    int* d_csrColInd = NULL;
    float* d_csrVal = NULL;

    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrColInd, sizeof(int) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrVal, sizeof(float) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrRowPtr, sizeof(int) * (m + 1)));
    
    CHECK_CUDA(hipMemcpy(
        dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(
        d_csrColInd, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(
        hipMemcpy(d_csrVal, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
    alphasparseXcoo2csr(dArow, nnz, m, d_csrRowPtr);
    hipsparseDirection_t dir;
    if(dir_alpha == ALPHA_SPARSE_LAYOUT_ROW_MAJOR) dir = HIPSPARSE_DIRECTION_ROW;
    else dir = HIPSPARSE_DIRECTION_COLUMN;
    float* bsrValA = NULL;
    int* bsrRowPtrA = NULL;
    int* bsrColIndA = NULL;
    int nnzb; //base
    hipsparseMatDescr_t descr;
    hipsparseMatDescr_t descrA;
    int mb = (m + blockdim-1)/blockdim;
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrRowPtrA, sizeof(int) *(mb+1)));

    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));  
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));

    CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(handle, dir, m, n,
                                      descr, d_csrRowPtr, d_csrColInd, blockdim,
                                      descrA, bsrRowPtrA, &nnzb));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrColIndA, sizeof(int)*nnzb));

    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrValA, sizeof(float)*(blockdim*blockdim)*nnzb));
    CHECK_CUSPARSE(hipsparseScsr2bsr(handle, dir, m, n,
                                    descr, d_csrVal, d_csrRowPtr, d_csrColInd, blockdim,
                                    descrA, bsrValA, bsrRowPtrA, bsrColIndA));

    bsric02Info_t info_M = 0;

    int pBufferSize_M;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;
    const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has non-unit diagonal

    // step 2: create a empty info structure
    // we need one info for bsric02 and two info's for csrsv2
    CHECK_CUSPARSE(hipsparseCreateBsric02Info(&info_M));
    // step 3: query how much memory used in bsric02 and csrsv2, and allocate the buffer
    CHECK_CUSPARSE(hipsparseSbsric02_bufferSize(handle, dir, mb, nnzb,
    descrA, bsrValA, bsrRowPtrA, bsrColIndA, blockdim, info_M, &pBufferSize_M));
    pBufferSize = pBufferSize_M;
    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize);
    // step 4: perform analysis of incomplete Cholesky on M
    // perform analysis of triangular solve on L
    // perform analysis of triangular solve on L'
    // The lower triangular part of M has the same sparsity pattern as L, so
    // we can do analysis of bsric02 and csrsv2 simultaneously.
    CHECK_CUSPARSE(hipsparseSbsric02_analysis(handle, dir, mb, nnzb, descrA,
    bsrValA, bsrRowPtrA, bsrColIndA, blockdim, info_M,
    policy_M, pBuffer));
    hipsparseStatus_t status = hipsparseXbsric02_zeroPivot(handle, info_M, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }
    // step 5: M = L * L'
    CHECK_CUSPARSE(hipsparseSbsric02(handle, dir, mb, nnzb, descrA,
    bsrValA, bsrRowPtrA, bsrColIndA, blockdim, info_M, policy_M, pBuffer));
    status = hipsparseXbsric02_zeroPivot(handle, info_M, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
        printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
    }
    // step 6: solve L*z = x
    // CHECK_CUSPARSE(hipsparseScsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L,
    // d_csrVal, d_csrRowPtr, d_csrColInd, info_L,
    // d_x, d_z, policy_L, pBuffer));
    // // step 7: solve L'*y = z
    // CHECK_CUSPARSE(hipsparseScsrsv2_solve(handle, trans_Lt, m, nnz, &alpha, descr_L,
    // d_csrVal, d_csrRowPtr, d_csrColInd, info_Lt,
    // d_z, d_y, policy_Lt, pBuffer));
    free(cuda_val);
    cuda_val = (float *)malloc(sizeof(float)*nnzb);
    hipMemcpy(cuda_val, bsrValA, sizeof(float)*nnzb, hipMemcpyDeviceToHost);
    cuda_nnzb = nnzb;
    // step 6: free resources
    hipFree(pBuffer);
    hipsparseDestroyMatDescr(descr);
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyBsric02Info(info_M);
    hipsparseDestroy(handle);
}

void alpha_ilu02()
{
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    hipsparseHandle_t chandle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&chandle));

    // Offload data to device
    int* d_csrRowPtr = NULL;
    int* dArow = NULL;
    int* d_csrColInd = NULL;
    float* d_csrVal = NULL;
 
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrColInd, sizeof(int) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrVal, sizeof(float) * nnz));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&d_csrRowPtr, sizeof(int) * (m + 1)));

    hipsparseDirection_t dir;
    if(dir_alpha == ALPHA_SPARSE_LAYOUT_ROW_MAJOR) dir = HIPSPARSE_DIRECTION_ROW;
    else dir = HIPSPARSE_DIRECTION_COLUMN;
    
    CHECK_CUDA(hipMemcpy(
        dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(
        d_csrColInd, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(
        hipMemcpy(d_csrVal, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
    alphasparseXcoo2csr(dArow, nnz, m, d_csrRowPtr);
    alpha_bsric02Info_t info_M = ALPHA_SPARSE_OPAQUE;

    float* bsrValA = NULL;
    int* bsrRowPtrA = NULL;
    int* bsrColIndA = NULL;
    int nnzb; //base
    hipsparseMatDescr_t descr;
    hipsparseMatDescr_t descrA;
    int mb = (m + blockdim-1)/blockdim;
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrRowPtrA, sizeof(int) *(mb+1)));

    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));  
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));

    CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(chandle, dir, m, n,
                                      descr, d_csrRowPtr, d_csrColInd, blockdim,
                                      descrA, bsrRowPtrA, &nnzb));
    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrColIndA, sizeof(int)*nnzb));

    PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrValA, sizeof(float)*(blockdim*blockdim)*nnzb));
    CHECK_CUSPARSE(hipsparseScsr2bsr(chandle, dir, m, n,
                                    descr, d_csrVal, d_csrRowPtr, d_csrColInd, blockdim,
                                    descrA, bsrValA, bsrRowPtrA, bsrColIndA));

    int pBufferSize_M;
    void *pBuffer = 0;

    const alphasparseSolvePolicy_t policy_M = ALPHA_SPARSE_SOLVE_POLICY_NO_LEVEL;
    
    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has non-unit diagonal
    // step 2: create a empty info structure
    // we need one info for bsric02 and two info's for csrsv2
    // step 3: query how much memory used in bsric02 and csrsv2, and allocate the buffer
    alphasparseMatDescr_t descr_M = 0;
    alphasparseCreateMatDescr(&descr_M);
    alphasparseSbsric02_bufferSize(handle, dir_alpha, mb, nnzb,
    descr_M, bsrValA, bsrRowPtrA, bsrColIndA, blockdim, info_M, &pBufferSize_M);
    // pBuffer returned by hipMalloc is automatically aligned to 128 bytes.
    hipMalloc((void**)&pBuffer, pBufferSize_M);
    // step 4: perform analysis of incomplete Cholesky on M
    // perform analysis of triangular solve on L
    // perform analysis of triangular solve on L'
    // The lower triangular part of M has the same sparsity pattern as L, so
    // we can do analysis of bsric02 and csrsv2 simultaneously.
    alphasparseSbsric02_analysis(handle, dir_alpha, mb, nnzb, descr_M,
    bsrValA, bsrRowPtrA, bsrColIndA, blockdim, info_M,
    policy_M, pBuffer);
    // step 5: M = L * L'
    alphasparseSbsric02(handle, dir_alpha, mb, nnzb, descr_M,
    bsrValA, bsrRowPtrA, bsrColIndA, blockdim, info_M, policy_M, pBuffer);
    
    free(ict_val);
    ict_val = (float *)malloc(sizeof(float)*nnzb);
    hipMemcpy(ict_val, bsrValA, sizeof(float)*nnzb, hipMemcpyDeviceToHost);
    ict_nnzb = nnzb;
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  dir_alpha = (alphasparseDirection_t)alpha_args_get_layout(argc, argv);

  // read coo
  alpha_read_coo<float>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, float>(nnz, coo_row_index, coo_col_index, coo_values);
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;
  // init x y

  ict_val = (float*)alpha_malloc(nnz * sizeof(float));
  cuda_val = (float*)alpha_malloc(nnz * sizeof(float));

  cuda_ilu02();
  alpha_ilu02();
  check((float*)cuda_val, cuda_nnzb, (float*)ict_val, ict_nnzb);
  for (int i = 0; i < min(20,cuda_nnzb); i++) {
    std::cout << cuda_val[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20,ict_nnzb); i++) {
    std::cout << ict_val[i] << ", ";
  }
  std::cout << std::endl;
  // for (int i = 0; i < ict_nnzb; i++)
  // {
  //   if((cuda_val[i] - ict_val[i]) / cuda_val[i] > 1e-6) 
  //   {
  //     std::cout << "pos " << i << " col indx " << coo_col_index[i] <<" cuda val ("<< std::setprecision(10) << cuda_val[i] << ") ict val (" << std::setprecision(10) << ict_val[i] << ")" << std::endl;
  //   }
  // }
  return 0;
}