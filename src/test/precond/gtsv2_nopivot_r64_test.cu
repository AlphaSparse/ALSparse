#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "alphasparse.h"

bool check_flag;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int m = 1024, n = 1024, ldb, size;
double *hdl, *hd, *hdu, *hictB, *hcudaB;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void
cuda_gtsv2()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  double* ddl = NULL;
  double* dd = NULL;
  double* ddu = NULL;
  double* dB = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(double) * m))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(double) * m))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(double) * m))
  CHECK_CUDA(hipMalloc((void**)&dB, sizeof(double) * size))

  // Copy data to device
  CHECK_CUDA(hipMemcpy(ddl, hdl, sizeof(double) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dd, hd, sizeof(double) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(ddu, hdu, sizeof(double) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dB, hcudaB, sizeof(double) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  CHECK_CUSPARSE(hipsparseDgtsv2_nopivot_bufferSizeExt(
    handle, m, n, ddl, dd, ddu, dB, ldb, &buffer_size))

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));
  CHECK_CUSPARSE(
    hipsparseDgtsv2_nopivot(handle, m, n, ddl, dd, ddu, dB, ldb, temp_buffer));

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(
    hipMemcpy(hcudaB, dB, sizeof(double) * size, hipMemcpyDeviceToHost));

  CHECK_CUSPARSE(hipsparseDestroy(handle))
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(temp_buffer))
}

void
alpha_gtsv2()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  double* ddl = NULL;
  double* dd = NULL;
  double* ddu = NULL;
  double* dB = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(double) * m))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(double) * m))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(double) * m))
  CHECK_CUDA(hipMalloc((void**)&dB, sizeof(double) * size))

  // Copy data to device
  CHECK_CUDA(hipMemcpy(ddl, hdl, sizeof(double) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dd, hd, sizeof(double) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(ddu, hdu, sizeof(double) * m, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dB, hictB, sizeof(double) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  alphasparseDgtsv2_nopivot_bufferSizeExt(
    handle, m, n, ddl, dd, ddu, dB, ldb, &buffer_size);

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));

  alphasparseDgtsv2_nopivot(handle, m, n, ddl, dd, ddu, dB, ldb, temp_buffer);

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(
    hipMemcpy(hictB, dB, sizeof(double) * size, hipMemcpyDeviceToHost));

  alphasparse_destory_handle(handle);
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dB))
  CHECK_CUDA(hipFree(temp_buffer))
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  m = args_get_rows(argc, argv, m);
  n = args_get_cols(argc, argv, n);
  ldb = m;
  size = ldb * n;
  hdl = (double*)alpha_malloc(m * sizeof(double));
  hd = (double*)alpha_malloc(m * sizeof(double));
  hdu = (double*)alpha_malloc(m * sizeof(double));
  hcudaB = (double*)alpha_malloc(size * sizeof(double));
  hictB = (double*)alpha_malloc(size * sizeof(double));

  alpha_fill_random(hdl, 2, m);
  hdl[0] = 0.f;
  alpha_fill_random(hd, 3, m);
  alpha_fill_random(hdu, 4, m);
  hdu[m - 1] = 0.f;
  alpha_fill_random(hcudaB, 1, size);
  alpha_fill_random(hictB, 1, size);
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcudaB[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hictB[i] << ", ";
  }
  std::cout << std::endl;
  cuda_gtsv2();
  alpha_gtsv2();

  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcudaB[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hictB[i] << ", ";
  }
  std::cout << std::endl;

  check(hictB, size, hcudaB, size);

  return 0;
}
