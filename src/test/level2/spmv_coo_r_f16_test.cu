#include "hip/hip_runtime.h"

#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_coo.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;

int m, n, nnz;
int *coo_row_index, *coo_col_index;
half* coo_values;

// coo format
half* x_val;
half* ict_y;
half* cuda_y;

// parms for kernel
const float alpha = 2.f;
const float beta = 3.f;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mv()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  half* dX = NULL;
  half* dY = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  half* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half) * nnz));
  CHECK_CUDA(hipMemcpy(dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dAval, coo_values, nnz * sizeof(half), hipMemcpyHostToDevice));
  hipsparseDnVecDescr_t vecX, vecY;
  hipsparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(half)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(half)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(half), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, cuda_y, m * sizeof(half), hipMemcpyHostToDevice));
  // Create dense vector X
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, dX, HIP_R_16F));
  // Create dense vector y
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, m, dY, HIP_R_16F));
  CHECK_CUSPARSE(hipsparseCreateCoo(&matA,
                                   m,
                                   n,
                                   nnz,
                                   dArow,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_16F));
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         vecX,
                                         &beta,
                                         vecY,
                                         HIP_R_32F,
                                         HIPSPARSE_SPMV_ALG_DEFAULT,
                                         &bufferSize));
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpMV(handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              matA,
                              vecX,
                              &beta,
                              vecY,
                              HIP_R_32F,
                              HIPSPARSE_SPMV_ALG_DEFAULT,
                              dBuffer));
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(cuda_y, dY, sizeof(half) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipsparseDestroy(handle);
}

static void
alpha_mv()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  half* dX = NULL;
  half* dY = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  half* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half) * nnz));
  CHECK_CUDA(hipMemcpy(dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dAval, coo_values, nnz * sizeof(half), hipMemcpyHostToDevice));
  alphasparseDnVecDescr_t vecX, vecY;
  alphasparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(half)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(half)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(half), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, ict_y, m * sizeof(half), hipMemcpyHostToDevice));

  alphasparseDnVecDescr_t x{};
  alphasparseCreateDnVec(&x, n, (void*)dX, ALPHA_R_16F);

  alphasparseDnVecDescr_t y_ict{};
  alphasparseCreateDnVec(&y_ict, m, (void*)dY, ALPHA_R_16F);

  alphasparseSpMatDescr_t coo;
  alphasparseCreateCoo(&coo,
                       m,
                       n,
                       nnz,
                       dArow,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_16F);
  void* dBuffer = NULL;
  size_t bufferSize = 0;
  alphasparseSpMV_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             coo,
                             x,
                             &beta,
                             y_ict,
                             ALPHA_R_32F,
                             ALPHA_SPARSE_SPMV_ALG_COO,
                             &bufferSize);
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
  alphasparseSpMV(handle,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha,
                  coo,
                  x,
                  &beta,
                  y_ict,
                  ALPHA_R_32F,
                  ALPHA_SPARSE_SPMV_ALG_COO,
                  dBuffer);
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(ict_y, dY, sizeof(half) * m, hipMemcpyDeviceToHost));
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);

  // read coo
  alpha_read_coo<half>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, half>(nnz, coo_row_index, coo_col_index, coo_values);
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;
  // init x y
  x_val = (half*)alpha_malloc(n * sizeof(half));
  ict_y = (half*)alpha_malloc(m * sizeof(half));
  cuda_y = (half*)alpha_malloc(m * sizeof(half));

  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  cuda_mv();
  alpha_mv();
  check((half*)cuda_y, m, (half*)ict_y, m);
  for(int i=0;i<20;i++){
    std::cout<<__half2float(cuda_y[i])<<", ";
  }
  std::cout<<std::endl;
  for(int i=0;i<20;i++){
    std::cout<<__half2float(ict_y[i])<<", ";
  }
  return 0;
}
