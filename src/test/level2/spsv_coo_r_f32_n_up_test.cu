
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_coo.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;

int m, n, nnz;
int* csrRowPtr = NULL;
int *coo_row_index, *coo_col_index;
float* coo_values;

// coo format
float* x_val;
float* ict_y;
float* cuda_y;

// parms for kernel
const float alpha = 2.f;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_spsv()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  float* dX = NULL;
  float* dY = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  hipsparseDnVecDescr_t vecX, vecY;
  hipsparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(float)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, cuda_y, m * sizeof(float), hipMemcpyHostToDevice));
  // Create dense vector X
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, dX, HIP_R_32F));
  // Create dense vector y
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, m, dY, HIP_R_32F));
  CHECK_CUSPARSE(hipsparseCreateCoo(&matA,
                                   m,
                                   n,
                                   nnz,
                                   dArow,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));
  hipsparseSpSVDescr_t spsvDescr;
  hipsparseSpSV_createDescr(&spsvDescr);
  hipsparseFillMode_t fillmode = HIPSPARSE_FILL_MODE_UPPER;
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(
    matA, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(fillmode)))
  // Specify Unit|Non-Unit diagonal type.
  hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(
    matA, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype)))
  void* dBuffer = NULL;
  size_t bufferSize = 0;
  // allocate an external buffer for analysis
  CHECK_CUSPARSE(hipsparseSpSV_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         vecX,
                                         vecY,
                                         HIP_R_32F,
                                         HIPSPARSE_SPSV_ALG_DEFAULT,
                                         spsvDescr,
                                         &bufferSize))
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
  CHECK_CUSPARSE(hipsparseSpSV_analysis(handle,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha,
                                       matA,
                                       vecX,
                                       vecY,
                                       HIP_R_32F,
                                       HIPSPARSE_SPSV_ALG_DEFAULT,
                                       spsvDescr,
                                       dBuffer))
  // execute SpSV
  CHECK_CUSPARSE(hipsparseSpSV_solve(handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha,
                                    matA,
                                    vecX,
                                    vecY,
                                    HIP_R_32F,
                                    HIPSPARSE_SPSV_ALG_DEFAULT,
                                    spsvDescr))
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(cuda_y, dY, sizeof(float) * m, hipMemcpyDeviceToHost));

  // destroy matrix/vector descriptors
  CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
  CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX))
  CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY))
  CHECK_CUSPARSE(hipsparseSpSV_destroyDescr(spsvDescr));
  CHECK_CUSPARSE(hipsparseDestroy(handle))
}

static void
alpha_spsv()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  float* dX = NULL;
  float* dY = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(float)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, ict_y, m * sizeof(float), hipMemcpyHostToDevice));

  alphasparseDnVecDescr_t x{};
  alphasparseCreateDnVec(&x, n, (void*)dX, ALPHA_R_32F);

  alphasparseDnVecDescr_t y_ict{};
  alphasparseCreateDnVec(&y_ict, m, (void*)dY, ALPHA_R_32F);

  alphasparseSpMatDescr_t coo;
  alphasparseCreateCoo(&coo,
                       m,
                       n,
                       nnz,
                       dArow,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_32F);
  alphasparseSpSVDescr_t spsvDescr;
  alphasparseSpSV_createDescr(&spsvDescr);
  // Specify Lower|Upper fill mode.
  alphasparse_fill_mode_t fillmode = ALPHA_SPARSE_FILL_MODE_UPPER;
  alphasparseSpMatSetAttribute(
    coo, ALPHASPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(fillmode));
  // Specify Unit|Non-Unit diagonal type.
  alphasparse_diag_type_t diagtype = ALPHA_SPARSE_DIAG_NON_UNIT;
  alphasparseSpMatSetAttribute(
    coo, ALPHASPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype));
  void* dBuffer = NULL;
  size_t bufferSize = 0;
  alphasparseSpSV_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             coo,
                             x,
                             y_ict,
                             ALPHA_R_32F,
                             ALPHA_SPARSE_SPSV_ALG_DEFAULT,
                             spsvDescr,
                             &bufferSize);
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
  alphasparseSpSV_analysis(handle,
                           ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                           &alpha,
                           coo,
                           x,
                           y_ict,
                           ALPHA_R_32F,
                           ALPHA_SPARSE_SPSV_ALG_DEFAULT,
                           spsvDescr,
                           dBuffer);
  alphasparseSpSV_solve(handle,
                        ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        coo,
                        x,
                        y_ict,
                        ALPHA_R_32F,
                        ALPHA_SPARSE_SPSV_ALG_DEFAULT,
                        spsvDescr);
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(ict_y, dY, sizeof(float) * m, hipMemcpyDeviceToHost));
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);

  // read coo
  alpha_read_coo<float>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, float>(nnz, coo_row_index, coo_col_index, coo_values);
  csrRowPtr = (int*)alpha_malloc(sizeof(int) * (m + 1));
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;
  // init x y
  x_val = (float*)alpha_malloc(n * sizeof(float));
  ict_y = (float*)alpha_malloc(m * sizeof(float));
  cuda_y = (float*)alpha_malloc(m * sizeof(float));

  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  cuda_spsv();
  alpha_spsv();
  check((float*)cuda_y, m, (float*)ict_y, m);
  for (int i = 0; i < 20; i++) {
    std::cout << cuda_y[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << ict_y[i] << ", ";
  }
  return 0;
}
