
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_coo.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;

int m, n, nnz;
int *coo_row_index, *coo_col_index;
nv_bfloat162* coo_values;

// coo format
nv_bfloat162* x_val;
nv_bfloat162* ict_y;
nv_bfloat162* cuda_y;

// parms for kernel
const hipFloatComplex alpha = {1.f,2.f};
const hipFloatComplex beta = {3.f,4.f};

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mv()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  nv_bfloat162* dX = NULL;
  nv_bfloat162* dY = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  nv_bfloat162* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(nv_bfloat162) * nnz));
  CHECK_CUDA(hipMemcpy(dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dAval, coo_values, nnz * sizeof(nv_bfloat162), hipMemcpyHostToDevice));
  hipsparseDnVecDescr_t vecX, vecY;
  hipsparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(nv_bfloat162)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(nv_bfloat162)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(nv_bfloat162), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, cuda_y, m * sizeof(nv_bfloat162), hipMemcpyHostToDevice));
  // Create dense vector X
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, dX, HIP_C_16BF));
  // Create dense vector y
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, m, dY, HIP_C_16BF));
  CHECK_CUSPARSE(hipsparseCreateCoo(&matA,
                                   m,
                                   n,
                                   nnz,
                                   dArow,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_C_16BF));
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         vecX,
                                         &beta,
                                         vecY,
                                         HIP_C_32F,
                                         HIPSPARSE_SPMV_ALG_DEFAULT,
                                         &bufferSize));
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpMV(handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              matA,
                              vecX,
                              &beta,
                              vecY,
                              HIP_C_32F,
                              HIPSPARSE_SPMV_ALG_DEFAULT,
                              dBuffer));
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(cuda_y, dY, sizeof(nv_bfloat162) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipsparseDestroy(handle);
}

static void
alpha_mv()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  nv_bfloat162* dX = NULL;
  nv_bfloat162* dY = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  nv_bfloat162* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(nv_bfloat162) * nnz));
  CHECK_CUDA(hipMemcpy(dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dAval, coo_values, nnz * sizeof(nv_bfloat162), hipMemcpyHostToDevice));
  alphasparseDnVecDescr_t vecX, vecY;
  alphasparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(nv_bfloat162)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(nv_bfloat162)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(nv_bfloat162), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, ict_y, m * sizeof(nv_bfloat162), hipMemcpyHostToDevice));

  alphasparseDnVecDescr_t x{};
  alphasparseCreateDnVec(&x, n, (void*)dX, ALPHA_C_16BF);

  alphasparseDnVecDescr_t y_ict{};
  alphasparseCreateDnVec(&y_ict, m, (void*)dY, ALPHA_C_16BF);

  alphasparseSpMatDescr_t coo;
  alphasparseCreateCoo(&coo,
                       m,
                       n,
                       nnz,
                       dArow,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_C_16BF);
  void* dBuffer = NULL;
  size_t bufferSize = 0;
  alphasparseSpMV_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             coo,
                             x,
                             &beta,
                             y_ict,
                             ALPHA_C_32F,
                             ALPHA_SPARSE_SPMV_ALG_COO,
                             &bufferSize);
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
  alphasparseSpMV(handle,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha,
                  coo,
                  x,
                  &beta,
                  y_ict,
                  ALPHA_C_32F,
                  ALPHA_SPARSE_SPMV_ALG_COO,
                  dBuffer);
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(ict_y, dY, sizeof(nv_bfloat162) * m, hipMemcpyDeviceToHost));
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);

  // read coo
  alpha_read_coo<nv_bfloat162>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, nv_bfloat162>(nnz, coo_row_index, coo_col_index, coo_values);
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;
  // init x y
  x_val = (nv_bfloat162*)alpha_malloc(n * sizeof(nv_bfloat162));
  ict_y = (nv_bfloat162*)alpha_malloc(m * sizeof(nv_bfloat162));
  cuda_y = (nv_bfloat162*)alpha_malloc(m * sizeof(nv_bfloat162));

  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  cuda_mv();
  alpha_mv();
  check((nv_bfloat162*)cuda_y, m, (nv_bfloat162*)ict_y, m);
  for(int i=0;i<20;i++){
    std::cout<<cuda_y[i]<<", ";
  }
  std::cout<<std::endl;
  for(int i=0;i<20;i++){
    std::cout<<ict_y[i]<<", ";
  }
  return 0;
}
