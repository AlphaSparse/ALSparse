#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "alphasparse.h"

#include "../test_common.h"

const char *file;
bool check_flag;
int iter;

// sparse vector
int *alpha_x_idx;
int *cuda_x_idx;
hipDoubleComplex *x_val, *cuda_y, *alpha_y;
const hipDoubleComplex alpha = {1.1f,2.4f};
const hipDoubleComplex beta = {3.2f,4.3f};
alphasparseOperation_t transAT;

int m, n, nnz;
int *coo_row_index, *coo_col_index;
hipDoubleComplex* coo_values;
int lda;
int sizeA;
hipDoubleComplex * A;

int idx_n = 1000;

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        exit(-1);                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(-1);                                                   \
    }                                                                          \
}

static void cuda_gemvi() {
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    hipDoubleComplex *dx_val = NULL;
    hipDoubleComplex *dy = NULL;
    hipDoubleComplex *dA = NULL;

    hipsparseOperation_t transA;
    if(transAT == ALPHA_SPARSE_OPERATION_NON_TRANSPOSE) transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    else if(transAT == ALPHA_SPARSE_OPERATION_TRANSPOSE) transA = HIPSPARSE_OPERATION_TRANSPOSE;
    else transA = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

    CHECK_CUDA(hipMalloc((void **)&dx_idx, sizeof(int) * nnz));
    CHECK_CUDA(hipMalloc((void **)&dx_val, sizeof(hipDoubleComplex) * n));
    CHECK_CUDA(hipMalloc((void **)&dy, sizeof(hipDoubleComplex) * m));
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(hipDoubleComplex) * sizeA));

    CHECK_CUDA(hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dx_val, x_val, sizeof(hipDoubleComplex) * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dy, cuda_y, sizeof(hipDoubleComplex) * m, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA, A, sizeof(hipDoubleComplex) * sizeA, hipMemcpyHostToDevice));   

    int pBufferSize;
    void * pBuffer;
    hipsparseZgemvi_bufferSize(handle, transA, m, n, nnz, &pBufferSize);
    CHECK_CUDA(hipMalloc((void **)&pBuffer, sizeof(HIP_R_32F) * pBufferSize));
    hipsparseIndexBase_t base = HIPSPARSE_INDEX_BASE_ZERO;
    CHECK_CUSPARSE(hipsparseZgemvi(handle, transA, m, n, &alpha, dA, lda, nnz, dx_val, dx_idx, &beta, dy, base, pBuffer));

    // Device synchronization
    hipDeviceSynchronize();

    CHECK_CUDA(hipMemcpy(cuda_y, dy, sizeof(hipDoubleComplex) * m, hipMemcpyDeviceToHost));

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    hipFree(dA);
    hipsparseDestroy(handle);
}

static void alpha_gemvi()
{
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    // Offload data to device
    int *dx_idx = NULL;
    hipDoubleComplex *dx_val     = NULL;
    hipDoubleComplex *dy         = NULL;
    hipDoubleComplex *dA         = NULL;

    CHECK_CUDA(hipMalloc((void **)&dx_idx, sizeof(int) * nnz));
    CHECK_CUDA(hipMalloc((void **)&dx_val, sizeof(hipDoubleComplex) * n));
    CHECK_CUDA(hipMalloc((void **)&dy, sizeof(hipDoubleComplex) * m));
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(hipDoubleComplex) * sizeA));

    CHECK_CUDA(hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dx_val, x_val, sizeof(hipDoubleComplex) * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dy, alpha_y, sizeof(hipDoubleComplex) * m, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA, A, sizeof(hipDoubleComplex) * sizeA, hipMemcpyHostToDevice));

    int pBufferSize;
    void * pBuffer = NULL;
    hipMalloc((void **)&pBuffer, sizeof(hipDoubleComplex) );
    alphasparseIndexBase_t base = ALPHA_SPARSE_INDEX_BASE_ZERO;
    alphasparseZgemvi(handle, transAT, m, n, &alpha, dA, lda, nnz, dx_val, dx_idx, &beta, dy, base, pBuffer),

    // Device synchronization
    hipDeviceSynchronize();

    CHECK_CUDA(hipMemcpy(alpha_y, dy, sizeof(hipDoubleComplex) * m, hipMemcpyDeviceToHost));

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    hipFree(dA);
    // alphasparse_destory_handle(handle);
}

int main(int argc, const char *argv[])
{
    // args
    args_help(argc, argv);
    file = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    transAT = alpha_args_get_transA(argc, argv);

    alpha_read_coo<hipDoubleComplex>(
      file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);

    if(transAT != ALPHA_SPARSE_OPERATION_NON_TRANSPOSE) 
    {
      int t = n;
      n = m;
      m = t;
    }
    float spasity = (float)nnz / (m * n) ;
    nnz = m * spasity; 
    nnz = nnz > n ? n : nnz;
    lda = n;
    sizeA = m * lda;

    alpha_x_idx=(int *)alpha_memalign(sizeof(int) * nnz, DEFAULT_ALIGNMENT);
    cuda_x_idx = (int *)alpha_memalign(sizeof(int) * nnz,  DEFAULT_ALIGNMENT);

    x_val   = (hipDoubleComplex *)alpha_memalign(sizeof(hipDoubleComplex) * n, DEFAULT_ALIGNMENT);
    alpha_y = (hipDoubleComplex *)alpha_memalign(sizeof(hipDoubleComplex) * m, DEFAULT_ALIGNMENT);
    cuda_y  = (hipDoubleComplex *)alpha_memalign(sizeof(hipDoubleComplex) * m, DEFAULT_ALIGNMENT);
    A       = (hipDoubleComplex *)alpha_memalign(sizeof(hipDoubleComplex) * sizeA, DEFAULT_ALIGNMENT);

    alpha_fill_random(alpha_y, 1, m);
    memcpy(cuda_y, alpha_y, sizeof(hipDoubleComplex)* m);
    alpha_fill_random(x_val, 0, n);
    alpha_fill_random(A, 1, sizeA);
    int fen = n / nnz;
    for (int i = 0; i < nnz; i++) {
        if(i == 0)
        {
          alpha_x_idx[i] = rand() % fen;
          cuda_x_idx[i]  = alpha_x_idx[i];
        }
        else
        {
          alpha_x_idx[i] = alpha_x_idx[i - 1] + rand() % fen;
          cuda_x_idx[i]  = alpha_x_idx[i];
        }
    }

    alpha_gemvi();

    if (check_flag) {
        cuda_gemvi();
        check(alpha_y, m, cuda_y, m);

        for(int i=0;i<10;i++)
            std::cout<<alpha_y[i]<<"  ";

        std::cout<<std::endl;
        for(int i=0;i<10;i++)
            std::cout<<cuda_y[i]<<"  ";   

        std::cout<<std::endl;   
    }
    printf("\n");

    alpha_free(x_val);
    alpha_free(cuda_x_idx);
    alpha_free(alpha_x_idx);
    return 0;
}