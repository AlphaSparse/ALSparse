
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char *file, *metrics_file;
int thread_num;
bool check_flag;
bool metrics_flag;
int iter, warm_up = 0, trials = 1;
float error;
alphasparseOperation_t transA;

int m, n, nnz;
int *csrRowPtr = NULL;
int *coo_row_index, *coo_col_index;
float *coo_values;

// coo format
float *x_val;
float *ict_y;
float *cuda_y;

// parms for kernel
const float alpha = 2.f;
const float beta = 3.f;

std::vector<double> cuda_times, alpha_times;

#define CHECK_CUDA(func)                                         \
  {                                                              \
    hipError_t status = (func);                                 \
    if (status != hipSuccess)                                   \
    {                                                            \
      printf("CUDA API failed at line %d with error: %s (%d)\n", \
             __LINE__,                                           \
             hipGetErrorString(status),                         \
             status);                                            \
      exit(-1);                                                  \
    }                                                            \
  }

#define CHECK_CUSPARSE(func)                                         \
  {                                                                  \
    hipsparseStatus_t status = (func);                                \
    if (status != HIPSPARSE_STATUS_SUCCESS)                           \
    {                                                                \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
             __LINE__,                                               \
             hipsparseGetErrorString(status),                         \
             status);                                                \
      exit(-1);                                                      \
    }                                                                \
  }

static void
cuda_mv()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  float *dX = NULL;
  float *dY = NULL;
  int *dCsrRowPtr = NULL;
  int *dArow = NULL;
  int *dAcol = NULL;
  float *dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dCsrRowPtr, sizeof(int) * (m + 1)));
  CHECK_CUDA(hipMemcpy(
      dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
      dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);
  hipsparseDnVecDescr_t vecX, vecY;
  hipsparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void **)&dX, n * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&dY, m * sizeof(float)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, cuda_y, m * sizeof(float), hipMemcpyHostToDevice));
  // Create dense vector X
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, dX, HIP_R_32F));
  // Create dense vector y
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, m, dY, HIP_R_32F));
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
                                   m,
                                   n,
                                   nnz,
                                   dCsrRowPtr,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         vecX,
                                         &beta,
                                         vecY,
                                         HIP_R_32F,
                                         HIPSPARSE_SPMV_ALG_DEFAULT,
                                         &bufferSize));
  void *dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  for (int i = 0; i < warm_up; ++i)
  {
    CHECK_CUSPARSE(hipsparseSpMV(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha,
                                matA,
                                vecX,
                                &beta,
                                vecY,
                                HIP_R_32F,
                                HIPSPARSE_SPMV_ALG_DEFAULT,
                                dBuffer));
    hipDeviceSynchronize();
  }

  for (int i = 0; i < trials; ++i)
  {
    double time = get_time_us();
    CHECK_CUSPARSE(hipsparseSpMV(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha,
                                matA,
                                vecX,
                                &beta,
                                vecY,
                                HIP_R_32F,
                                HIPSPARSE_SPMV_ALG_DEFAULT,
                                dBuffer));
    hipDeviceSynchronize();
    time = (get_time_us() - time) / (1e3);
    cuda_times.push_back(time);
  }
  CHECK_CUDA(hipMemcpy(cuda_y, dY, sizeof(float) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipsparseDestroy(handle);
}

static void
alpha_mv()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  float *dX = NULL;
  float *dY = NULL;
  int *dCsrRowPtr = NULL;
  int *dArow = NULL;
  int *dAcol = NULL;
  float *dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void **)&dCsrRowPtr, sizeof(int) * (m + 1)));

  CHECK_CUDA(hipMemcpy(
      dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
      dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
      hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);

  alphasparseDnVecDescr_t vecX, vecY;
  alphasparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void **)&dX, n * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&dY, m * sizeof(float)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, ict_y, m * sizeof(float), hipMemcpyHostToDevice));

  alphasparseDnVecDescr_t x{};
  alphasparseCreateDnVec(&x, n, (void *)dX, ALPHA_R_32F);

  alphasparseDnVecDescr_t y_ict{};
  alphasparseCreateDnVec(&y_ict, m, (void *)dY, ALPHA_R_32F);

  alphasparseSpMatDescr_t csr;
  alphasparseCreateCsr(&csr,
                       m,
                       n,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_32F);
  void *dBuffer = NULL;
  size_t bufferSize = 0;
  alphasparseSpMV_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             csr,
                             x,
                             &beta,
                             y_ict,
                             ALPHA_R_32F,
                             ALPHA_SPARSE_SPMV_ALG_COO,
                             &bufferSize);
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
  for (int i = 0; i < warm_up; ++i)
  {
    alphasparseSpMV(handle,
                    ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    csr,
                    x,
                    &beta,
                    y_ict,
                    ALPHA_R_32F,
                    ALPHA_SPARSE_SPMV_ALG_COO,
                    dBuffer);
    hipDeviceSynchronize();
  }

  for (int i = 0; i < trials; ++i)
  {
    double time = get_time_us();
    alphasparseSpMV(handle,
                    ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    csr,
                    x,
                    &beta,
                    y_ict,
                    ALPHA_R_32F,
                    ALPHA_SPARSE_SPMV_ALG_COO,
                    dBuffer);
    hipDeviceSynchronize();
    time = (get_time_us() - time) / (1e3);
    alpha_times.push_back(time);
  }
  CHECK_CUDA(hipMemcpy(ict_y, dY, sizeof(float) * m, hipMemcpyDeviceToHost));
}

int main(int argc, const char *argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  metrics_file = args_save_metrics_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  metrics_flag = args_get_if_calculate_metrics(argc, argv);
  transA = alpha_args_get_transA(argc, argv);

  // read coo
  alpha_read_coo<float>(
      file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, float>(nnz, coo_row_index, coo_col_index, coo_values);
  csrRowPtr = (int *)alpha_malloc(sizeof(int) * (m + 1));
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE)
  {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;
  // init x y
  x_val = (float *)alpha_malloc(n * sizeof(float));
  ict_y = (float *)alpha_malloc(m * sizeof(float));
  cuda_y = (float *)alpha_malloc(m * sizeof(float));

  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  if (metrics_flag)
  {
    warm_up = 1;
    trials = 3;
  }
  else if (!metrics_flag && check_flag)
  {
    warm_up = 0;
    trials = 1;
  }
  alpha_mv();
  if (check_flag || metrics_flag)
  {
    cuda_mv();
    if (metrics_flag)
    {

      // 打开文件，如果文件不存在则创建一个新文件
      std::ofstream filename(metrics_file, std::ios::app);
      int check_pass = check((float *)cuda_y, m, (float *)ict_y, m);
      char *if_pass = "";
      if (check_pass == 0)
      {
        if_pass = "PASS";
      }
      else
      {
        if_pass = "FAIL";
      }
      double time = get_avg_time(alpha_times);
      double bandwidth = static_cast<double>(sizeof(float)) * (2 * m + nnz) + sizeof(int) * (m + 1 + nnz) / time / 1e6;
      double gflops = static_cast<double>(2 * nnz) / time / 1e6;
      filename << "Parameters:LIB=\"AlphaSparse\",FUNCTIONS=\"SpMV\",FORMAT=\"CSR\",OPERATION=\"N_TRANS\",ALGO=\"ADAPTIVE\",A DATATYPE=\"R_32F\",X DATATYPE=\"R_32F\",Y DATATYPE=\"R_32F\",COMPUTE=\"R_32F\"\n";
      filename << "Results:TEST Mat=" << file << ",Residual=" << error << ",CHECK=" << if_pass << ",time=" << time << ",Perf=" << gflops << "\n";
      time = get_avg_time(cuda_times);
      bandwidth = static_cast<double>(sizeof(float)) * (2 * m + nnz) + sizeof(int) * (m + 1 + nnz) / time / 1e6;
      gflops = static_cast<double>(2 * nnz) / time / 1e6;
      filename << "Parameters:LIB=\"cuSPARSE\",FUNCTIONS=\"SpMV\",FORMAT=\"CSR\",OPERATION=\"N_TRANS\",ALGO=\"DEFAULT\",A DATATYPE=\"R_32F\",X DATATYPE=\"R_32F\",Y DATATYPE=\"R_32F\",COMPUTE=\"R_32F\"\n";
      filename << "Results:TEST Mat=" << file << ",Residual=0,CHECK=PASS,time=" << time << ",Perf=" << gflops << "\n";
      filename.close();
    }
    else
    {
      check((float *)cuda_y, m, (float *)ict_y, m);
    }
  }

  // for (int i = 0; i < 20; i++) {
  //   std::cout << cuda_y[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << ict_y[i] << ", ";
  // }
  return 0;
}
