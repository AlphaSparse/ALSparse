
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/csr2csc.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;

int m, n, nnz;
int* csrRowPtr = NULL;
int *coo_row_index, *coo_col_index;
half2* coo_values;

// coo format
half2* x_val;
half2* ict_y;
half2* cuda_y;

// parms for kernel
const hipFloatComplex alpha = {2.f, 3.f};
const hipFloatComplex beta = {3.f, 2.f};

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mv()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  half2* dX = NULL;
  half2* dY = NULL;
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  half2* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half2) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (m + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half2), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);
  hipsparseDnVecDescr_t vecX, vecY;
  hipsparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(half2)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(half2)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(half2), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, cuda_y, m * sizeof(half2), hipMemcpyHostToDevice));
  // Create dense vector X
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, dX, HIP_C_16F));
  // Create dense vector y
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, m, dY, HIP_C_16F));
  int* dCscColPtr = NULL;
  int* dCscRowInd = NULL;
  half2* dCscVal = NULL;
  size_t csc_bufferSize = 0;
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscRowInd, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscVal, sizeof(half2) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscColPtr, sizeof(int) * (n + 1)));
  hipsparseCsr2cscEx2_bufferSize(handle,
    m,
    n,
    nnz,
    dAval,
    dCsrRowPtr,
    dAcol,
    dCscVal,
    dCscColPtr,
    dCscRowInd,
    HIP_C_16F,
    HIPSPARSE_ACTION_NUMERIC,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIPSPARSE_CSR2CSC_ALG1,
    &csc_bufferSize);
  void* csc_dBuffer = NULL;
  CHECK_CUDA(hipMalloc((void**)&csc_dBuffer, csc_bufferSize * sizeof(half2)));
  hipsparseCsr2cscEx2(handle,
    m,
    n,
    nnz,
    dAval,
    dCsrRowPtr,
    dAcol,
    dCscVal,
    dCscColPtr,
    dCscRowInd,
    HIP_C_16F,
    HIPSPARSE_ACTION_NUMERIC,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIPSPARSE_CSR2CSC_ALG1,
    csc_dBuffer);

  hipsparseSpMatDescr_t csc;
  hipsparseCreateCsc(&csc,
    m,
    n,
    nnz,
    dCscColPtr,
    dCscRowInd,
    dCscVal,
    HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIP_C_16F);
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         csc,
                                         vecX,
                                         &beta,
                                         vecY,
                                         HIP_C_32F,
                                         HIPSPARSE_SPMV_ALG_DEFAULT,
                                         &bufferSize));
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpMV(handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              csc,
                              vecX,
                              &beta,
                              vecY,
                              HIP_C_32F,
                              HIPSPARSE_SPMV_ALG_DEFAULT,
                              dBuffer));
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(cuda_y, dY, sizeof(half2) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipsparseDestroy(handle);
}

static void
alpha_mv()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  half2* dX = NULL;
  half2* dY = NULL;
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  half2* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half2) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (m + 1)));

  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half2), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);
  alphasparseDnVecDescr_t vecX, vecY;
  alphasparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(half2)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(half2)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(half2), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, ict_y, m * sizeof(half2), hipMemcpyHostToDevice));

  alphasparseDnVecDescr_t x{};
  alphasparseCreateDnVec(&x, n, (void*)dX, ALPHA_C_16F);

  alphasparseDnVecDescr_t y_ict{};
  alphasparseCreateDnVec(&y_ict, m, (void*)dY, ALPHA_C_16F);

  alphasparseSpMatDescr_t csr;
  alphasparseCreateCsr(&csr,
                       m,
                       n,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_C_16F);
  alphasparseSpMatDescr_t csc;
  alphasparseCsr2csc<int, half2>(csr, csc);
  
  void* dBuffer = NULL;
  size_t bufferSize = 0;
  alphasparseSpMV_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             csc,
                             x,
                             &beta,
                             y_ict,
                             ALPHA_C_32F,
                             ALPHA_SPARSE_SPMV_ALG_DEFAULT,
                             &bufferSize);
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize * sizeof(half2)))
  alphasparseSpMV(handle,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha,
                  csc,
                  x,
                  &beta,
                  y_ict,
                  ALPHA_C_32F,
                  ALPHA_SPARSE_SPMV_ALG_DEFAULT,
                  dBuffer);
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(ict_y, dY, sizeof(half2) * m, hipMemcpyDeviceToHost));
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);

  // read coo
  alpha_read_coo<half2>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, half2>(nnz, coo_row_index, coo_col_index, coo_values);
  csrRowPtr = (int*)alpha_malloc(sizeof(int) * (m + 1));
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;
  // init x y
  x_val = (half2*)alpha_malloc(n * sizeof(half2));
  ict_y = (half2*)alpha_malloc(m * sizeof(half2));
  cuda_y = (half2*)alpha_malloc(m * sizeof(half2));
  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  std::cout<<"\nx_val: \n";
  for (int i = 0; i < 20; i++) {
    std::cout << x_val[i] << ", ";
  }
  std::cout<<"\ncuda_y: \n";
  for (int i = 0; i < 20; i++) {
    std::cout << cuda_y[i] << ", ";
  }
  std::cout<<"\nict_y: \n";
  for (int i = 0; i < 20; i++) {
    std::cout << ict_y[i] << ", ";
  }
  cuda_mv();
  alpha_mv();
  check((half2*)cuda_y, m, (half2*)ict_y, m);
  for (int i = 0; i < 20; i++) {
    std::cout << cuda_y[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << ict_y[i] << ", ";
  }
  return 0;
}
