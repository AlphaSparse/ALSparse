#include "alphasparse_gtsv2.h"
#include <iostream>

alphasparseStatus_t
alphasparseSgtsv2_bufferSizeExt(alphasparseHandle_t handle,
                                int m,
                                int n,
                                const float* dl,
                                const float* d,
                                const float* du,
                                float* B,
                                int ldb,
                                size_t* bufferSizeInBytes)
{
  gtsv_buffer_size_template<float>(
    handle, m, n, dl, d, du, B, ldb, bufferSizeInBytes);
  return ALPHA_SPARSE_STATUS_SUCCESS;
}

alphasparseStatus_t
alphasparseDgtsv2_bufferSizeExt(alphasparseHandle_t handle,
                                int m,
                                int n,
                                const double* dl,
                                const double* d,
                                const double* du,
                                double* B,
                                int ldb,
                                size_t* bufferSizeInBytes)
{
  gtsv_buffer_size_template<double>(
    handle, m, n, dl, d, du, B, ldb, bufferSizeInBytes);
  return ALPHA_SPARSE_STATUS_SUCCESS;
}

alphasparseStatus_t
alphasparseCgtsv2_bufferSizeExt(alphasparseHandle_t handle,
                                int m,
                                int n,
                                const hipFloatComplex* dl,
                                const hipFloatComplex* d,
                                const hipFloatComplex* du,
                                hipFloatComplex* B,
                                int ldb,
                                size_t* bufferSizeInBytes)
{
  gtsv_buffer_size_template<hipFloatComplex>(
    handle, m, n, dl, d, du, B, ldb, bufferSizeInBytes);
  return ALPHA_SPARSE_STATUS_SUCCESS;
}

alphasparseStatus_t
alphasparseZgtsv2_bufferSizeExt(alphasparseHandle_t handle,
                                int m,
                                int n,
                                const hipDoubleComplex* dl,
                                const hipDoubleComplex* d,
                                const hipDoubleComplex* du,
                                hipDoubleComplex* B,
                                int ldb,
                                size_t* bufferSizeInBytes)
{
  gtsv_buffer_size_template<hipDoubleComplex>(
    handle, m, n, dl, d, du, B, ldb, bufferSizeInBytes);
  return ALPHA_SPARSE_STATUS_SUCCESS;
}

alphasparseStatus_t
alphasparseSgtsv2(alphasparseHandle_t handle,
                  int m,
                  int n,
                  const float* dl,
                  const float* d,
                  const float* du,
                  float* B,
                  int ldb,
                  void* pBuffer)
{
  gtsv_template<float>(handle, m, n, dl, d, du, B, ldb, pBuffer);
  return ALPHA_SPARSE_STATUS_SUCCESS;
}

alphasparseStatus_t
alphasparseDgtsv2(alphasparseHandle_t handle,
                  int m,
                  int n,
                  const double* dl,
                  const double* d,
                  const double* du,
                  double* B,
                  int ldb,
                  void* pBuffer)
{
  gtsv_template<double>(handle, m, n, dl, d, du, B, ldb, pBuffer);
  return ALPHA_SPARSE_STATUS_SUCCESS;
}

alphasparseStatus_t
alphasparseCgtsv2(alphasparseHandle_t handle,
                  int m,
                  int n,
                  const hipFloatComplex* dl,
                  const hipFloatComplex* d,
                  const hipFloatComplex* du,
                  hipFloatComplex* B,
                  int ldb,
                  void* pBuffer)
{
  gtsv_template<hipFloatComplex>(handle, m, n, dl, d, du, B, ldb, pBuffer);
  return ALPHA_SPARSE_STATUS_SUCCESS;
}

alphasparseStatus_t
alphasparseZgtsv2(alphasparseHandle_t handle,
                  int m,
                  int n,
                  const hipDoubleComplex* dl,
                  const hipDoubleComplex* d,
                  const hipDoubleComplex* du,
                  hipDoubleComplex* B,
                  int ldb,
                  void* pBuffer)
{
  gtsv_template<hipDoubleComplex>(handle, m, n, dl, d, du, B, ldb, pBuffer);
  return ALPHA_SPARSE_STATUS_SUCCESS;
}