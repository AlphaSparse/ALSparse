
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "../../format/csr2csc.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
int8_t* coo_values;

// parms for kernel
int8_t *hmatB;
float *matC_ict, *matC_roc;
long long C_rows, C_cols;
long long B_cols;
long long ldb, ldc;
long long B_size, C_size;
const float alpha = 2;
const float beta = 3;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  int8_t* dAval = NULL;

  int nnz = rnnz;

  int8_t* dmatB = NULL;
  float* dmatC = NULL;

  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(int8_t) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(int8_t), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);
  int* dCscColPtr = NULL;
  int* dCscRowInd = NULL;
  int8_t* dCscVal = NULL;
  size_t csc_bufferSize = 0;
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscRowInd, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscVal, sizeof(int8_t) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCscColPtr, sizeof(int) * (A_cols + 1)));
  hipsparseCsr2cscEx2_bufferSize(handle,
                                A_rows,
                                A_cols,
                                nnz,
                                dAval,
                                dCsrRowPtr,
                                dAcol,
                                dCscVal,
                                dCscColPtr,
                                dCscRowInd,
                                HIP_R_8I,
                                HIPSPARSE_ACTION_NUMERIC,
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIPSPARSE_CSR2CSC_ALG1,
                                &csc_bufferSize);
  void* csc_dBuffer = NULL;
  CHECK_CUDA(hipMalloc((void**)&csc_dBuffer, csc_bufferSize * sizeof(int8_t)));
  hipsparseCsr2cscEx2(handle,
                     A_rows,
                     A_cols,
                     nnz,
                     dAval,
                     dCsrRowPtr,
                     dAcol,
                     dCscVal,
                     dCscColPtr,
                     dCscRowInd,
                     HIP_R_8I,
                     HIPSPARSE_ACTION_NUMERIC,
                     HIPSPARSE_INDEX_BASE_ZERO,
                     HIPSPARSE_CSR2CSC_ALG1,
                     csc_dBuffer);

  hipsparseSpMatDescr_t csc;
  hipsparseCreateCsc(&csc,
                    A_rows,
                    A_cols,
                    nnz,
                    dCscColPtr,
                    dCscRowInd,
                    dCscVal,
                    HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO,
                    HIP_R_8I);

  hipMalloc((void**)&dmatB, sizeof(int8_t) * B_size);
  hipMalloc((void**)&dmatC, sizeof(float) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(int8_t) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(int8_t), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(int8_t) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(float) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, HIP_R_8I, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_R_32F, HIPSPARSE_ORDER_COL))
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         csc,
                                         matB,
                                         &beta,
                                         matC,
                                         HIP_R_32F,
                                         HIPSPARSE_SPMM_ALG_DEFAULT,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpMM(handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              csc,
                              matB,
                              &beta,
                              matC,
                              HIP_R_32F,
                              HIPSPARSE_SPMM_ALG_DEFAULT,
                              dBuffer))
  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(float), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  int8_t* dAval = NULL;

  int nnz = rnnz;

  int8_t* dmatB = NULL;
  float* dmatC = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(int8_t) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));

  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(int8_t), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  alphasparseSpMatDescr_t csr;
  alphasparseCreateCsr(&csr,
                       A_rows,
                       A_cols,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_8I);
  alphasparseSpMatDescr_t csc;
  alphasparseCsr2csc<int, int8_t>(csr, csc);

  hipMalloc((void**)&dmatB, sizeof(int8_t) * B_size);
  hipMalloc((void**)&dmatC, sizeof(float) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(int8_t) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(int8_t), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(int8_t) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(float) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, ALPHA_R_8I, ALPHASPARSE_ORDER_COL);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_R_32F, ALPHASPARSE_ORDER_COL);
  size_t bufferSize = 0;
  alphasparseSpMM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             csc,
                             matB,
                             &beta,
                             matC,
                             ALPHA_R_32F,
                             ALPHASPARSE_SPMM_ALG_DEFAULT,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  alphasparseSpMM(handle,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha,
                  csc,
                  matB,
                  &beta,
                  matC,
                  ALPHA_R_32F,
                  ALPHASPARSE_SPMM_ALG_DEFAULT,
                  dBuffer);
  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(float), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  transB = alpha_args_get_transB(argc, argv);

  // read coo
  alpha_read_coo<int8_t>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, int8_t>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, A_rows); // 默认C是方阵
  C_rows = A_rows;
  C_cols = columns;
  B_cols = columns;
  ldb = A_cols;
  ldc = C_rows;
  B_size = ldb * B_cols;
  C_size = ldc * B_cols;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_row_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_col_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;
  // init x y
  // init B C
  hmatB = (int8_t*)alpha_malloc(B_size * sizeof(int8_t));
  matC_ict = (float*)alpha_malloc(C_size * sizeof(float));
  matC_roc = (float*)alpha_malloc(C_size * sizeof(float));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_fill_random(matC_roc, 1, C_size);

  cuda_mm();
  alpha_mm();

  for (int i = 0; i < 20; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  check((float*)matC_roc, C_size, (float*)matC_ict, C_size);
  return 0;
}
