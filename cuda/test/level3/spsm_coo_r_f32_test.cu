
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_coo.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;
alpha_matrix_descr descrT;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
float* coo_values;

// parms for kernel
float *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
long long B_cols;
long long ldb, ldc;
long long B_size, C_size;
const float alpha = 2.f;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));
  hipsparseSpSMDescr_t spsmDescr;
  CHECK_CUSPARSE(hipsparseSpSM_createDescr(&spsmDescr))
  // Offload data to device
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  int nnz = rnnz;

  float* dmatB = NULL;
  float* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(float) * B_size);
  hipMalloc((void**)&dmatC, sizeof(float) * C_size);
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  hipMemcpy(dmatB, hmatB, sizeof(float) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(float) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, HIP_R_32F, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_R_32F, HIPSPARSE_ORDER_COL))
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCoo(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dArow,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));

  hipsparseFillMode_t fillmode;
  hipsparseDiagType_t diagtype;
  if (descrT.mode == ALPHA_SPARSE_FILL_MODE_LOWER)
    fillmode = HIPSPARSE_FILL_MODE_LOWER;
  else
    fillmode = HIPSPARSE_FILL_MODE_UPPER;
  if (descrT.diag == ALPHA_SPARSE_DIAG_NON_UNIT)
    diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  else
    diagtype = HIPSPARSE_DIAG_TYPE_UNIT;
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(
    matA, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(fillmode)))
  // Specify Unit|Non-Unit diagonal type.
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(
    matA, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype)))
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpSM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         matB,
                                         matC,
                                         HIP_R_32F,
                                         HIPSPARSE_SPSM_ALG_DEFAULT,
                                         spsmDescr,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpSM_analysis(handle,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha,
                                       matA,
                                       matB,
                                       matC,
                                       HIP_R_32F,
                                       HIPSPARSE_SPSM_ALG_DEFAULT,
                                       spsmDescr,
                                       dBuffer))
  CHECK_CUSPARSE(hipsparseSpSM_solve(handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha,
                                    matA,
                                    matB,
                                    matC,
                                    HIP_R_32F,
                                    HIPSPARSE_SPSM_ALG_DEFAULT,
                                    spsmDescr))
  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(float), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);
  alphasparseSpSMDescr_t spsmDescr;
  alphasparseSpSM_createDescr(&spsmDescr);
  // Offload data to device
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  int nnz = rnnz;

  float* dmatB = NULL;
  float* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(float) * B_size);
  hipMalloc((void**)&dmatC, sizeof(float) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));

  hipMemcpy(dmatB, hmatB, sizeof(float) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(float) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, ALPHA_R_32F, ALPHASPARSE_ORDER_COL);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_R_32F, ALPHASPARSE_ORDER_COL);
  alphasparseSpMatDescr_t matA;
  alphasparseCreateCoo(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dArow,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_32F);
  // Specify Lower|Upper fill mode.
  alphasparseSpMatSetAttribute(
    matA, ALPHASPARSE_SPMAT_FILL_MODE, &descrT.mode, sizeof(descrT.mode));
  // Specify Unit|Non-Unit diagonal type.
  alphasparseSpMatSetAttribute(
    matA, ALPHASPARSE_SPMAT_DIAG_TYPE, &descrT.diag, sizeof(descrT.diag));
  size_t bufferSize = 0;
  alphasparseSpSM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             matA,
                             matB,
                             matC,
                             ALPHA_R_32F,
                             ALPHASPARSE_SPSM_ALG_DEFAULT,
                             spsmDescr,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  alphasparseSpSM_solve(handle,
                        ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                        ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        matA,
                        matB,
                        matC,
                        ALPHA_R_32F,
                        ALPHASPARSE_SPSM_ALG_DEFAULT,
                        spsmDescr);
  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(float), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  transB = alpha_args_get_transB(argc, argv);
  descrT = alpha_args_get_matrix_descrA(argc, argv);

  // read coo
  alpha_read_coo<float>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, float>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = 1024; // 默认C是方阵
  C_rows = A_rows;
  C_cols = columns;
  B_cols = columns;
  ldb = A_cols;
  ldc = C_rows;
  B_size = ldb * B_cols;
  C_size = ldc * B_cols;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_row_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_col_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;
  // init x y
  // init B C
  hmatB = (float*)alpha_malloc(B_size * sizeof(float));
  matC_ict = (float*)alpha_malloc(C_size * sizeof(float));
  matC_roc = (float*)alpha_malloc(C_size * sizeof(float));
  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_fill_random(matC_roc, 1, C_size);
  cuda_mm();
  alpha_mm();

  for (int i = 0; i < 20; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  check((float*)matC_roc, C_size, (float*)matC_ict, C_size);
  return 0;
}
