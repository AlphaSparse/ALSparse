#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csc.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

hipsparseSpMMAlg_t cu_alg = HIPSPARSE_SPMM_ALG_DEFAULT;
alphasparseSpMMAlg_t alpha_alg = ALPHASPARSE_SPMM_CSR_ALG1;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
float* coo_values;

// parms for kernel
float *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
int B_rows;
long long ldb, ldc;
long long B_size, C_size;
const float alpha = 2.f;
const float beta = 3.f;

hipEvent_t event_start, event_stop;
float elapsed_time = 0.0;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  int* dCscColPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  int nnz = rnnz;

  float* dmatB = NULL;
  float* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(float) * B_size);
  hipMalloc((void**)&dmatC, sizeof(float) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCscColPtr, sizeof(int) * (A_cols + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dAcol, nnz, A_cols, dCscColPtr);

  hipMemcpy(dmatB, hmatB, sizeof(float) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(float) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, C_cols, ldb, dmatB, HIP_R_32F, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_R_32F, HIPSPARSE_ORDER_COL))
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCsc(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dCscColPtr,
                                   dArow,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));
  std::vector<double> times;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         matB,
                                         &beta,
                                         matC,
                                         HIP_R_32F,
                                         cu_alg,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpMM(handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              matA,
                              matB,
                              &beta,
                              matC,
                              HIP_R_32F,
                              cu_alg,
                              dBuffer))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  times.push_back(elapsed_time);
  printf("cusparse: %lf\n", get_avg_time(times));
  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(float), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  int* dCscColPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  int nnz = rnnz;

  float* dmatB = NULL;
  float* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(float) * B_size);
  hipMalloc((void**)&dmatC, sizeof(float) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCscColPtr, sizeof(int) * (A_cols + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dAcol, nnz, A_cols, dCscColPtr);

  hipMemcpy(dmatB, hmatB, sizeof(float) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(float) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, C_cols, ldb, dmatB, ALPHA_R_32F, ALPHASPARSE_ORDER_COL);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_R_32F, ALPHASPARSE_ORDER_COL);
  alphasparseSpMatDescr_t matA;
  alphasparseCreateCsc(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dCscColPtr,
                       dArow,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_32F);
  std::vector<double> times;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  size_t bufferSize = 0;
  alphasparseSpMM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             matA,
                             matB,
                             &beta,
                             matC,
                             ALPHA_R_32F,
                             alpha_alg,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  alphasparseSpMM(handle,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha,
                  matA,
                  matB,
                  &beta,
                  matC,
                  ALPHA_R_32F,
                  alpha_alg,
                  dBuffer);
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  times.push_back(elapsed_time);
  printf("alphasparse: %lf\n", get_avg_time(times));
  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(float), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);

  // read coo
  alpha_read_coo<float>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, float>(rnnz, coo_col_index, coo_row_index, coo_values);
  columns = args_get_cols(argc, argv, 256);
  C_rows = A_rows;
  C_cols = columns;
  B_rows = A_cols;
  ldb = B_rows;
  ldc = C_rows;
  B_size = ldb * C_cols;
  C_size = ldc * C_cols;
  // init x y
  // init B C
  hmatB = (float*)alpha_malloc(B_size * sizeof(float));
  matC_ict = (float*)alpha_malloc(C_size * sizeof(float));
  matC_roc = (float*)alpha_malloc(C_size * sizeof(float));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_fill_random(matC_roc, 1, C_size);
  // std::cout << "matc:" <<std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << matC_ict[i] << ", ";
  // }
  // std::cout << "matc:" << std::endl;
  cuda_mm();
  alpha_mm();

  for (int i = 0; i < 8; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 8; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  check((float*)matC_roc, C_size, (float*)matC_ict, C_size);
  return 0;
}
