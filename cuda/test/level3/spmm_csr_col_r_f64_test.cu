#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

hipsparseSpMMAlg_t cu_alg = HIPSPARSE_SPMM_ALG_DEFAULT;
alphasparseSpMMAlg_t alpha_alg = ALPHASPARSE_SPMM_CSR_ALG1;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
double* coo_values;

// parms for kernel
double *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
long long B_rows;
long long ldb, ldc;
long long B_size, C_size;
const double alpha = 2.f;
const double beta = 3.f;

hipEvent_t event_start, event_stop;
float elapsed_time = 0.0;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  double* dAval = NULL;

  int nnz = rnnz;

  double* dmatB = NULL;
  double* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(double) * B_size);
  hipMalloc((void**)&dmatC, sizeof(double) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(double) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(double) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, C_cols, ldb, dmatB, HIP_R_64F, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_R_64F, HIPSPARSE_ORDER_COL))
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dCsrRowPtr,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_64F));
  std::vector<double> times;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         matB,
                                         &beta,
                                         matC,
                                         HIP_R_64F,
                                         cu_alg,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpMM(handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              matA,
                              matB,
                              &beta,
                              matC,
                              HIP_R_64F,
                              cu_alg,
                              dBuffer))
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  times.push_back(elapsed_time);
  printf("cusparse: %lf\n", get_avg_time(times));
  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(double), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  double* dAval = NULL;

  int nnz = rnnz;

  double* dmatB = NULL;
  double* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(double) * B_size);
  hipMalloc((void**)&dmatC, sizeof(double) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(double) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(double) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, C_cols, ldb, dmatB, ALPHA_R_64F, ALPHASPARSE_ORDER_COL);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_R_64F, ALPHASPARSE_ORDER_COL);
  alphasparseSpMatDescr_t matA;
  alphasparseCreateCsr(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_64F);
  std::vector<double> times;
  GPU_TIMER_START(elapsed_time, event_start, event_stop);
  size_t bufferSize = 0;
  alphasparseSpMM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             matA,
                             matB,
                             &beta,
                             matC,
                             ALPHA_R_64F,
                             alpha_alg,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  alphasparseSpMM(handle,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha,
                  matA,
                  matB,
                  &beta,
                  matC,
                  ALPHA_R_64F,
                  alpha_alg,
                  dBuffer);
  GPU_TIMER_END(elapsed_time, event_start, event_stop);
  times.push_back(elapsed_time);
  printf("alphasparse: %lf\n", get_avg_time(times));
  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(double), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);

  // read coo
  alpha_read_coo<double>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, double>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, 256);
  C_rows = A_rows;
  C_cols = columns;
  B_rows = A_cols;
  ldb = B_rows;
  ldc = C_rows;
  B_size = ldb * C_cols;
  C_size = ldc * C_cols;
  // init x y
  // init B C
  hmatB = (double*)alpha_malloc(B_size * sizeof(double));
  matC_ict = (double*)alpha_malloc(C_size * sizeof(double));
  matC_roc = (double*)alpha_malloc(C_size * sizeof(double));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_fill_random(matC_roc, 1, C_size);
  // std::cout << "matc:" <<std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << matC_ict[i] << ", ";
  // }
  // std::cout << "matc:" << std::endl;
  cuda_mm();
  alpha_mm();

  for (int i = 0; i < 8; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 8; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  check((double*)matC_roc, C_size, (double*)matC_ict, C_size);
  return 0;
}
