#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const int iteration = 210;
const int warmup_times = 200;
const char *file, *metric_file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

double alpha_time;
std::vector<alphasparseSpMMAlg_t> alpha_alg_list = {ALPHASPARSE_SPMM_ALG_DEFAULT, ALPHASPARSE_SPMM_CSR_ALG1, ALPHASPARSE_SPMM_CSR_ALG2, ALPHASPARSE_SPMM_CSR_ALG3, ALPHASPARSE_SPMM_CSR_ALG4, ALPHASPARSE_SPMM_CSR_ALG5};
// std::vector<alphasparseSpMMAlg_t> alpha_alg_list = {ALPHASPARSE_SPMM_CSR_ALG2};
int alg_num;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
hipDoubleComplex* coo_values;

// parms for kernel
hipDoubleComplex *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
int B_rows;
long long ldb, ldc;
long long B_size, C_size;
const hipDoubleComplex alpha = {2.f, 3.f};
const hipDoubleComplex beta = {3.f, 2.f};

hipEvent_t event_start, event_stop;
float elapsed_time = 0.0;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  hipDoubleComplex* dAval = NULL;

  int nnz = rnnz;

  hipDoubleComplex* dmatB = NULL;
  hipDoubleComplex* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(hipDoubleComplex) * B_size);
  hipMalloc((void**)&dmatC, sizeof(hipDoubleComplex) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(hipDoubleComplex) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(hipDoubleComplex) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(hipDoubleComplex) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, C_cols, ldb, dmatB, ALPHA_C_64F, ALPHASPARSE_ORDER_ROW);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_C_64F, ALPHASPARSE_ORDER_ROW);
  alphasparseSpMatDescr_t matA;
  alphasparseCreateCsr(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_C_64F);
  auto alg = alpha_alg_list[alg_num];
  size_t bufferSize = 0;
  alphasparseSpMM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             matA,
                             matB,
                             &beta,
                             matC,
                             ALPHA_C_64F,
                             alg,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  std::vector<double> times;
  for (int i = 0; i < iteration; i++) {
    if ( i>= warmup_times) hipMemcpy(dmatC, matC_ict, sizeof(hipDoubleComplex) * C_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    GPU_TIMER_START(elapsed_time, event_start, event_stop);
    alphasparseSpMM(handle,
                    ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                    ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    matA,
                    matB,
                    &beta,
                    matC,
                    ALPHA_C_64F,
                    alg,
                    dBuffer);
    GPU_TIMER_END(elapsed_time, event_start, event_stop);
    if (i >= warmup_times)
      times.push_back(elapsed_time);
  }
  alpha_time = get_avg_time_2(times);
  printf("alphasparse %d: %lf ms\n", alg, alpha_time);

  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
	metric_file = args_save_metrics_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  alg_num = args_get_alg_num(argc, argv);

  // read coo
  alpha_read_coo<hipDoubleComplex>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, hipDoubleComplex>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, 256);
  C_rows = A_rows;
  C_cols = columns;
  B_rows = A_cols;
  ldb = columns;
  ldc = C_cols;
  B_size = ldb * B_rows;
  C_size = ldc * C_rows;
  // init x y
  // init B C
  hmatB = (hipDoubleComplex*)alpha_malloc(B_size * sizeof(hipDoubleComplex));
  matC_ict = (hipDoubleComplex*)alpha_malloc(C_size * sizeof(hipDoubleComplex));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  // std::cout << "matc:" <<std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << matC_ict[i] << ", ";
  // }
  // std::cout << "matc:" << std::endl;
  alpha_mm();

  std::ofstream filename(metric_file, std::ios::app);
  filename << file << "," << alpha_time << "\n";
  filename.close();
  return 0;
}
