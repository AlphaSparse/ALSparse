#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const int iteration = 210;
const int warmup_times = 200;
const char *file, *metric_file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

double alpha_time;
std::vector<alphasparseSpMMAlg_t> alpha_alg_list = {ALPHASPARSE_SPMM_ALG_DEFAULT, ALPHASPARSE_SPMM_CSR_ALG1, ALPHASPARSE_SPMM_CSR_ALG2, ALPHASPARSE_SPMM_CSR_ALG3, ALPHASPARSE_SPMM_CSR_ALG4, ALPHASPARSE_SPMM_CSR_ALG5};
// std::vector<alphasparseSpMMAlg_t> alpha_alg_list = {ALPHASPARSE_SPMM_CSR_ALG2};
int alg_num;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
double* coo_values;

// parms for kernel
double *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
int B_rows;
long long ldb, ldc;
long long B_size, C_size;
const double alpha = 2.f;
const double beta = 3.f;

hipEvent_t event_start, event_stop;
float elapsed_time = 0.0;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  double* dAval = NULL;

  int nnz = rnnz;

  double* dmatB = NULL;
  double* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(double) * B_size);
  hipMalloc((void**)&dmatC, sizeof(double) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(double) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(double) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, C_cols, ldb, dmatB, ALPHA_R_64F, ALPHASPARSE_ORDER_ROW);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_R_64F, ALPHASPARSE_ORDER_ROW);
  alphasparseSpMatDescr_t matA;
  alphasparseCreateCsr(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_64F);
  auto alg = alpha_alg_list[alg_num];
  size_t bufferSize = 0;
  alphasparseSpMM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             matA,
                             matB,
                             &beta,
                             matC,
                             ALPHA_R_64F,
                             alg,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  std::vector<double> times;
  for (int i = 0; i < iteration; i++) {
    if (i >= warmup_times) hipMemcpy(dmatC, matC_ict, sizeof(double) * C_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    GPU_TIMER_START(elapsed_time, event_start, event_stop);
    alphasparseSpMM(handle,
                    ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                    ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                    &alpha,
                    matA,
                    matB,
                    &beta,
                    matC,
                    ALPHA_R_32F,
                    alg,
                    dBuffer);
    GPU_TIMER_END(elapsed_time, event_start, event_stop);
    if (i >= warmup_times) times.push_back(elapsed_time);
  }
  alpha_time = get_avg_time_2(times);
  printf("alphasparse %d: %lf ms\n", alg, alpha_time);

  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(double), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
	metric_file = args_save_metrics_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  alg_num = args_get_alg_num(argc, argv);

  // read coo
  alpha_read_coo<double>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, double>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, 256);
  C_rows = A_rows;
  C_cols = columns;
  B_rows = A_cols;
  ldb = columns;
  ldc = C_cols;
  B_size = ldb * B_rows;
  C_size = ldc * C_rows;
  // init x y
  // init B C
  hmatB = (double*)alpha_malloc(B_size * sizeof(double));
  matC_ict = (double*)alpha_malloc(C_size * sizeof(double));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_mm();

  std::ofstream filename(metric_file, std::ios::app);
  filename << file << "," << alpha_time << "\n";
  filename.close();

  return 0;
}
