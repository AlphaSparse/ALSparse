#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const int iteration = 210;
const int warmup_times = 200;
const char *file, *metric_file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

double cu_time;
std::vector<hipsparseSpMMAlg_t> cu_alg_list = {HIPSPARSE_SPMM_ALG_DEFAULT, HIPSPARSE_SPMM_CSR_ALG1, HIPSPARSE_SPMM_CSR_ALG2, HIPSPARSE_SPMM_CSR_ALG3};
// std::vector<hipsparseSpMMAlg_t> cu_alg_list = {HIPSPARSE_SPMM_CSR_ALG2};
int alg_num;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
float* coo_values;

// parms for kernel
float *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
int B_rows;
long long ldb, ldc;
long long B_size, C_size;
const float alpha = 2.f;
const float beta = 3.f;

hipEvent_t event_start, event_stop;
float elapsed_time = 0.0;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  int nnz = rnnz;

  float* dmatB = NULL;
  float* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(float) * B_size);
  hipMalloc((void**)&dmatC, sizeof(float) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(float) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(float) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, C_cols, ldb, dmatB, HIP_R_32F, HIPSPARSE_ORDER_ROW))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_R_32F, HIPSPARSE_ORDER_ROW))
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dCsrRowPtr,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_32F));
  auto alg = cu_alg_list[alg_num];
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         matB,
                                         &beta,
                                         matC,
                                         HIP_R_32F,
                                         alg,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  std::vector<double> times;
  for (int i = 0; i < iteration; i++) {
    if (i >= warmup_times) hipMemcpy(dmatC, matC_roc, sizeof(float) * C_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    GPU_TIMER_START(elapsed_time, event_start, event_stop);
    CHECK_CUSPARSE(hipsparseSpMM(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha,
                                matA,
                                matB,
                                &beta,
                                matC,
                                HIP_R_32F,
                                alg,
                                dBuffer))
    GPU_TIMER_END(elapsed_time, event_start, event_stop);
    if (i >= warmup_times) times.push_back(elapsed_time);
  }
  cu_time = get_avg_time_2(times);
  printf("cusparse %d: %lf ms\n", alg, cu_time);

  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(float), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
	metric_file = args_save_metrics_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  alg_num = args_get_alg_num(argc, argv);

  // read coo
  alpha_read_coo<float>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, float>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, 256);
  C_rows = A_rows;
  C_cols = columns;
  B_rows = A_cols;
  ldb = columns;
  ldc = C_cols;
  B_size = ldb * B_rows;
  C_size = ldc * C_rows;
  // init x y
  // init B C
  hmatB = (float*)alpha_malloc(B_size * sizeof(float));
  matC_roc = (float*)alpha_malloc(C_size * sizeof(float));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_roc, 1, C_size);
  cuda_mm();

  std::ofstream filename(metric_file, std::ios::app);
  filename << file << "," << cu_time << "\n";
  filename.close();

  return 0;
}
