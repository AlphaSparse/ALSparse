#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const int iteration = 210;
const int warmup_times = 200;
const char *file, *metric_file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;

double cu_time;
std::vector<hipsparseSpMMAlg_t> cu_alg_list = {HIPSPARSE_SPMM_ALG_DEFAULT, HIPSPARSE_SPMM_CSR_ALG1, HIPSPARSE_SPMM_CSR_ALG2, HIPSPARSE_SPMM_CSR_ALG3};
// std::vector<hipsparseSpMMAlg_t> cu_alg_list = {HIPSPARSE_SPMM_CSR_ALG2};
int alg_num;


long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
double* coo_values;

// parms for kernel
double *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
long long B_rows;
long long ldb, ldc;
long long B_size, C_size;
const double alpha = 2.f;
const double beta = 3.f;

hipEvent_t event_start, event_stop;
float elapsed_time = 0.0;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  double* dAval = NULL;

  int nnz = rnnz;

  double* dmatB = NULL;
  double* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(double) * B_size);
  hipMalloc((void**)&dmatC, sizeof(double) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(double) * nnz));
  PRINT_IF_CUDA_ERROR(
    hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (A_rows + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(double), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, A_rows, dCsrRowPtr);

  hipMemcpy(dmatB, hmatB, sizeof(double) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(double) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, C_cols, ldb, dmatB, HIP_R_64F, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_R_64F, HIPSPARSE_ORDER_COL))
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dCsrRowPtr,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_R_64F));
  auto alg = cu_alg_list[alg_num];
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         matB,
                                         &beta,
                                         matC,
                                         HIP_R_64F,
                                         alg,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  std::vector<double> times;
  for (int i = 0; i < iteration; i++) {
    if (i >= warmup_times) hipMemcpy(dmatC, matC_roc, sizeof(double) * C_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    GPU_TIMER_START(elapsed_time, event_start, event_stop);
    CHECK_CUSPARSE(hipsparseSpMM(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha,
                                matA,
                                matB,
                                &beta,
                                matC,
                                HIP_R_64F,
                                alg,
                                dBuffer))
    GPU_TIMER_END(elapsed_time, event_start, event_stop);
    if (i >= warmup_times) times.push_back(elapsed_time);
  }
  cu_time = get_avg_time_2(times);
  printf("cusparse %d: %lf ms\n", alg, cu_time);

  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(double), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
	metric_file = args_save_metrics_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  alg_num = args_get_alg_num(argc, argv);

  // read coo
  alpha_read_coo<double>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, double>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = args_get_cols(argc, argv, 256);
  C_rows = A_rows;
  C_cols = columns;
  B_rows = A_cols;
  ldb = B_rows;
  ldc = C_rows;
  B_size = ldb * C_cols;
  C_size = ldc * C_cols;
  // init x y
  // init B C
  hmatB = (double*)alpha_malloc(B_size * sizeof(double));
  matC_roc = (double*)alpha_malloc(C_size * sizeof(double));

  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_roc, 1, C_size);

  cuda_mm();

  std::ofstream filename(metric_file, std::ios::app);
  filename << file << "," << cu_time << "\n";
  filename.close();

  return 0;
}
