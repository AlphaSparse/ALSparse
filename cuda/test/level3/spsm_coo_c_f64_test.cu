
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_coo.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseOperation_t transB;
alpha_matrix_descr descrT;

long long columns;
int A_rows, A_cols, rnnz;
int *coo_row_index, *coo_col_index;
hipDoubleComplex* coo_values;

// parms for kernel
hipDoubleComplex *hmatB, *matC_ict, *matC_roc;
long long C_rows, C_cols;
long long B_cols;
long long ldb, ldc;
long long B_size, C_size;
const hipDoubleComplex alpha = {2.f, 3.f};

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mm()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));
  hipsparseSpSMDescr_t spsmDescr;
  CHECK_CUSPARSE(hipsparseSpSM_createDescr(&spsmDescr))
  // Offload data to device
  int* dArow = NULL;
  int* dAcol = NULL;
  hipDoubleComplex* dAval = NULL;

  int nnz = rnnz;

  hipDoubleComplex* dmatB = NULL;
  hipDoubleComplex* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(hipDoubleComplex) * B_size);
  hipMalloc((void**)&dmatC, sizeof(hipDoubleComplex) * C_size);
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(hipDoubleComplex) * nnz));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
  hipMemcpy(dmatB, hmatB, sizeof(hipDoubleComplex) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_roc, sizeof(hipDoubleComplex) * C_size, hipMemcpyHostToDevice);
  hipsparseDnMatDescr_t matB, matC;
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, HIP_C_64F, HIPSPARSE_ORDER_COL))
  // Create dense matrix C
  CHECK_CUSPARSE(hipsparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, HIP_C_64F, HIPSPARSE_ORDER_COL))
  hipsparseSpMatDescr_t matA;
  CHECK_CUSPARSE(hipsparseCreateCoo(&matA,
                                   A_rows,
                                   A_cols,
                                   nnz,
                                   dArow,
                                   dAcol,
                                   dAval,
                                   HIPSPARSE_INDEX_32I,
                                   HIPSPARSE_INDEX_BASE_ZERO,
                                   HIP_C_64F));

  hipsparseFillMode_t fillmode;
  hipsparseDiagType_t diagtype;
  if (descrT.mode == ALPHA_SPARSE_FILL_MODE_LOWER)
    fillmode = HIPSPARSE_FILL_MODE_LOWER;
  else
    fillmode = HIPSPARSE_FILL_MODE_UPPER;
  if (descrT.diag == ALPHA_SPARSE_DIAG_NON_UNIT)
    diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  else
    diagtype = HIPSPARSE_DIAG_TYPE_UNIT;
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(
    matA, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof(fillmode)))
  // Specify Unit|Non-Unit diagonal type.
  CHECK_CUSPARSE(hipsparseSpMatSetAttribute(
    matA, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof(diagtype)))
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpSM_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         matA,
                                         matB,
                                         matC,
                                         HIP_C_64F,
                                         HIPSPARSE_SPSM_ALG_DEFAULT,
                                         spsmDescr,
                                         &bufferSize))
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpSM_analysis(handle,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                       &alpha,
                                       matA,
                                       matB,
                                       matC,
                                       HIP_C_64F,
                                       HIPSPARSE_SPSM_ALG_DEFAULT,
                                       spsmDescr,
                                       dBuffer))
  CHECK_CUSPARSE(hipsparseSpSM_solve(handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha,
                                    matA,
                                    matB,
                                    matC,
                                    HIP_C_64F,
                                    HIPSPARSE_SPSM_ALG_DEFAULT,
                                    spsmDescr))
  CHECK_CUDA(
    hipMemcpy(matC_roc, dmatC, C_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
  hipsparseDestroy(handle);
}

static void
alpha_mm()
{
  alphasparseHandle_t handle = NULL;
  initHandle(&handle);
  alphasparseGetHandle(&handle);
  alphasparseSpSMDescr_t spsmDescr;
  alphasparseSpSM_createDescr(&spsmDescr);
  // Offload data to device
  int* dArow = NULL;
  int* dAcol = NULL;
  hipDoubleComplex* dAval = NULL;

  int nnz = rnnz;

  hipDoubleComplex* dmatB = NULL;
  hipDoubleComplex* dmatC = NULL;

  hipMalloc((void**)&dmatB, sizeof(hipDoubleComplex) * B_size);
  hipMalloc((void**)&dmatC, sizeof(hipDoubleComplex) * C_size);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(hipDoubleComplex) * nnz));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  hipMemcpy(dmatB, hmatB, sizeof(hipDoubleComplex) * B_size, hipMemcpyHostToDevice);
  hipMemcpy(dmatC, matC_ict, sizeof(hipDoubleComplex) * C_size, hipMemcpyHostToDevice);
  alphasparseDnMatDescr_t matB, matC;
  alphasparseCreateDnMat(
    &matB, A_cols, B_cols, ldb, dmatB, ALPHA_C_64F, ALPHASPARSE_ORDER_COL);
  // Create dense matrix C
  alphasparseCreateDnMat(
    &matC, C_rows, C_cols, ldc, dmatC, ALPHA_C_64F, ALPHASPARSE_ORDER_COL);
  alphasparseSpMatDescr_t matA;
  alphasparseCreateCoo(&matA,
                       A_rows,
                       A_cols,
                       nnz,
                       dArow,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_C_64F);
  // Specify Lower|Upper fill mode.
  alphasparseSpMatSetAttribute(
    matA, ALPHASPARSE_SPMAT_FILL_MODE, &descrT.mode, sizeof(descrT.mode));
  // Specify Unit|Non-Unit diagonal type.
  alphasparseSpMatSetAttribute(
    matA, ALPHASPARSE_SPMAT_DIAG_TYPE, &descrT.diag, sizeof(descrT.diag));
  size_t bufferSize = 0;
  alphasparseSpSM_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             matA,
                             matB,
                             matC,
                             ALPHA_C_64F,
                             ALPHASPARSE_SPSM_ALG_DEFAULT,
                             spsmDescr,
                             &bufferSize);
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  alphasparseSpSM_solve(handle,
                        ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                        ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha,
                        matA,
                        matB,
                        matC,
                        ALPHA_C_64F,
                        ALPHASPARSE_SPSM_ALG_DEFAULT,
                        spsmDescr);
  CHECK_CUDA(
    hipMemcpy(matC_ict, dmatC, C_size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost))
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dmatB);
  hipFree(dmatC);
}

int
main(int argc, const char* argv[])
{
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  transB = alpha_args_get_transB(argc, argv);
  descrT = alpha_args_get_matrix_descrA(argc, argv);

  // read coo
  alpha_read_coo<hipDoubleComplex>(
    file, &A_rows, &A_cols, &rnnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, hipDoubleComplex>(rnnz, coo_row_index, coo_col_index, coo_values);
  columns = 1024; // 默认C是方阵
  C_rows = A_rows;
  C_cols = columns;
  B_cols = columns;
  ldb = A_cols;
  ldc = C_rows;
  B_size = ldb * B_cols;
  C_size = ldc * B_cols;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_row_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_col_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;
  // init x y
  // init B C
  hmatB = (hipDoubleComplex*)alpha_malloc(B_size * sizeof(hipDoubleComplex));
  matC_ict = (hipDoubleComplex*)alpha_malloc(C_size * sizeof(hipDoubleComplex));
  matC_roc = (hipDoubleComplex*)alpha_malloc(C_size * sizeof(hipDoubleComplex));
  alpha_fill_random(hmatB, 0, B_size);
  alpha_fill_random(matC_ict, 1, C_size);
  alpha_fill_random(matC_roc, 1, C_size);
  cuda_mm();
  alpha_mm();

  for (int i = 0; i < 20; i++) {
    std::cout << matC_roc[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << matC_ict[i] << ", ";
  }
  check((hipDoubleComplex*)matC_roc, C_size, (hipDoubleComplex*)matC_ict, C_size);
  return 0;
}
