#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "alphasparse.h"

bool check_flag;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int m = 16, n = 32, batchStride=16, batchCount, size;
hipFloatComplex *hdl, *hd, *hdu, *hict_x, *hcuda_x;
float error;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void
cuda_gtsv2()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  hipFloatComplex* ddl = NULL;
  hipFloatComplex* dd = NULL;
  hipFloatComplex* ddu = NULL;
  hipFloatComplex* dx = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dx, sizeof(hipFloatComplex) * size))

  // Copy data to device
  CHECK_CUDA(hipMemcpy(ddl, hdl, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dd, hd, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(ddu, hdu, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dx, hcuda_x, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  CHECK_CUSPARSE(hipsparseCgtsv2StridedBatch_bufferSizeExt(
    handle, m, ddl, dd, ddu, dx, batchCount, batchStride, &buffer_size))

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));
  CHECK_CUSPARSE(hipsparseCgtsv2StridedBatch(
    handle, m, ddl, dd, ddu, dx, batchCount, batchStride, temp_buffer));

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(
    hipMemcpy(hcuda_x, dx, sizeof(hipFloatComplex) * size, hipMemcpyDeviceToHost));

  CHECK_CUSPARSE(hipsparseDestroy(handle))
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dx))
  CHECK_CUDA(hipFree(temp_buffer))
}

void
alpha_gtsv2()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  hipFloatComplex* ddl = NULL;
  hipFloatComplex* dd = NULL;
  hipFloatComplex* ddu = NULL;
  hipFloatComplex* dx = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(hipFloatComplex) * size))
  CHECK_CUDA(hipMalloc((void**)&dx, sizeof(hipFloatComplex) * size))

  // Copy data to device
  CHECK_CUDA(hipMemcpy(ddl, hdl, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dd, hd, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(ddu, hdu, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dx, hict_x, sizeof(hipFloatComplex) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  alphasparseCgtsv2StridedBatch_bufferSizeExt(
    handle, m, ddl, dd, ddu, dx, batchCount, batchStride, &buffer_size);

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));

  alphasparseCgtsv2StridedBatch(
    handle, m, ddl, dd, ddu, dx, batchCount, batchStride, temp_buffer);

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(
    hipMemcpy(hict_x, dx, sizeof(hipFloatComplex) * size, hipMemcpyDeviceToHost));

  alphasparse_destory_handle(handle);
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dx))
  CHECK_CUDA(hipFree(temp_buffer))
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  m = args_get_rows(argc, argv, m);
  n = args_get_cols(argc, argv, n);
  batchStride = args_get_batch_stride(argc, argv, batchStride);
  assert(batchStride >= m);
  batchCount = n;
  size = batchCount * batchStride;
  hdl = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hd = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hdu = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hcuda_x = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));
  hict_x = (hipFloatComplex*)alpha_malloc(size * sizeof(hipFloatComplex));

  alpha_fill_random(hdl, 899, size);
  alpha_fill_random(hd, 101, size);
  alpha_fill_random(hdu, 77, size);
  alpha_fill_random(hcuda_x, 1, size);
  alpha_fill_random(hict_x, 1, size);
  for (size_t i = 0; i < m; ++i)
  {
    if (hd[i].x > 0)
      hd[i].x += 100.0;
    else
      hd[i].x -= 100.0;

    if (hd[i].y > 0)
      hd[i].y += 100.0;
    else
      hd[i].y -= 100.0;
  }
  for(int j = 0; j < batchCount; ++j)
  {
      hdl[j * batchStride + 0]     = {};
      hdu[j * batchStride + m - 1] = {};
  }
  std::cout << "===========hdl=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hdl[i] << ", ";
  }
  std::cout << std::endl << "===========hdl=============" << std::endl;
  std::cout << std::endl << "===========hd=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hd[i] << ", ";
  }
  std::cout << std::endl << "===========hd=============" << std::endl;
  std::cout << std::endl << "===========hdu=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hdu[i] << ", ";
  }
  std::cout << std::endl << "===========hdu=============" << std::endl;
  std::cout << std::endl << "===========hcuda_x=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcuda_x[i] << ", ";
  }
  std::cout << std::endl << "===========hcuda_x=============" << std::endl;
  std::cout << std::endl << "===========hict_x=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hict_x[i] << ", ";
  }
  std::cout << std::endl << "===========hict_x=============" << std::endl;

  cuda_gtsv2();
  alpha_gtsv2();

  std::cout << std::endl << "===========result=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcuda_x[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hict_x[i] << ", ";
  }
  std::cout << std::endl << "===========result=============" << std::endl;
  check(hict_x, size, hcuda_x, size, &error);

  return 0;
}
