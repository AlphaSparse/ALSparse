#include "hip/hip_runtime.h"
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#include "alphasparse.h"

bool check_flag;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int m = 32, n = 16, batchCount, batchStride, size;
double *hdl, *hd, *hdu, *hict_x, *hcuda_x, *hresult;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

void
test_gtsv2(double* hx)
{
  for (int j = 0; j < batchCount; j++) {
    int offset = batchStride * j;
    hresult[offset] =
      hd[offset + 0] * hx[offset] + hdu[offset + 0] * hx[offset + 1];
    hresult[offset + m - 1] = hdl[offset + m - 1] * hx[offset + m - 2] +
                              hd[offset + m - 1] * hx[offset + m - 1];
#ifdef _OPENMP
#pragma omp parallel for schedule(dynamic, 1024)
#endif
    for (int i = 1; i < m - 1; i++) {
      hresult[offset + i] = hdl[offset + i] * hx[offset + i - 1] +
                            hd[offset + i] * hx[offset + i] +
                            hdu[offset + i] * hx[offset + i + 1];
    }
  }
}

void
cuda_gtsv2()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  double* ddl = NULL;
  double* dd = NULL;
  double* ddu = NULL;
  double* dx = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(double) * size))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(double) * size))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(double) * size))
  CHECK_CUDA(hipMalloc((void**)&dx, sizeof(double) * size))

  // Copy data to device
  CHECK_CUDA(
    hipMemcpy(ddl, hdl, sizeof(double) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dd, hd, sizeof(double) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(ddu, hdu, sizeof(double) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dx, hcuda_x, sizeof(double) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  CHECK_CUSPARSE(hipsparseDgtsv2StridedBatch_bufferSizeExt(
    handle, m, ddl, dd, ddu, dx, batchCount, batchStride, &buffer_size))

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));
  CHECK_CUSPARSE(hipsparseDgtsv2StridedBatch(
    handle, m, ddl, dd, ddu, dx, batchCount, batchStride, temp_buffer));

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(
    hipMemcpy(hcuda_x, dx, sizeof(double) * size, hipMemcpyDeviceToHost));

  CHECK_CUSPARSE(hipsparseDestroy(handle))
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dx))
  CHECK_CUDA(hipFree(temp_buffer))
}

void
alpha_gtsv2()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  double* ddl = NULL;
  double* dd = NULL;
  double* ddu = NULL;
  double* dx = NULL;

  CHECK_CUDA(hipMalloc((void**)&ddl, sizeof(double) * size))
  CHECK_CUDA(hipMalloc((void**)&dd, sizeof(double) * size))
  CHECK_CUDA(hipMalloc((void**)&ddu, sizeof(double) * size))
  CHECK_CUDA(hipMalloc((void**)&dx, sizeof(double) * size))

  // Copy data to device
  CHECK_CUDA(
    hipMemcpy(ddl, hdl, sizeof(double) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(hipMemcpy(dd, hd, sizeof(double) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(ddu, hdu, sizeof(double) * size, hipMemcpyHostToDevice))
  CHECK_CUDA(
    hipMemcpy(dx, hict_x, sizeof(double) * size, hipMemcpyHostToDevice))

  // Obtain required buffer size
  size_t buffer_size;
  alphasparseDgtsv2StridedBatch_bufferSizeExt(
    handle, m, ddl, dd, ddu, dx, batchCount, batchStride, &buffer_size);

  void* temp_buffer;
  CHECK_CUDA(hipMalloc(&temp_buffer, buffer_size));

  alphasparseDgtsv2StridedBatch(
    handle, m, ddl, dd, ddu, dx, batchCount, batchStride, temp_buffer);

  // Device synchronization
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(
    hipMemcpy(hict_x, dx, sizeof(double) * size, hipMemcpyDeviceToHost));

  alphasparse_destory_handle(handle);
  CHECK_CUDA(hipFree(ddl))
  CHECK_CUDA(hipFree(dd))
  CHECK_CUDA(hipFree(ddu))
  CHECK_CUDA(hipFree(dx))
  CHECK_CUDA(hipFree(temp_buffer))
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  m = args_get_rows(argc, argv, m);
  n = args_get_cols(argc, argv, n);
  batchStride = m;
  batchCount = n;
  size = batchCount * batchStride;
  hdl = (double*)alpha_malloc(size * sizeof(double));
  hd = (double*)alpha_malloc(size * sizeof(double));
  hdu = (double*)alpha_malloc(size * sizeof(double));
  hcuda_x = (double*)alpha_malloc(size * sizeof(double));
  hict_x = (double*)alpha_malloc(size * sizeof(double));
  hresult = (double*)alpha_malloc(size * sizeof(double));
  double* hx_original = (double*)alpha_malloc(size * sizeof(double));

  alpha_fill_random(hdl, 899, size);
  alpha_fill_random(hd, 101, size);
  alpha_fill_random(hdu, 77, size);
  alpha_fill_random(hcuda_x, 1, size);
  alpha_fill_random(hict_x, 1, size);
  alpha_fill_random(hx_original, 1, size);
  for(int j = 0; j < batchCount; ++j)
  {
      hdl[j * batchStride + 0]     = {};
      hdu[j * batchStride + m - 1] = {};
  }
  std::cout << "===========hdl=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hdl[i] << ", ";
  }
  std::cout << std::endl << "===========hdl=============" << std::endl;
  std::cout << std::endl << "===========hd=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hd[i] << ", ";
  }
  std::cout << std::endl << "===========hd=============" << std::endl;
  std::cout << std::endl << "===========hdu=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hdu[i] << ", ";
  }
  std::cout << std::endl << "===========hdu=============" << std::endl;
  std::cout << std::endl << "===========hcuda_x=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcuda_x[i] << ", ";
  }
  std::cout << std::endl << "===========hcuda_x=============" << std::endl;
  std::cout << std::endl << "===========hict_x=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hict_x[i] << ", ";
  }
  std::cout << std::endl << "===========hict_x=============" << std::endl;

  cuda_gtsv2();
  alpha_gtsv2();
  // test_gtsv2(hict_x);

  std::cout << std::endl << "===========result=============" << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hcuda_x[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < min(20, size); i++) {
    std::cout << hict_x[i] << ", ";
  }
  std::cout << std::endl << "===========result=============" << std::endl;
  check(hict_x, size, hcuda_x, size);

  return 0;
}
