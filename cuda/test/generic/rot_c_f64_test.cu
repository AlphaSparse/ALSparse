#include "../test_common.h"

/**
 * @brief ict dcu mv csr test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include <alphasparse.h>

const char *file;
bool check_flag;
int iter;

// sparse vector
int nnz = 10000;
int *alpha_x_idx;
int *roc_x_idx;
hipDoubleComplex *x_val;
hipDoubleComplex *cuda_y, *alpha_y;
hipDoubleComplex c = {2., 3.}, s = {4., 5.};

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        exit(-1);                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(-1);                                                   \
    }                                                                          \
}

static void roc_rot() {
  // cusparse handle
  hipsparseHandle_t handle;
  CHECK_CUSPARSE( hipsparseCreate(&handle) )

  hipDeviceProp_t devProp;
  int device_id = 0;

  hipGetDevice(&device_id);
  hipGetDeviceProperties(&devProp, device_id);
  std::cout << "Device: " << devProp.name << std::endl;

  // Offload data to device
  int *dx_idx = NULL;
  hipDoubleComplex *dx_val = NULL;
  hipDoubleComplex *dy = NULL;

  hipMalloc((void **)&dx_idx, sizeof(int) * nnz);
  hipMalloc((void **)&dx_val, sizeof(hipDoubleComplex) * nnz);
  hipMalloc((void **)&dy, sizeof(hipDoubleComplex) * nnz * 20);

  hipMemcpy(dx_idx, roc_x_idx, sizeof(int) * nnz,
            hipMemcpyHostToDevice);
  hipMemcpy(dx_val, x_val, sizeof(hipDoubleComplex) * nnz, hipMemcpyHostToDevice);
  hipMemcpy(dy, cuda_y, sizeof(hipDoubleComplex) * nnz * 20, hipMemcpyHostToDevice);

hipsparseSpVecDescr_t x;
    hipsparseCreateSpVec(&x, nnz * 20, nnz, (void *)dx_idx,
                                (void *)dx_val, HIPSPARSE_INDEX_32I,
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIP_C_64F);

hipsparseDnVecDescr_t y;
hipsparseCreateDnVec(&y, nnz * 20, (void *)dy,
                            HIP_C_64F);

  // Call cusparse csrmv
  CHECK_CUSPARSE( hipsparseRot(handle, &c, &s, x, y) )

  // Device synchronization
  hipDeviceSynchronize();

  hipMemcpy(cuda_y, dy, sizeof(hipDoubleComplex) * nnz * 20, hipMemcpyDeviceToHost);

  // Clear up on device
  hipFree(dx_val);
  hipFree(dx_idx);
  hipFree(dy);
  hipsparseDestroy(handle);
}

static void alpha_rot()
{
    // cusparse handle
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    hipDoubleComplex *dx_val     = NULL;
    hipDoubleComplex *dy         = NULL;

    hipMalloc((void **)&dx_idx, sizeof(int) * nnz);
    hipMalloc((void **)&dx_val, sizeof(hipDoubleComplex) * nnz);
    hipMalloc((void **)&dy, sizeof(hipDoubleComplex) * nnz * 20);

    hipMemcpy(dx_idx, roc_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dx_val, x_val, sizeof(hipDoubleComplex) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(dy, alpha_y, sizeof(hipDoubleComplex) * nnz * 20, hipMemcpyHostToDevice);

    alphasparseSpVecDescr_t x{};
    alphasparseCreateSpVec(&x, nnz * 20,nnz,(void *)dx_idx,(void *)dx_val,ALPHA_SPARSE_INDEXTYPE_I32,ALPHA_SPARSE_INDEX_BASE_ZERO,ALPHA_C_64F);

    alphasparseDnVecDescr_t y{};
    alphasparseCreateDnVec(&y, nnz * 20,(void *)dy,ALPHA_C_64F);

    // Call cusparse csrmv
    alphasparseRot(handle, &c, &s, x, y);

    // Device synchronization
    hipDeviceSynchronize();

    hipMemcpy(alpha_y, dy, sizeof(hipDoubleComplex) * nnz * 20, hipMemcpyDeviceToHost);

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    alphasparse_destory_handle(handle);
}

int main(int argc, const char *argv[])
{
    // args
    args_help(argc, argv);
    file  = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    iter  = args_get_iter(argc, argv);
    nnz  = args_get_nnz(argc, argv);

    alpha_x_idx = (int *)alpha_memalign(sizeof(int) * nnz, DEFAULT_ALIGNMENT);
    roc_x_idx   = (int *)alpha_memalign(sizeof(int) * nnz,
                                                DEFAULT_ALIGNMENT);
    x_val       = (hipDoubleComplex *)alpha_memalign(sizeof(hipDoubleComplex) * nnz, DEFAULT_ALIGNMENT);
    cuda_y       = (hipDoubleComplex *)alpha_memalign(sizeof(hipDoubleComplex) * nnz * 20, DEFAULT_ALIGNMENT);
    alpha_y     = (hipDoubleComplex *)alpha_memalign(sizeof(hipDoubleComplex) * nnz * 20, DEFAULT_ALIGNMENT);

    alpha_fill_random(cuda_y, 1, nnz * 20);
    alpha_fill_random(alpha_y, 1, nnz * 20);
    alpha_fill_random(x_val, 0, nnz);

    for (int i = 0; i < nnz; i++) {
        alpha_x_idx[i] = i * 20;
        roc_x_idx[i]   = i * 20;
    }

    alpha_rot();

    if (check_flag) {
      roc_rot();
      check(cuda_y, nnz * 20, alpha_y, nnz * 20);
    }
    printf("\n");

    alpha_free(x_val);
    alpha_free(roc_x_idx);
    alpha_free(alpha_x_idx);
    alpha_free(cuda_y);
    alpha_free(alpha_y);
    return 0;
}
