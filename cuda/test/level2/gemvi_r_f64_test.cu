#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "alphasparse.h"

#include "../test_common.h"

const char *file;
bool check_flag;
int iter;

// sparse vector
int *alpha_x_idx;
int *cuda_x_idx;
double *x_val, *cuda_y, *alpha_y;
double alpha = 2.;
double beta = 3.;
alphasparseOperation_t transAT;

int m, n, nnz;
int *coo_row_index, *coo_col_index;
double* coo_values;
int lda;
int sizeA;
double * A;

int idx_n = 1000;

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        exit(-1);                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(-1);                                                   \
    }                                                                          \
}

static void cuda_gemvi() {
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    double *dx_val = NULL;
    double *dy = NULL;
    double *dA = NULL;

    hipsparseOperation_t transA;
    if(transAT == ALPHA_SPARSE_OPERATION_NON_TRANSPOSE) transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    else if(transAT == ALPHA_SPARSE_OPERATION_TRANSPOSE) transA = HIPSPARSE_OPERATION_TRANSPOSE;
    else transA = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

    CHECK_CUDA(hipMalloc((void **)&dx_idx, sizeof(int) * nnz));
    CHECK_CUDA(hipMalloc((void **)&dx_val, sizeof(double) * n));
    CHECK_CUDA(hipMalloc((void **)&dy, sizeof(double) * m));
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(double) * sizeA));

    CHECK_CUDA(hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dx_val, x_val, sizeof(double) * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dy, cuda_y, sizeof(double) * m, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA, A, sizeof(double) * sizeA, hipMemcpyHostToDevice));   

    int pBufferSize;
    void * pBuffer;
    hipsparseDgemvi_bufferSize(handle, transA, m, n, nnz, &pBufferSize);
    CHECK_CUDA(hipMalloc((void **)&pBuffer, sizeof(HIP_R_32F) * pBufferSize));
    hipsparseIndexBase_t base = HIPSPARSE_INDEX_BASE_ZERO;
    CHECK_CUSPARSE(hipsparseDgemvi(handle, transA, m, n, &alpha, dA, lda, nnz, dx_val, dx_idx, &beta, dy, base, pBuffer));

    // Device synchronization
    hipDeviceSynchronize();

    CHECK_CUDA(hipMemcpy(cuda_y, dy, sizeof(double) * m, hipMemcpyDeviceToHost));

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    hipFree(dA);
    hipsparseDestroy(handle);
}

static void alpha_gemvi()
{
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    // Offload data to device
    int *dx_idx = NULL;
    double *dx_val     = NULL;
    double *dy         = NULL;
    double *dA         = NULL;

    CHECK_CUDA(hipMalloc((void **)&dx_idx, sizeof(int) * nnz));
    CHECK_CUDA(hipMalloc((void **)&dx_val, sizeof(double) * n));
    CHECK_CUDA(hipMalloc((void **)&dy, sizeof(double) * m));
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(double) * sizeA));

    CHECK_CUDA(hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dx_val, x_val, sizeof(double) * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dy, alpha_y, sizeof(double) * m, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA, A, sizeof(double) * sizeA, hipMemcpyHostToDevice));

    int pBufferSize;
    void * pBuffer = NULL;
    hipMalloc((void **)&pBuffer, sizeof(double) );
    alphasparseIndexBase_t base = ALPHA_SPARSE_INDEX_BASE_ZERO;
    alphasparseDgemvi(handle, transAT, m, n, &alpha, dA, lda, nnz, dx_val, dx_idx, &beta, dy, base, pBuffer),

    // Device synchronization
    hipDeviceSynchronize();

    CHECK_CUDA(hipMemcpy(alpha_y, dy, sizeof(double) * m, hipMemcpyDeviceToHost));

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    hipFree(dA);
    // alphasparse_destory_handle(handle);
}

int main(int argc, const char *argv[])
{
    // args
    args_help(argc, argv);
    file = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    transAT = alpha_args_get_transA(argc, argv);

    alpha_read_coo<double>(
      file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);

    if(transAT != ALPHA_SPARSE_OPERATION_NON_TRANSPOSE) 
    {
      int t = n;
      n = m;
      m = t;
    }
    double spasity = (double)nnz / (m * n) ;
    nnz = m * spasity; 
    nnz = nnz > n ? n : nnz;
    lda = n;
    sizeA = m * lda;

    alpha_x_idx=(int *)alpha_memalign(sizeof(int) * nnz, DEFAULT_ALIGNMENT);
    cuda_x_idx = (int *)alpha_memalign(sizeof(int) * nnz,  DEFAULT_ALIGNMENT);

    x_val   = (double *)alpha_memalign(sizeof(double) * n, DEFAULT_ALIGNMENT);
    alpha_y = (double *)alpha_memalign(sizeof(double) * m, DEFAULT_ALIGNMENT);
    cuda_y  = (double *)alpha_memalign(sizeof(double) * m, DEFAULT_ALIGNMENT);
    A       = (double *)alpha_memalign(sizeof(double) * sizeA, DEFAULT_ALIGNMENT);

    alpha_fill_random(alpha_y, 1, m);
    memcpy(cuda_y, alpha_y, sizeof(double)* m);
    alpha_fill_random(x_val, 0, n);
    alpha_fill_random(A, 1, sizeA);
    int fen = n / nnz;
    for (int i = 0; i < nnz; i++) {
        if(i == 0)
        {
          alpha_x_idx[i] = rand() % fen;
          cuda_x_idx[i]  = alpha_x_idx[i];
        }
        else
        {
          alpha_x_idx[i] = alpha_x_idx[i - 1] + rand() % fen;
          cuda_x_idx[i]  = alpha_x_idx[i];
        }
    }

    alpha_gemvi();

    if (check_flag) {
        cuda_gemvi();
        check(alpha_y, m, cuda_y, m);

        for(int i=0;i<10;i++)
            std::cout<<alpha_y[i]<<"  ";

        std::cout<<std::endl;
        for(int i=0;i<10;i++)
            std::cout<<cuda_y[i]<<"  ";   

        std::cout<<std::endl;   
    }
    printf("\n");

    alpha_free(x_val);
    alpha_free(cuda_x_idx);
    alpha_free(alpha_x_idx);
    return 0;
}