
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/alphasparseXcsr2bsr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transAT;
alphasparseDirection_t dir_alpha;
struct alpha_matrix_descr descrT;

int m, n, nnz;
int* csrRowPtr = NULL;
int *coo_row_index, *coo_col_index;
hipFloatComplex* coo_values;

// coo format
hipFloatComplex* x_val;
hipFloatComplex* ict_y;
hipFloatComplex* cuda_y;

// parms for kernel
const hipFloatComplex alpha = {2.3f, 3.4f};

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_sv2()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  hipFloatComplex* dX = NULL;
  hipFloatComplex* dY = NULL;
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  hipFloatComplex* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(hipFloatComplex) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (m + 1)));
  
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);

  int blockDim = 2;
  hipsparseDirection_t dir;
  if(dir_alpha == ALPHA_SPARSE_LAYOUT_ROW_MAJOR) dir = HIPSPARSE_DIRECTION_ROW;
  else dir = HIPSPARSE_DIRECTION_COLUMN;
  hipFloatComplex* bsrValC = NULL;
  int* bsrRowPtrC = NULL;
  int* bsrColIndC = NULL;
  int nnzb; //base
  hipsparseMatDescr_t descrA;
  hipsparseMatDescr_t descrC;
  int mb = (m + blockDim-1)/blockDim;
  // int nb = (n + blockDim-1)/blockDim;
  hipsparseOperation_t transA;
  if(transAT == ALPHA_SPARSE_OPERATION_NON_TRANSPOSE) transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  else if(transAT == ALPHA_SPARSE_OPERATION_TRANSPOSE) transA = HIPSPARSE_OPERATION_TRANSPOSE;
  else transA = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrRowPtrC, sizeof(int) *(mb+1)));
  // nnzTotalDevHostPtr points to host memory
  // int *nnzTotalDevHostPtr = &nnzb;

  bsrsv2Info_t info = 0;
  int pBufferSize;
  void *pBuffer = 0;

  int structural_zero;
  int numerical_zero;
  const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;

  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
  CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
  CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));  
  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));

  CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(handle, dir, m, n,
                                    descrA, dCsrRowPtr, dAcol, blockDim,
                                    descrC, bsrRowPtrC, &nnzb));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrColIndC, sizeof(int)*nnzb));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrValC, sizeof(hipFloatComplex)*(blockDim*blockDim)*nnzb));
  CHECK_CUSPARSE(hipsparseCcsr2bsr(handle, dir, m, n,
                                  descrA, dAval, dCsrRowPtr, dAcol, blockDim,
                                  descrC, bsrValC, bsrRowPtrC, bsrColIndC));
  hipsparseFillMode_t FillMode;
  hipsparseDiagType_t DiagType;
  
  if(descrT.mode == ALPHA_SPARSE_FILL_MODE_LOWER) FillMode = HIPSPARSE_FILL_MODE_LOWER;
  else FillMode = HIPSPARSE_FILL_MODE_UPPER;
  if(descrT.diag == ALPHA_SPARSE_DIAG_NON_UNIT) DiagType = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  else DiagType = HIPSPARSE_DIAG_TYPE_UNIT;

  CHECK_CUSPARSE(hipsparseSetMatFillMode(descrC, FillMode));
  CHECK_CUSPARSE(hipsparseSetMatDiagType(descrC, DiagType));
  CHECK_CUSPARSE(hipsparseCreateBsrsv2Info(&info));

  hipsparseCbsrsv2_bufferSize(handle, dir, transA, mb, nnzb, descrC,
              bsrValC, bsrRowPtrC, bsrColIndC, blockDim, info, &pBufferSize);

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&pBuffer, pBufferSize));   
  
  CHECK_CUSPARSE(hipsparseCbsrsv2_analysis(handle, dir, transA, mb, nnzb, descrC,
                                    bsrValC, bsrRowPtrC, bsrColIndC, blockDim,
                                    info, policy, pBuffer));   

  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(hipFloatComplex)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(hipFloatComplex)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, cuda_y, m * sizeof(hipFloatComplex), hipMemcpyHostToDevice));    

  // L has unit diagonal, so no structural zero is reported.
  hipsparseStatus_t status = hipsparseXbsrsv2_zeroPivot(handle, info, &structural_zero);
  if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
    printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
  }
  // step 5: solve L*y = x
  CHECK_CUSPARSE(hipsparseCbsrsv2_solve(handle, dir, transA, mb, nnzb, &alpha, descrC,
                                  bsrValC, bsrRowPtrC, bsrColIndC, blockDim, info,
                                  dX, dY, policy, pBuffer));
  // L has unit diagonal, so no numerical zero is reported.
  status = hipsparseXbsrsv2_zeroPivot(handle, info, &numerical_zero);
  if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
    printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
  }     

  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(cuda_y, dY, sizeof(hipFloatComplex) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipFree(pBuffer);
  hipsparseDestroyBsrsv2Info(info);
  hipsparseDestroyMatDescr(descrA);  
  hipsparseDestroyMatDescr(descrC);  
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipFree(bsrValC);
  hipFree(bsrRowPtrC);
  hipFree(bsrColIndC);
  hipsparseDestroy(handle);
}

static void
alpha_sv2()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);
  hipsparseHandle_t chandle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&chandle));

  // Offload data to device
  hipFloatComplex* dX = NULL;
  hipFloatComplex* dY = NULL;
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  hipFloatComplex* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(hipFloatComplex) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (m + 1)));

  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);

  int blockDim = 2;
  hipsparseDirection_t dir;
  if(dir_alpha == ALPHA_SPARSE_LAYOUT_ROW_MAJOR) dir = HIPSPARSE_DIRECTION_ROW;
  else dir = HIPSPARSE_DIRECTION_COLUMN;
  hipFloatComplex* bsrValC = NULL;
  int* bsrRowPtrC = NULL;
  int* bsrColIndC = NULL;
  int nnzb; //base
  hipsparseMatDescr_t descrA;
  hipsparseMatDescr_t descrC;
  int mb = (m + blockDim-1)/blockDim;
  // int nb = (n + blockDim-1)/blockDim;
  alpha_bsrsv2Info_t info = ALPHA_SPARSE_OPAQUE;
  int pBufferSize = m;
  void *pBuffer = 0;
  const alphasparseSolvePolicy_t policy = ALPHA_SPARSE_SOLVE_POLICY_NO_LEVEL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrRowPtrC, sizeof(int) *(mb+1)));
  // nnzTotalDevHostPtr points to host memory
  // int *nnzTotalDevHostPtr = &nnzb;

  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
  CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
  CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));

  CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(chandle, dir, m, n,
                                    descrA, dCsrRowPtr, dAcol, blockDim,
                                    descrC, bsrRowPtrC, &nnzb));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrColIndC, sizeof(int)*nnzb));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrValC, sizeof(hipFloatComplex)*(blockDim*blockDim)*nnzb));
  hipsparseCcsr2bsr(chandle, dir, m, n,
                    descrA, dAval, dCsrRowPtr, dAcol, blockDim,
                    descrC, bsrValC, bsrRowPtrC, bsrColIndC);

  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(hipFloatComplex)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(hipFloatComplex)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, ict_y, m * sizeof(hipFloatComplex), hipMemcpyHostToDevice));

  alphasparseMatDescr_t descr_alpha ;
  alphasparseCreateMatDescr(&descr_alpha);
  alphasparseSetMatFillMode(descr_alpha, descrT.mode);
  alphasparseSetMatDiagType(descr_alpha, descrT.diag);

  if(descrT.diag == ALPHA_SPARSE_DIAG_UNIT) pBufferSize *= 2;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&pBuffer, sizeof(hipFloatComplex) * pBufferSize)); 
  hipMemset(pBuffer, 0, sizeof(hipFloatComplex) * pBufferSize);

  alphasparseCbsrsv2_solve(handle, dir_alpha, transAT, mb, nnzb, &alpha, descr_alpha,
                                  bsrValC, bsrRowPtrC, bsrColIndC, blockDim, info,
                                  dX, dY, policy, pBuffer);
  
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(ict_y, dY, sizeof(hipFloatComplex) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipsparseDestroyMatDescr(descrA);  
  hipsparseDestroyMatDescr(descrC);  
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipFree(bsrValC);
  hipFree(bsrRowPtrC);
  hipFree(bsrColIndC);
  hipFree(pBuffer);
  hipsparseDestroy(chandle);
  hipDeviceSynchronize();
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transAT = alpha_args_get_transA(argc, argv);
  descrT = alpha_args_get_matrix_descrA(argc, argv);
  dir_alpha = (alphasparseDirection_t)alpha_args_get_layout(argc, argv);

  // read coo
  alpha_read_coo<hipFloatComplex>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, hipFloatComplex>(nnz, coo_row_index, coo_col_index, coo_values);
  csrRowPtr = (int*)alpha_malloc(sizeof(int) * (m + 1));
  if (transAT == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transAT == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 3; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;
  // init x y
  x_val = (hipFloatComplex*)alpha_malloc(n * sizeof(hipFloatComplex));
  ict_y = (hipFloatComplex*)alpha_malloc(m * sizeof(hipFloatComplex));
  cuda_y = (hipFloatComplex*)alpha_malloc(m * sizeof(hipFloatComplex));

  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  cuda_sv2();
  alpha_sv2();
  check((hipFloatComplex*)cuda_y, m, (hipFloatComplex*)ict_y, m);
  for (int i = 0; i < 20; i++) {
    std::cout << cuda_y[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << ict_y[i] << ", ";
  }
  std::cout << std::endl;
  return 0;
}
