#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "alphasparse.h"

#include "../test_common.h"

const char *file;
bool check_flag;
int iter;

// sparse vector
int *alpha_x_idx;
int *cuda_x_idx;
hipFloatComplex *x_val, *cuda_y, *alpha_y;
const hipFloatComplex alpha = {1.1f,2.4f};
const hipFloatComplex beta = {3.2f,4.3f};
alphasparseOperation_t transAT;

int m, n, nnz;
int *coo_row_index, *coo_col_index;
hipFloatComplex* coo_values;
int lda;
int sizeA;
hipFloatComplex * A;

int idx_n = 1000;

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        exit(-1);                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        exit(-1);                                                   \
    }                                                                          \
}

static void cuda_gemvi() {
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    hipDeviceProp_t devProp;
    int device_id = 0;

    hipGetDevice(&device_id);
    hipGetDeviceProperties(&devProp, device_id);
    std::cout << "Device: " << devProp.name << std::endl;

    // Offload data to device
    int *dx_idx = NULL;
    hipFloatComplex *dx_val = NULL;
    hipFloatComplex *dy = NULL;
    hipFloatComplex *dA = NULL;

    hipsparseOperation_t transA;
    if(transAT == ALPHA_SPARSE_OPERATION_NON_TRANSPOSE) transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    else if(transAT == ALPHA_SPARSE_OPERATION_TRANSPOSE) transA = HIPSPARSE_OPERATION_TRANSPOSE;
    else transA = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

    CHECK_CUDA(hipMalloc((void **)&dx_idx, sizeof(int) * nnz));
    CHECK_CUDA(hipMalloc((void **)&dx_val, sizeof(hipFloatComplex) * n));
    CHECK_CUDA(hipMalloc((void **)&dy, sizeof(hipFloatComplex) * m));
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(hipFloatComplex) * sizeA));

    CHECK_CUDA(hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dx_val, x_val, sizeof(hipFloatComplex) * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dy, cuda_y, sizeof(hipFloatComplex) * m, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA, A, sizeof(hipFloatComplex) * sizeA, hipMemcpyHostToDevice));   

    int pBufferSize;
    void * pBuffer;
    hipsparseCgemvi_bufferSize(handle, transA, m, n, nnz, &pBufferSize);
    CHECK_CUDA(hipMalloc((void **)&pBuffer, sizeof(HIP_R_32F) * pBufferSize));
    hipsparseIndexBase_t base = HIPSPARSE_INDEX_BASE_ZERO;
    CHECK_CUSPARSE(hipsparseCgemvi(handle, transA, m, n, &alpha, dA, lda, nnz, dx_val, dx_idx, &beta, dy, base, pBuffer));

    // Device synchronization
    hipDeviceSynchronize();

    CHECK_CUDA(hipMemcpy(cuda_y, dy, sizeof(hipFloatComplex) * m, hipMemcpyDeviceToHost));

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    hipFree(dA);
    hipsparseDestroy(handle);
}

static void alpha_gemvi()
{
    alphasparseHandle_t handle;
    initHandle(&handle);
    alphasparseGetHandle(&handle);

    // Offload data to device
    int *dx_idx = NULL;
    hipFloatComplex *dx_val     = NULL;
    hipFloatComplex *dy         = NULL;
    hipFloatComplex *dA         = NULL;

    CHECK_CUDA(hipMalloc((void **)&dx_idx, sizeof(int) * nnz));
    CHECK_CUDA(hipMalloc((void **)&dx_val, sizeof(hipFloatComplex) * n));
    CHECK_CUDA(hipMalloc((void **)&dy, sizeof(hipFloatComplex) * m));
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(hipFloatComplex) * sizeA));

    CHECK_CUDA(hipMemcpy(dx_idx, alpha_x_idx, sizeof(int) * nnz, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dx_val, x_val, sizeof(hipFloatComplex) * n, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dy, alpha_y, sizeof(hipFloatComplex) * m, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA, A, sizeof(hipFloatComplex) * sizeA, hipMemcpyHostToDevice));

    int pBufferSize;
    void * pBuffer = NULL;
    hipMalloc((void **)&pBuffer, sizeof(hipFloatComplex) );
    alphasparseIndexBase_t base = ALPHA_SPARSE_INDEX_BASE_ZERO;
    alphasparseCgemvi(handle, transAT, m, n, &alpha, dA, lda, nnz, dx_val, dx_idx, &beta, dy, base, pBuffer),

    // Device synchronization
    hipDeviceSynchronize();

    CHECK_CUDA(hipMemcpy(alpha_y, dy, sizeof(hipFloatComplex) * m, hipMemcpyDeviceToHost));

    // Clear up on device
    hipFree(dx_val);
    hipFree(dx_idx);
    hipFree(dy);
    hipFree(dA);
    // alphasparse_destory_handle(handle);
}

int main(int argc, const char *argv[])
{
    // args
    args_help(argc, argv);
    file = args_get_data_file(argc, argv);
    check_flag = args_get_if_check(argc, argv);
    transAT = alpha_args_get_transA(argc, argv);

    alpha_read_coo<hipFloatComplex>(
      file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);

    if(transAT != ALPHA_SPARSE_OPERATION_NON_TRANSPOSE) 
    {
      int t = n;
      n = m;
      m = t;
    }
    float spasity = (float)nnz / (m * n) ;
    nnz = m * spasity; 
    nnz = nnz > n ? n : nnz;
    lda = n;
    sizeA = m * lda;

    alpha_x_idx=(int *)alpha_memalign(sizeof(int) * nnz, DEFAULT_ALIGNMENT);
    cuda_x_idx = (int *)alpha_memalign(sizeof(int) * nnz,  DEFAULT_ALIGNMENT);

    x_val   = (hipFloatComplex *)alpha_memalign(sizeof(hipFloatComplex) * n, DEFAULT_ALIGNMENT);
    alpha_y = (hipFloatComplex *)alpha_memalign(sizeof(hipFloatComplex) * m, DEFAULT_ALIGNMENT);
    cuda_y  = (hipFloatComplex *)alpha_memalign(sizeof(hipFloatComplex) * m, DEFAULT_ALIGNMENT);
    A       = (hipFloatComplex *)alpha_memalign(sizeof(hipFloatComplex) * sizeA, DEFAULT_ALIGNMENT);

    alpha_fill_random(alpha_y, 1, m);
    memcpy(cuda_y, alpha_y, sizeof(hipFloatComplex)* m);
    alpha_fill_random(x_val, 0, n);
    alpha_fill_random(A, 1, sizeA);
    int fen = n / nnz;
    for (int i = 0; i < nnz; i++) {
        if(i == 0)
        {
          alpha_x_idx[i] = rand() % fen;
          cuda_x_idx[i]  = alpha_x_idx[i];
        }
        else
        {
          alpha_x_idx[i] = alpha_x_idx[i - 1] + rand() % fen;
          cuda_x_idx[i]  = alpha_x_idx[i];
        }
    }

    alpha_gemvi();

    if (check_flag) {
        cuda_gemvi();
        check(alpha_y, m, cuda_y, m);

        for(int i=0;i<10;i++)
            std::cout<<alpha_y[i]<<"  ";

        std::cout<<std::endl;
        for(int i=0;i<10;i++)
            std::cout<<cuda_y[i]<<"  ";   

        std::cout<<std::endl;   
    }
    printf("\n");

    alpha_free(x_val);
    alpha_free(cuda_x_idx);
    alpha_free(alpha_x_idx);
    return 0;
}