
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/alphasparseXcsr2bsr.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;
alphasparseDirection_t dir_alpha;

int m, n, nnz;
int* csrRowPtr = NULL;
int *coo_row_index, *coo_col_index;
float* coo_values;

// coo format
float* x_val;
float* ict_y;
float* cuda_y;
int * h_bsrMaskPtr;
int sizeOfMask;

// parms for kernel
const float alpha = 2.3f;
const float beta = 3.4f;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mv()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  float* dX = NULL;
  float* dY = NULL;
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (m + 1)));
  
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);

  int blockDim = 2;
  hipsparseDirection_t dir;
  if(dir_alpha == ALPHA_SPARSE_LAYOUT_ROW_MAJOR) dir = HIPSPARSE_DIRECTION_ROW;
  else dir = HIPSPARSE_DIRECTION_COLUMN;

  float* bsrValC = NULL;
  int* bsrRowPtrC = NULL;
  int* bsrEndPtrC = NULL;
  int* h_bsrRowPtrC = NULL;
  int* bsrColIndC = NULL;
  int nnzb; //base
  int * bsrMaskPtr = NULL;
  hipsparseMatDescr_t descrA;
  hipsparseMatDescr_t descrC;
  int mb = (m + blockDim-1)/blockDim;
  int nb = (n + blockDim-1)/blockDim;
  hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrRowPtrC, sizeof(int) *(mb+1)));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrEndPtrC, sizeof(int) *(mb+1)));
  h_bsrRowPtrC = (int *)malloc(sizeof(int) *(mb+1));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrMaskPtr, sizeof(int) * sizeOfMask));
  // nnzTotalDevHostPtr points to host memory
  // int *nnzTotalDevHostPtr = &nnzb;
  CHECK_CUDA(hipMemcpy(bsrMaskPtr, h_bsrMaskPtr, sizeof(int) *sizeOfMask, hipMemcpyHostToDevice));

  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
  CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
  CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));

  CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(handle, dir, m, n,
                                    descrA, dCsrRowPtr, dAcol, blockDim,
                                    descrC, bsrRowPtrC, &nnzb));  

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrColIndC, sizeof(int)*nnzb));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrValC, sizeof(float)*(blockDim*blockDim)*nnzb));
  CHECK_CUSPARSE(hipsparseScsr2bsr(handle, dir, m, n,
                                  descrA, dAval, dCsrRowPtr, dAcol, blockDim,
                                  descrC, bsrValC, bsrRowPtrC, bsrColIndC));

  CHECK_CUDA(hipMemcpy(h_bsrRowPtrC, bsrRowPtrC, sizeof(int) *(mb+1), hipMemcpyDeviceToHost));

  for(int i = 0; i < mb + 1; i++)
  {
    h_bsrRowPtrC[i] += 1;
  }

  CHECK_CUDA(hipMemcpy(bsrEndPtrC, h_bsrRowPtrC, sizeof(int) *(mb+1), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(float)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, cuda_y, m * sizeof(float), hipMemcpyHostToDevice));  

  CHECK_CUSPARSE(hipsparseSbsrxmv(handle, dir, transA, sizeOfMask, mb, nb, nnzb, &alpha,
              descrC, bsrValC, bsrMaskPtr, bsrRowPtrC, bsrEndPtrC, bsrColIndC, blockDim, dX, &beta, dY));
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(cuda_y, dY, sizeof(float) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipsparseDestroyMatDescr(descrA);  
  hipsparseDestroyMatDescr(descrC);  
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipFree(bsrValC);
  hipFree(bsrRowPtrC);
  hipFree(bsrColIndC);
  hipsparseDestroy(handle);
}

static void
alpha_mv()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);
  hipsparseHandle_t chandle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&chandle));

  // Offload data to device
  float* dX = NULL;
  float* dY = NULL;
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  float* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(float) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (m + 1)));

  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(float), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);

  int blockDim = 2;
  hipsparseDirection_t dir;
  if(dir_alpha == ALPHA_SPARSE_LAYOUT_ROW_MAJOR) dir = HIPSPARSE_DIRECTION_ROW;
  else dir = HIPSPARSE_DIRECTION_COLUMN;
  float* bsrValC = NULL;
  int* bsrRowPtrC = NULL;
  int* bsrEndPtrC = NULL;
  int* h_bsrRowPtrC = NULL;
  int* bsrColIndC = NULL;
  int nnzb; //base  
  int * bsrMaskPtr = NULL;
  
  hipsparseMatDescr_t descrA;
  hipsparseMatDescr_t descrC;
  int mb = (m + blockDim-1)/blockDim;
  int nb = (n + blockDim-1)/blockDim;
  alphasparseOperation_t transA = ALPHA_SPARSE_OPERATION_NON_TRANSPOSE;
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrRowPtrC, sizeof(int) *(mb+1)));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrEndPtrC, sizeof(int) *(mb+1)));
  h_bsrRowPtrC = (int *)malloc(sizeof(int) *(mb+1));
  
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrMaskPtr, sizeof(int) * sizeOfMask));
  // nnzTotalDevHostPtr points to host memory
  // int *nnzTotalDevHostPtr = &nnzb;  

  CHECK_CUDA(hipMemcpy(bsrMaskPtr, h_bsrMaskPtr, sizeof(int) *sizeOfMask, hipMemcpyHostToDevice));

  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
  CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
  CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
  CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrC));

  CHECK_CUSPARSE(hipsparseXcsr2bsrNnz(chandle, dir, m, n,
                                    descrA, dCsrRowPtr, dAcol, blockDim,
                                    descrC, bsrRowPtrC, &nnzb));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrColIndC, sizeof(int)*nnzb));

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&bsrValC, sizeof(float)*(blockDim*blockDim)*nnzb));
  hipsparseScsr2bsr(chandle, dir, m, n,
                    descrA, dAval, dCsrRowPtr, dAcol, blockDim,
                    descrC, bsrValC, bsrRowPtrC, bsrColIndC);

  CHECK_CUDA(hipMemcpy(h_bsrRowPtrC, bsrRowPtrC, sizeof(int) *(mb+1), hipMemcpyDeviceToHost));

  for(int i = 0; i < mb + 1; i++)
  {
    h_bsrRowPtrC[i] += 1;
  }

  CHECK_CUDA(hipMemcpy(bsrEndPtrC, h_bsrRowPtrC, sizeof(int) *(mb+1), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(float)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(float)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, ict_y, m * sizeof(float), hipMemcpyHostToDevice));

  alphasparseMatDescr_t descr_alpha ;
  alphasparseCreateMatDescr(&descr_alpha);

  alphasparseSbsrxmv(handle, dir_alpha, transA, sizeOfMask, mb, nb, nnzb, &alpha,
              descr_alpha, bsrValC, bsrMaskPtr, bsrRowPtrC, bsrEndPtrC, bsrColIndC, blockDim, dX, &beta, dY);
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(ict_y, dY, sizeof(float) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipsparseDestroyMatDescr(descrA);  
  hipsparseDestroyMatDescr(descrC);  
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipFree(bsrValC);
  hipFree(bsrRowPtrC);
  hipFree(bsrColIndC);
  hipsparseDestroy(chandle);
  hipDeviceSynchronize();
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);
  dir_alpha = (alphasparseDirection_t)alpha_args_get_layout(argc, argv);

  // read coo
  alpha_read_coo<float>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, float>(nnz, coo_row_index, coo_col_index, coo_values);
  csrRowPtr = (int*)alpha_malloc(sizeof(int) * (m + 1));
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  for (int i = 0; i < 20; i++) {
    std::cout << coo_row_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_col_index[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << coo_values[i] << ", ";
  }
  std::cout << std::endl;
  // init x y
  x_val = (float*)alpha_malloc(n * sizeof(float));
  ict_y = (float*)alpha_malloc(m * sizeof(float));
  cuda_y = (float*)alpha_malloc(m * sizeof(float));

  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  sizeOfMask = (m / 2) / 4;
  h_bsrMaskPtr = (int *)malloc(sizeof(int) * sizeOfMask);
  for(int i = 0; i < sizeOfMask; i++)
  {
    if(i == 0) h_bsrMaskPtr[i] = rand() % 4;
    else h_bsrMaskPtr[i] = h_bsrMaskPtr[i - 1] + rand() % 4 + 1;
  }
  for(int i = 0; i < 5; i++)
    std::cout<<"mask "<<h_bsrMaskPtr[i]<<std::endl;

  cuda_mv();
  alpha_mv();
  check((float*)cuda_y, m, (float*)ict_y, m);
  for (int i = 0; i < 20; i++) {
    std::cout << cuda_y[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << ict_y[i] << ", ";
  }
  std::cout << std::endl;
  return 0;
}
