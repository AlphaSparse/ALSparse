
#include "../test_common.h"

/**
 * @brief ict dcu mv hyb test
 * @author HPCRC, ICT
 */

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#include <iomanip>
#include <iostream>
#include <vector>

#include "../../format/alphasparse_create_csr.h"
#include "../../format/csr2csc.h"
#include "../../format/coo2csr.h"
#include "../../format/coo_order.h"
#include "alphasparse.h"
#include <iostream>

const char* file;
int thread_num;
bool check_flag;
int iter;

alphasparseOperation_t transA;

int m, n, nnz;
int* csrRowPtr = NULL;
int *coo_row_index, *coo_col_index;
half* coo_values;

// coo format
half* x_val;
float* ict_y;
float* cuda_y;

// parms for kernel
const float alpha = 2.f;
const float beta = 3.f;

#define CHECK_CUDA(func)                                                       \
  {                                                                            \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
      printf("CUDA API failed at line %d with error: %s (%d)\n",               \
             __LINE__,                                                         \
             hipGetErrorString(status),                                       \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

#define CHECK_CUSPARSE(func)                                                   \
  {                                                                            \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n",           \
             __LINE__,                                                         \
             hipsparseGetErrorString(status),                                   \
             status);                                                          \
      exit(-1);                                                                \
    }                                                                          \
  }

static void
cuda_mv()
{
  hipsparseHandle_t handle = NULL;
  CHECK_CUSPARSE(hipsparseCreate(&handle));

  // Offload data to device
  half* dX = NULL;
  float* dY = NULL;
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  half* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (m + 1)));
  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);
  hipsparseDnVecDescr_t vecX, vecY;
  hipsparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(half)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(float)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(half), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, cuda_y, m * sizeof(float), hipMemcpyHostToDevice));
  // Create dense vector X
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, dX, HIP_R_16F));
  // Create dense vector y
  CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, m, dY, HIP_R_32F));
  int* dCscColPtr = NULL;
  int* dCscRowInd = NULL;
  half* dCscVal = NULL;
  size_t csc_bufferSize = 0;
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscRowInd, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscVal, sizeof(half) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCscColPtr, sizeof(int) * (n + 1)));
  hipsparseCsr2cscEx2_bufferSize(handle,
    m,
    n,
    nnz,
    dAval,
    dCsrRowPtr,
    dAcol,
    dCscVal,
    dCscColPtr,
    dCscRowInd,
    HIP_R_16F,
    HIPSPARSE_ACTION_NUMERIC,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIPSPARSE_CSR2CSC_ALG1,
    &csc_bufferSize);
  void* csc_dBuffer = NULL;
  CHECK_CUDA(hipMalloc((void**)&csc_dBuffer, csc_bufferSize));
  hipsparseCsr2cscEx2(handle,
    m,
    n,
    nnz,
    dAval,
    dCsrRowPtr,
    dAcol,
    dCscVal,
    dCscColPtr,
    dCscRowInd,
    HIP_R_16F,
    HIPSPARSE_ACTION_NUMERIC,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIPSPARSE_CSR2CSC_ALG1,
    csc_dBuffer);

  hipsparseSpMatDescr_t csc;
  hipsparseCreateCsc(&csc,
    m,
    n,
    nnz,
    dCscColPtr,
    dCscRowInd,
    dCscVal,
    HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_32I,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIP_R_16F);
  size_t bufferSize = 0;
  CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                         &alpha,
                                         csc,
                                         vecX,
                                         &beta,
                                         vecY,
                                         HIP_R_32F,
                                         HIPSPARSE_SPMV_ALG_DEFAULT,
                                         &bufferSize));
  void* dBuffer = NULL;
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
  CHECK_CUSPARSE(hipsparseSpMV(handle,
                              HIPSPARSE_OPERATION_NON_TRANSPOSE,
                              &alpha,
                              csc,
                              vecX,
                              &beta,
                              vecY,
                              HIP_R_32F,
                              HIPSPARSE_SPMV_ALG_DEFAULT,
                              dBuffer));
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(cuda_y, dY, sizeof(float) * m, hipMemcpyDeviceToHost));
  // Clear up on device
  hipFree(dArow);
  hipFree(dAcol);
  hipFree(dAval);
  hipFree(dX);
  hipFree(dY);
  hipsparseDestroy(handle);
}

static void
alpha_mv()
{
  alphasparseHandle_t handle;
  initHandle(&handle);
  alphasparseGetHandle(&handle);

  // Offload data to device
  half* dX = NULL;
  float* dY = NULL;
  int* dCsrRowPtr = NULL;
  int* dArow = NULL;
  int* dAcol = NULL;
  half* dAval = NULL;

  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dArow, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAcol, sizeof(int) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dAval, sizeof(half) * nnz));
  PRINT_IF_CUDA_ERROR(hipMalloc((void**)&dCsrRowPtr, sizeof(int) * (m + 1)));

  CHECK_CUDA(hipMemcpy(
    dArow, coo_row_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(
    dAcol, coo_col_index, nnz * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(
    hipMemcpy(dAval, coo_values, nnz * sizeof(half), hipMemcpyHostToDevice));
  alphasparseXcoo2csr(dArow, nnz, m, dCsrRowPtr);
  alphasparseDnVecDescr_t vecX, vecY;
  alphasparseSpMatDescr_t matA;
  CHECK_CUDA(hipMalloc((void**)&dX, n * sizeof(half)));
  CHECK_CUDA(hipMalloc((void**)&dY, m * sizeof(float)));
  CHECK_CUDA(hipMemcpy(dX, x_val, n * sizeof(half), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(dY, ict_y, m * sizeof(float), hipMemcpyHostToDevice));

  alphasparseDnVecDescr_t x{};
  alphasparseCreateDnVec(&x, n, (void*)dX, ALPHA_R_16F);

  alphasparseDnVecDescr_t y_ict{};
  alphasparseCreateDnVec(&y_ict, m, (void*)dY, ALPHA_R_32F);

  alphasparseSpMatDescr_t csr;
  alphasparseCreateCsr(&csr,
                       m,
                       n,
                       nnz,
                       dCsrRowPtr,
                       dAcol,
                       dAval,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEXTYPE_I32,
                       ALPHA_SPARSE_INDEX_BASE_ZERO,
                       ALPHA_R_16F);
  alphasparseSpMatDescr_t csc;
  alphasparseCsr2csc<int, half>(csr, csc);
  
  void* dBuffer = NULL;
  size_t bufferSize = 0;
  alphasparseSpMV_bufferSize(handle,
                             ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha,
                             csc,
                             x,
                             &beta,
                             y_ict,
                             ALPHA_R_32F,
                             ALPHA_SPARSE_SPMV_ALG_DEFAULT,
                             &bufferSize);
  CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
  alphasparseSpMV(handle,
                  ALPHA_SPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha,
                  csc,
                  x,
                  &beta,
                  y_ict,
                  ALPHA_R_32F,
                  ALPHA_SPARSE_SPMV_ALG_DEFAULT,
                  dBuffer);
  hipDeviceSynchronize();
  CHECK_CUDA(hipMemcpy(ict_y, dY, sizeof(float) * m, hipMemcpyDeviceToHost));
}

int
main(int argc, const char* argv[])
{
  // args
  args_help(argc, argv);
  file = args_get_data_file(argc, argv);
  check_flag = args_get_if_check(argc, argv);
  transA = alpha_args_get_transA(argc, argv);

  // read coo
  alpha_read_coo<half>(
    file, &m, &n, &nnz, &coo_row_index, &coo_col_index, &coo_values);
  coo_order<int32_t, half>(nnz, coo_row_index, coo_col_index, coo_values);
  csrRowPtr = (int*)alpha_malloc(sizeof(int) * (m + 1));
  if (transA == ALPHA_SPARSE_OPERATION_TRANSPOSE ||
      transA == ALPHA_SPARSE_OPERATION_CONJUGATE_TRANSPOSE) {
    int temp = n;
    n = m;
    m = temp;
  }
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_row_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_col_index[i] << ", ";
  // }
  // std::cout << std::endl;
  // for (int i = 0; i < 20; i++) {
  //   std::cout << coo_values[i] << ", ";
  // }
  // std::cout << std::endl;
  // init x y
  x_val = (half*)alpha_malloc(n * sizeof(half));
  ict_y = (float*)alpha_malloc(m * sizeof(float));
  cuda_y = (float*)alpha_malloc(m * sizeof(float));
  alpha_fill_random(x_val, 0, n);
  alpha_fill_random(ict_y, 1, m);
  alpha_fill_random(cuda_y, 1, m);
  for (int i = 0; i < 20; i++) {
    std::cout << cuda_y[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << ict_y[i] << ", ";
  }
  cuda_mv();
  alpha_mv();
  check((float*)cuda_y, m, (float*)ict_y, m);
  for (int i = 0; i < 20; i++) {
    std::cout << cuda_y[i] << ", ";
  }
  std::cout << std::endl;
  for (int i = 0; i < 20; i++) {
    std::cout << ict_y[i] << ", ";
  }
  return 0;
}
