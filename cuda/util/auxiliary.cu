#include "hip/hip_runtime.h"
#include <sstream>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_complex.h>
#include "alphasparse/util/auxiliary.h"

std::ostream& operator<<(std::ostream& out, const hipFloatComplex& z)
{
    std::stringstream ss;
    ss << '(' << z.x << ',' << z.y << ')';
    return out << ss.str();
}

std::ostream& operator<<(std::ostream& out, const hipDoubleComplex& z)
{
    std::stringstream ss;
    ss << '(' << z.x << ',' << z.y << ')';
    return out << ss.str();
}

std::ostream& operator<<(std::ostream& out, const half& z)
{
    std::stringstream ss;
    ss << __half2float(z);
    return out << ss.str();
}

std::ostream& operator<<(std::ostream& out, const half2& z)
{
    std::stringstream ss;
    ss << '(' << z.x << ',' << z.y << ')';
    return out << ss.str();
}

std::ostream& operator<<(std::ostream& out, const hip_bfloat16& z)
{
    std::stringstream ss;
    ss << __bfloat162float(z);
    return out << ss.str();
}

std::ostream& operator<<(std::ostream& out, const nv_bfloat162& z)
{
    std::stringstream ss;
    ss << '(' << z.x << ',' << z.y << ')';
    return out << ss.str();
}

std::ostream& operator<<(std::ostream& out, const int8_t& z)
{
    std::stringstream ss;
    ss << int(z);
    return out << ss.str();
}
